#include "hip/hip_runtime.h"
// decoder/simple-decoder.cc

// Copyright 2009-2011 Microsoft Corporation
//           2012-2013 Johns Hopkins University (author: Daniel Povey)

// See ../../COPYING for clarification regarding multiple authors
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//  http://www.apache.org/licenses/LICENSE-2.0
//
// THIS CODE IS PROVIDED *AS IS* BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, EITHER EXPRESS OR IMPLIED, INCLUDING WITHOUT LIMITATION ANY IMPLIED
// WARRANTIES OR CONDITIONS OF TITLE, FITNESS FOR A PARTICULAR PURPOSE,
// MERCHANTABLITY OR NON-INFRINGEMENT.
// See the Apache 2 License for the specific language governing permissions and
// limitations under the License.

#include "decoder/cuda-decoder.h"
#include "fstext/remove-eps-local.h"
#include <algorithm>
#include <nvToolsExt.h>
#include <hip/hip_runtime_api.h>
#include <float.h>
#include <math.h>
#include <hip/hip_cooperative_groups.h>
#include <hipcub/hipcub.hpp>

#define MEMADVISE

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
        hipError_t e=hipGetLastError();                                 \
        if(e!=hipSuccess) {                                              \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
    }

#define DIV_ROUND_UP(a,b) ((a+b-1)/b)
namespace kaldi {

  /***************************************CudaFst Implementation*****************************************/
  HOST DEVICE inline float CudaFst::Final(StateId state) const {
    #ifdef __CUDA_ARCH__
    return final_d[state];
    #else
    return final_h[state];
    #endif

  }
  void CudaFst::initialize(const fst::Fst<StdArc> &fst) {
    nvtxRangePushA("CudaFst constructor");
    bytes_cudaMalloc=0;
    //count states since Fst doesn't provide this functionality
    numStates=0;
    for( fst::StateIterator<fst::Fst<StdArc> > iter(fst); !iter.Done(); iter.Next()) {
      numStates++;
    }
    start=fst.Start();
    hipHostMalloc(&final_h,sizeof(float)*numStates);
    hipMalloc(&final_d,sizeof(float)*numStates);

    //allocate and initialize offset arrays
    e_offsets_h=(unsigned int *)malloc(sizeof(unsigned int)*(numStates+1));
    ne_offsets_h=(unsigned int *)malloc(sizeof(unsigned int)*(numStates+1));

    hipMalloc((void**)&e_offsets_d,sizeof(unsigned int)*(numStates+1)); bytes_cudaMalloc+=sizeof(unsigned int)*(numStates+1);
    hipMalloc((void**)&ne_offsets_d,sizeof(unsigned int)*(numStates+1)); bytes_cudaMalloc+=sizeof(unsigned int)*(numStates+1);

    memset(e_offsets_h,0,sizeof(unsigned int)*(numStates+1));
    memset(ne_offsets_h,0,sizeof(unsigned int)*(numStates+1));

    //iterate through states and arcs and count number of arcs per state
    e_count=0;
    ne_count=0;
    max_ilabel=0;

    for(int i=0;i<numStates;i++) {
      final_h[i]=fst.Final(i).Value();
      //count emmiting and non_emitting arcs
      for (fst::ArcIterator<fst::Fst<StdArc> > aiter(fst, i); !aiter.Done(); aiter.Next()) {
        StdArc arc = aiter.Value();
        int32 ilabel = arc.ilabel;
        int32 olabel = arc.olabel;

        if(ilabel>max_ilabel) {
          max_ilabel=ilabel;
        }

        if(ilabel!=0) { //emitting
          e_count++;
        } else { //non-emitting
          ne_count++;
        }
      }
      ne_offsets_h[i+1]=ne_count;
      e_offsets_h[i+1]=e_count;
    }

    //offset ne_offsets by the number of emitting arcs
    for(int i=0;i<numStates+1;i++) {
      e_offsets_h[i]+=1;          //add dummy arc at the beginingg.
      ne_offsets_h[i]+=e_count+1;   //add dummy arc and put e_arcs before
    }

    arc_count=e_count+ne_count+1;

    hipMemcpy(final_d,final_h,sizeof(float)*numStates,hipMemcpyHostToDevice);
    
    hipMemcpy(e_offsets_d,e_offsets_h,sizeof(unsigned int)*(numStates+1),hipMemcpyHostToDevice);
    hipMemcpy(ne_offsets_d,ne_offsets_h,sizeof(unsigned int)*(numStates+1),hipMemcpyHostToDevice);


    //Allocate non-zero arrays
    hipHostMalloc(&arc_weights_h,arc_count*sizeof(BaseFloat));
    hipHostMalloc(&arc_nextstates_h,arc_count*sizeof(StateId));
    hipHostMalloc(&arc_ilabels_h,arc_count*sizeof(int32));
    hipHostMalloc(&arc_olabels_h,arc_count*sizeof(int32));

    hipMalloc((void**)&arc_weights_d,arc_count*sizeof(BaseFloat));
    hipMalloc((void**)&arc_nextstates_d,arc_count*sizeof(StateId));
    hipMalloc((void**)&arc_ilabels_d,arc_count*sizeof(int32)); 

        //now populate arc data
    int e_idx=1;          //save room for dummy arc (so start at 1)
    int ne_idx=e_count+1; //starts where e_offsets ends

    //create dummy arc
    arc_weights_h[0]=StdWeight::One().Value();
    arc_nextstates_h[0]=fst.Start();
    arc_ilabels_h[0]=0;
    arc_olabels_h[0]=0;

    for(int i=0;i<numStates;i++) {
      //count emiting and non_emitting arcs

      for (fst::ArcIterator<fst::Fst<StdArc> > aiter(fst, i); !aiter.Done(); aiter.Next()) {
        StdArc arc = aiter.Value();
        int idx;
        if(arc.ilabel!=0) { //emitting
          idx=e_idx++;
        } else {
          idx=ne_idx++;
        }
        arc_weights_h[idx]=arc.weight.Value();
        arc_nextstates_h[idx]=arc.nextstate;
        arc_ilabels_h[idx]=arc.ilabel;
        arc_olabels_h[idx]=arc.olabel;
      }
    }

    hipMemcpy(arc_weights_d,arc_weights_h,arc_count*sizeof(BaseFloat),hipMemcpyHostToDevice);
    hipMemcpy(arc_nextstates_d,arc_nextstates_h,arc_count*sizeof(StateId),hipMemcpyHostToDevice);
    hipMemcpy(arc_ilabels_d,arc_ilabels_h, arc_count*sizeof(int32),hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    cudaCheckError();

    nvtxRangePop();
  }

  void CudaFst::finalize() {
    nvtxRangePushA("CudaFst destructor");
    hipHostFree(final_h);
    hipFree(final_d);
    free(e_offsets_h);
    free(ne_offsets_h);

    hipFree(e_offsets_d);
    hipFree(ne_offsets_d);

    hipHostFree(arc_weights_h);
    hipHostFree(arc_nextstates_h);
    hipHostFree(arc_ilabels_h);
    hipHostFree(arc_olabels_h);

    hipFree(arc_weights_d);
    hipFree(arc_nextstates_d);
    hipFree(arc_ilabels_d);
    nvtxRangePop();
  }

  /***************************************End CudaFst****************************************************/

  CudaDecoder::CudaDecoder(const CudaFst &fst, const CudaDecoderConfig &config): fst_(fst), beam_(config.beam),
  bytes_cudaMalloc(0), max_tokens(config.max_tokens) {
    printf("CudaDecoder2 Constructor\n");

    int max_token = config.max_tokens; // for CUB

    // Comments about variables are in the .h file

    hipStreamCreate(&compute_st);

    hipEventCreate(&q_token_from_narcs_evt);

    hipMalloc(&d_curr_token, sizeof(int));
    hipMalloc(&d_q_token_from, sizeof(int));
    hipMalloc(&d_q_token_to, sizeof(int));
    hipMalloc(&d_q_token_end, sizeof(int));

    hipMalloc(&d_q_token_from_narcs, sizeof(int));
  
    hipMalloc(&d_allToken, config.max_tokens * sizeof(StateId));
    hipMalloc(&d_allTokenInfo, config.max_tokens * sizeof(InfoToken));

    hipHostMalloc(&h_q_token_from_size, sizeof(int));  

    // TODO move back to params
    int max_token_frame = 5000000;
    // we could use same pointer
    hipMalloc(&d_degrees_scan, max_token_frame * sizeof(int));
    hipMalloc(&d_block_sums_scan, (max_token_frame / 256 + 2)* sizeof(int)); // TODO remove hardcoded
    hipMalloc(&d_q_arc_offset, max_token_frame * sizeof(int));

    hipMalloc(&loglikelihoods_d, sizeof(BaseFloat)*(fst_.max_ilabel+1));  

    hipMalloc(&d_state_cost,sizeof(BaseFloat)*fst_.numStates);

    hipHostMalloc(&h_reached_final, sizeof(int));
    hipHostMalloc(&h_q_token_from_narcs, sizeof(int));

    // TODO use directly pinned, no device mem
    // TODO hardcoded params
    hipMalloc(&d_reversed_path, 50000 * sizeof(int)); // TODO pinned
    h_reversed_path = (int*)malloc(50000 * sizeof(int));

    hipMalloc(&d_cutoff, sizeof(float));
    
    hipMalloc(&d_path_size, sizeof(int));
    hipMalloc(&d_n_CTA_done, sizeof(int));

    cudaCheckError();
  }

  CudaDecoder::~CudaDecoder() {
        printf("CUDA DECODER DESTRUCTOR\n");
      // TODO
  }

  void CudaDecoder::InitDecoding() {
    printf("CUDA DECODER InitDecoding\n");


    InitLookup();

    StateId start_state = fst_.Start();
    KALDI_ASSERT(start_state != fst::kNoStateId);

    cudaCheckError();
    InfoToken it_init;
    it_init.cost = StdWeight::One().Value();
    it_init.prev_token = INT_MIN;
    it_init.arc_idx = -1;

    hipMemcpy(d_allToken, &start_state, sizeof(StateId), hipMemcpyHostToDevice);
    hipMemcpy(d_allTokenInfo, &it_init, sizeof(InfoToken), hipMemcpyHostToDevice);

    // We simulate a regular execution for the first iteration
    hipMemcpy(&d_state_cost[start_state], &(it_init.cost), sizeof(BaseFloat), hipMemcpyHostToDevice);

    hipMemset(d_curr_token, 0, sizeof(int));
    hipMemset(d_q_token_from, 0, sizeof(int));

    // Init state is in queue
    int one = 1;
    hipMemcpy(d_q_token_to, &one, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_q_token_end, &one, sizeof(int), hipMemcpyHostToDevice);
    *h_q_token_from_size = 1;

    float cutoff = FLT_MAX;
    hipMemcpy(d_cutoff, &cutoff, sizeof(float), hipMemcpyHostToDevice);

    hipMemset(d_n_CTA_done, 0, sizeof(int));
    
    cudaCheckError();

    debug_max_narcs = 0;
    num_frames_decoded_ = 0;

    printf("CUDA DECODER InitDecoding 1/2\n");
    ProcessNonemitting();
    printf("CUDA DECODER InitDecoding 2/2\n");
 }


// Used to trigger the fire&forget version of atomicMin (only av for int/long)
__device__ int floatToOrderedInt(float floatVal) {

    int intVal = __float_as_int( floatVal );

    return (intVal >= 0 ) ? intVal : intVal ^ 0x7FFFFFFF;

}



__device__ float orderedIntToFloat(int intVal) {

    return __int_as_float( (intVal >= 0) ? intVal : intVal ^ 0x7FFFFFFF );

} 


// Used before first frame
__global__ void init_lookup_kernel(int *state_cost, int size) {
    for(int idx = blockIdx.x*blockDim.x + threadIdx.x;
            idx < size;
            idx += blockDim.x*gridDim.x) {
        state_cost[idx]  = floatToOrderedInt(FLT_MAX);
    }
}

void CudaDecoder::InitLookup() {
    int nstates = fst_.numStates;


    dim3 grid,block;
    block.x = 256;
    grid.x = DIV_ROUND_UP(nstates, block.x);

    init_lookup_kernel<<<grid,block,0,hipStreamPerThread>>>(d_state_cost, nstates);
}

typedef CudaDecoder::StateId StateId;

// Used to reset lookup table between frames
// Using the queue to reset only the values needed
// Also takes care of resetting cutof
// TODO rename to something like "ResetForNewFrame"
__global__ void reset_lookup_kernel(StateId *d_q, int *d_q_offset, int *d_q_end, int *state_cost, float *d_cutoff) {
    int q_offset = *d_q_offset;
    int q_end = *d_q_end; 

    for(int idx = q_offset + blockIdx.x*blockDim.x + threadIdx.x;
            idx < q_end;
            idx += blockDim.x*gridDim.x) {

        StateId state = d_q[idx];

        state_cost[state]  = floatToOrderedInt(FLT_MAX);
    }

    // Avoiding a kernel call just to reset the cutoff
    if(blockIdx.x == 0 && threadIdx.x == 0)
        *d_cutoff = FLT_MAX; 
}

void CudaDecoder::ResetLookup() {
    int size = *h_q_token_from_size;

    dim3 grid,block;
    block.x = 256;
    grid.x = DIV_ROUND_UP(size, block.x);

    reset_lookup_kernel<<<grid,block,0,compute_st>>>(d_allToken, d_q_token_from, d_q_token_to, d_state_cost, d_cutoff);
}


void CudaDecoder::AdvanceDecoding(DecodableInterface *decodable,
        int32 max_num_frames) {
    printf("AdvanceDecoding\n");

    KALDI_ASSERT(num_frames_decoded_ >= 0 &&
        "You must call InitDecoding() before AdvanceDecoding()");
    int32 num_frames_ready = decodable->NumFramesReady();

    // num_frames_ready must be >= num_frames_decoded, or else
    // the number of frames ready must have decreased (which doesn't
    // make sense) or the decodable object changed between calls
    // (which isn't allowed).
    KALDI_ASSERT(num_frames_ready >= num_frames_decoded_);
    int32 target_frames_decoded = num_frames_ready;
    if (max_num_frames >= 0)
      target_frames_decoded = std::min(target_frames_decoded,
          num_frames_decoded_ + max_num_frames);

    ComputeLogLikelihoods(decodable);

    while (num_frames_decoded_ < target_frames_decoded) {
        //KALDI_LOG << "New frame";

        num_frames_decoded_++; 
        ComputeLogLikelihoods(decodable);

        //KALDI_LOG << "Emitting, frame=" << num_frames_decoded_;
        ProcessEmitting();

        //KALDI_LOG << "Non Emitting";
        ProcessNonemitting(); 


        if(num_frames_decoded_ > 3) {
            //KALDI_ASSERT(0); 
        }

        //computes log likelihoods for the next frame - check order
    }   


    printf("AdvanceDecoding Done\n");
    nvtxRangePop();
}


  void CudaDecoder::ComputeLogLikelihoods(DecodableInterface *decodable) {

    int32 frame = num_frames_decoded_;

    decodable->ComputeLogLikelihoods(loglikelihoods_d,frame,fst_.max_ilabel+1, compute_st);
  }


// Below that value, we launch the persistent kernel for NonEmitting
#define NONEM_LT_MAX_NARCS 4096
bool CudaDecoder::ProcessToken(unsigned int *d_arc_offsets,
                        bool is_emitting) {


    // Compute degrees, reduce by key, apply cutoff
    // Compute first part of the prefix sums of the degrees
    // At the end of that step, the kernel
    // set the value of h_q_token_from_narcs
    // (the number of arcs in the current queue processed)
    // TODO rename to something more explicit
    ComputeDegrees(d_arc_offsets);
    
    // Recording an event to signal h_q_token_from_narcs 
    // as ready to use 
    hipEventRecord(q_token_from_narcs_evt, compute_st);
            cudaCheckError();

    // last time we use the lookup for old_q is in compute degrees
    if(is_emitting)
        ResetLookup();

    // Finalize the scan 
    // partial scans + block offsets -> global scan
    // If we want to speed up the binary search in expand
    // This is where we can compute lower and upper bound 
    // on the fly
    FinalizeDegreesScan();
    
    // We need h_q_token_from_narcs to be ready
    hipEventSynchronize(q_token_from_narcs_evt);
    int h_old_q_narcs = *h_q_token_from_narcs;

    ExpandArcParams params;

    params.d_q = d_allToken; 
    params.d_q_info = d_allTokenInfo;

    params.d_q_token_from = d_q_token_from;
    params.d_q_token_to = d_q_token_to;
    params.d_q_token_end = d_q_token_end;

    params.d_degrees_scan = d_degrees_scan; 

    params.d_q_arc_offsets = d_q_arc_offset;
    params.arc_ilabels = fst_.arc_ilabels_d;
    params.d_q_token_from_narcs = d_q_token_from_narcs;
 
    params.arc_weights = fst_.arc_weights_d; 
    params.arc_nextstates = fst_.arc_nextstates_d; 
    params.d_cutoff = d_cutoff;
    params.beam = beam_;
    params.d_loglikelihoods= loglikelihoods_d;
    params.d_lookup = d_state_cost;
    params.is_emitting = is_emitting;

    params.d_curr_token = d_curr_token;
    params.h_q_token_from_size = h_q_token_from_size;
    params.d_n_CTA_done = d_n_CTA_done;

    bool done = false;

    if(h_old_q_narcs) {
        if(!params.is_emitting 
            && h_old_q_narcs < NONEM_LT_MAX_NARCS) { 
            NonEmittingLongTail(d_arc_offsets, params); 

            cudaCheckError();

            // Persistent kernel finishes the job
            done = true;
        }
        else {
            ExpandArcs(h_old_q_narcs, params);
        }

        hipStreamSynchronize(compute_st); 
    }

    cudaCheckError();
    return done;
}


void CudaDecoder::ProcessEmitting() {
    nvtxRangePushA("ProcessEmitting");
    
    // Using emitting arc offsets
    ProcessToken(fst_.e_offsets_d, true); 

    cudaCheckError();
    nvtxRangePop();
}

  void CudaDecoder::ProcessNonemitting() {
    nvtxRangePushA("ProcessNonemitting");

    // While not done, call it
    while(!ProcessToken(fst_.ne_offsets_d, false));

    cudaCheckError();
    nvtxRangePop();
  }


// TODO use struct for params, 
// large # of args slow things down

/*

This kernel is responsible for :

1) Read a token from the input queue [from, to[
2) Compute the outgoing degree of that token.next_state. For that :
   -> If that token is suboptimal (cutoff, best_cost), degree = 0
   -> Otherwise, we set degree using CSR graph

The distinction between emitting / non emitting depends on the argument passed
as "d_q_arc_offset"

3) Compute prefix sums of those degrees within the block :
    -> We store those "local prefix sums" in d_degrees_scan. Another kernel will finish the job
    -> We save the sum of all degrees in that block (block_sums)

4) The last block alive compute the prefix sums of block_sums. 
    -> We save it, it will be needed to compute global_scan
    -> We now have the total number of arcs overall, we save it to h_q_token_from_narcs

*/

#define COMPUTE_DEGREES_DIMX 256
  __global__ void compute_degrees_kernel(StateId *d_q, InfoToken *d_q_info, const int *d_q_token_from, const int
  *d_q_token_to, int *d_degrees_scan, unsigned int
  *d_offsets, int *d_state_cost, BaseFloat *d_cutoff, int *d_q_arc_offset,
  int *d_block_sums, int *d_block_sums_scan, int *h_q_token_from_narcs, int *d_q_token_from_narcs, int *d_n_CTA_done) {

       typedef hipcub::BlockScan<int, COMPUTE_DEGREES_DIMX> BlockScan;
       __shared__ typename BlockScan::TempStorage temp_storage;

       __shared__ int blk_scan_offset;
       __shared__ int is_last_CTA;


        int queue_offset = *d_q_token_from;
        int queue_end = *d_q_token_to;
        int queue_size = queue_end - queue_offset;

        BaseFloat cutoff = *d_cutoff;

        for(int block_offset = blockDim.x*blockIdx.x;
                block_offset < queue_size;
                block_offset += gridDim.x*blockDim.x) {
            int idx = queue_offset + block_offset + threadIdx.x;
            int degree = 0;

            if(idx < queue_end) {

                StateId state_idx = d_q[idx];
                BaseFloat cost = d_q_info[idx].cost;

                if(cost < cutoff) {
                    BaseFloat best_cost = orderedIntToFloat(d_state_cost[state_idx]);
                    if(cost == best_cost) {
                        int start = d_offsets[state_idx];
                        int end = d_offsets[state_idx+1];
                        degree = end - start;
                        d_q_arc_offset[idx-queue_offset] = start;
                    }
                }
            }

            int scan;
            BlockScan(temp_storage).ExclusiveSum(degree, scan);

            if(idx < queue_end)
                d_degrees_scan[idx-queue_offset] = scan;

            if(threadIdx.x == (COMPUTE_DEGREES_DIMX-1)) {
                d_block_sums[block_offset/COMPUTE_DEGREES_DIMX] = (scan + degree); // scan is exclusive 
            }

            if((block_offset + gridDim.x*blockDim.x) < queue_end) {
                // if there's another iteration, we'll reuse temp_storage
                __syncthreads();
            }
        }

        if(threadIdx.x == 0) {
            int old = atomicAdd(d_n_CTA_done, 1);
            blk_scan_offset = 0; // will be used if last CTA, avoiding a second sync
            is_last_CTA = (old == (gridDim.x -1));
        }

        __syncthreads(); // is_last_CTA + temp_storage reuse if last CTA

        if(is_last_CTA) {
                // The last block alive takes care of scan of block sums 
                __threadfence();
                if(threadIdx.x == 0) {
                    *d_n_CTA_done = 0;
                }

                // following value can be different than gridDim.x
                int total_blk_val = (queue_size + COMPUTE_DEGREES_DIMX -1) / COMPUTE_DEGREES_DIMX;

                for(int blk_idx_off = 0;
                    blk_idx_off < total_blk_val;
                    blk_idx_off += blockDim.x) {
                    int blk_idx = blk_idx_off + threadIdx.x;

                    int blk_sum = (blk_idx < total_blk_val) ? d_block_sums[blk_idx] : 0;

                    int blk_scan;
                    BlockScan(temp_storage).ExclusiveSum(blk_sum, blk_scan);
                    blk_scan += blk_scan_offset; 
                
                    if(blk_idx < total_blk_val) {
                        d_block_sums_scan[blk_idx] = blk_scan;
                    }
                    
                    if(threadIdx.x == (COMPUTE_DEGREES_DIMX-1)) {
                        int total = blk_scan + blk_sum;
                        blk_scan_offset = total;
                    }

                    __syncthreads(); // blk_scan_offset + reuse temp_storage
                }

            if(threadIdx.x == 0) {
                *d_q_token_from_narcs = blk_scan_offset; // pinned memory
                *h_q_token_from_narcs = blk_scan_offset; // pinned memory
            }
        }
  }

  void CudaDecoder::ComputeDegrees(unsigned int *d_offsets) {
    dim3 grid,block;
    block.x = COMPUTE_DEGREES_DIMX;
    grid.x = DIV_ROUND_UP(*h_q_token_from_size, block.x);

    compute_degrees_kernel<<<grid,block,0,compute_st>>>(d_allToken, d_allTokenInfo, d_q_token_from, d_q_token_to, d_degrees_scan,
    d_offsets, d_state_cost, d_cutoff, d_q_arc_offset, d_block_sums_scan, d_block_sums_scan, h_q_token_from_narcs,
    d_q_token_from_narcs, d_n_CTA_done);
  }


/*

Part 2 of the scan. Computes global prefix sum with block prefix sum and block offsets

If we want to speed up expand, we can compute lower and upper bound to restrain 
the binary search in expand
This can be done on the fly here, and removes main bottleneck of expand
Not done for now, because expand is fast enough

*/
 __global__ void finalize_degrees_scan_kernel(int *d_scan, int *d_blk_scan, const int *d_q_token_from, const int
  *d_q_token_to) {

        int q_off = *d_q_token_from;
        int q_end = *d_q_token_to;
        int q_size = q_end - q_off;

        for(int idx = blockDim.x*blockIdx.x + threadIdx.x;
                idx < q_size;
                idx += blockDim.x*gridDim.x) {

            int blk_idx = idx / blockDim.x;
            int blk_scan_offset = d_blk_scan[blk_idx]; // we rely on L1 for this one, avoiding syncs

            d_scan[idx] += blk_scan_offset;
        }

 }

  void CudaDecoder::FinalizeDegreesScan() {
      dim3 grid,block;
      block.x = COMPUTE_DEGREES_DIMX;
      grid.x = DIV_ROUND_UP(*h_q_token_from_size, block.x);

      finalize_degrees_scan_kernel<<<grid,block,0,compute_st>>>(d_degrees_scan, d_block_sums_scan, d_q_token_from, d_q_token_to); 
  }
    

__forceinline__ __device__ int binsearch_maxle(const int *vec, const int val, int low, int high) {
    while(true) {
        if(low == high)
            return low; //we know it exists
        if((low + 1) == high)
            return (vec[high] <= val) ? high : low;

        int mid = low + (high- low) / 2;

        if(vec[mid] > val)
            high = mid-1;
        else
            low = mid;
    }
}


// Temporary used for cutoff - will be removed
__device__ float fatomicMin(float *addr, float value)

{

    float old = *addr, assumed;
    if(old <= value) return old;

    do
    {
        assumed = old;
        old = atomicCAS((unsigned int*)addr,
                __float_as_int(assumed),
                __float_as_int(value));

    } while(old!=assumed);

    return old;

}

typedef CudaDecoder::ExpandArcParams ExpandArcParams; // TODO

#define EXPAND_ARCS_DIMX 256

/*

This kernel propagates arcs from the current queue [from,to[
to the new queue [to,end[

The main bottleneck is the first binary search. 
If we want to remove that bottleneck, cf comments on FinalizeScan


TODO merge reduce and scan for code simplicity + remove syncs

The last block alive moves the queues indexes :
new from is old to
new to is new end
new end stays new end


*/


void __global__ expand_arcs_kernel(ExpandArcParams params) {
    typedef hipcub::BlockScan<int, EXPAND_ARCS_DIMX> BlockScan;
    typedef hipcub::BlockReduce<BaseFloat, EXPAND_ARCS_DIMX> BlockReduce;
    
    __shared__ typename BlockScan::TempStorage temp_storage_scan;
    __shared__ typename BlockReduce::TempStorage temp_storage_reduce;

    __shared__ int new_q_block_off;
    __shared__ BaseFloat global_cutoff;
 
    const int total_narcs = *params.d_q_token_from_narcs;
    const int old_q_offset = *params.d_q_token_from;
    const int old_q_size = *params.d_q_token_to - old_q_offset;

    if(threadIdx.x == 0) {
        global_cutoff = *params.d_cutoff;
    }

    __syncthreads();
 
    // Keeping the whole CTA alive, we'll have syncs
    for(int block_offset = blockDim.x*blockIdx.x;
            block_offset < total_narcs;
            block_offset += gridDim.x*blockDim.x) {

        int th_idx = block_offset + threadIdx.x;
        bool valid_input = (th_idx < total_narcs);

        StateId prev_state;
        BaseFloat total_cost = FLT_MAX;
        int arc_idx;
        StateId arc_next_state;
        int q_idx;

        if(valid_input) {
            //we can do better than that
            q_idx = old_q_offset + binsearch_maxle(params.d_degrees_scan, th_idx, 0, old_q_size-1); 
            
            int lower_bound = params.d_degrees_scan[q_idx - old_q_offset];
            prev_state = params.d_q[q_idx];

            int arc_offset_start = params.d_q_arc_offsets[q_idx - old_q_offset];
            arc_idx = arc_offset_start + (block_offset + threadIdx.x - lower_bound);

            arc_next_state = params.arc_nextstates[arc_idx];
            BaseFloat arc_weight = params.arc_weights[arc_idx];
            
            int arc_ilabel = params.is_emitting ? params.arc_ilabels[arc_idx] : 0;

            BaseFloat accoustic_cost = (arc_ilabel != 0) ? -params.d_loglikelihoods[arc_ilabel] : 0.0; 
            BaseFloat next_state_cost = orderedIntToFloat(params.d_lookup[arc_next_state]);

            BaseFloat old_tok_cost = params.d_q_info[q_idx].cost;

            total_cost = accoustic_cost + arc_weight + old_tok_cost;

            if(total_cost >= next_state_cost) {
                total_cost = FLT_MAX;
                valid_input = false; 
            } 
        }
        
        BaseFloat thread_cutoff = (total_cost < FLT_MAX) ? (total_cost + params.beam) : FLT_MAX;
        BaseFloat new_block_cutoff = BlockReduce(temp_storage_reduce).Reduce(thread_cutoff, hipcub::Min());

        if(threadIdx.x == 0) {
            if(new_block_cutoff < global_cutoff) {
                BaseFloat new_global_cutoff = fatomicMin(params.d_cutoff, new_block_cutoff);
                new_global_cutoff = min(new_global_cutoff, new_block_cutoff);
                global_cutoff = new_global_cutoff;
            }
        }
        
        __syncthreads();

        BaseFloat cutoff = global_cutoff;

        int has_successor = (total_cost < cutoff && valid_input) ? 1 : 0;

        if(has_successor) {
            // reduce, not atomic (no return)
            atomicMin(&params.d_lookup[arc_next_state], floatToOrderedInt(total_cost));
        }

        int new_q_idx_block;

        BlockScan(temp_storage_scan).ExclusiveSum(has_successor, new_q_idx_block); // we could merge the reduce and
        //the scan

        
        //printf("thx=%i, next_state=%i, new_arc_idx=%i, arc_idx=%i \n", threadIdx.x, arc_next_state, new_q_idx_block,
        //arc_idx);

        if(threadIdx.x == (EXPAND_ARCS_DIMX - 1)) {
            int total_block = new_q_idx_block + has_successor; // exclusive sum
            new_q_block_off = atomicAdd(params.d_q_token_end, total_block);
        }

        __syncthreads(); // newQueue_block_off + we'll reuse temp_storage_scan + global cutoff

        int new_q_index = new_q_block_off + new_q_idx_block;

        if(has_successor) {
            params.d_q[new_q_index] = arc_next_state;

            InfoToken new_tok_info;
            new_tok_info.cost = total_cost;
            // Negative means we'll have to reindex at the end of advancedecoding
            new_tok_info.prev_token = q_idx;
            new_tok_info.arc_idx = arc_idx;
    
            params.d_q_info[new_q_index] = new_tok_info;

            //printf("Posted one NOT NULL tok=%i to %i, arc_idx=%i, with pred=%i (q=%i), cost=%f\n", new_q_index,
            //arc_next_state, arc_idx, prev_state, q_idx, total_cost);
        }
    }


    // Last block alive moves queue 

    if(threadIdx.x == 0) {
        int old = atomicAdd(params.d_n_CTA_done, 1);
        if(old == (gridDim.x -1)) {
            // The last block alive takes care of preparing for next iter
            __threadfence(); // we want last value of d_q_token_end
            int final_end = *params.d_q_token_end;

            *params.h_q_token_from_size = final_end - *params.d_q_token_to;

            *params.d_n_CTA_done = 0;
            *params.d_q_token_from = *params.d_q_token_to;
            *params.d_q_token_to = final_end;

            if(params.is_emitting) {
                // Saving position of curr_token for this frame
                // We'll need to reset d_q_token_from for next frame
                *params.d_curr_token = *params.d_q_token_from;
            }
        }
    }

}

void CudaDecoder::ExpandArcs(int nthreads, const ExpandArcParams &params) {
    dim3 grid,block;
    block.x = 256;
    grid.x = DIV_ROUND_UP(nthreads, block.x);

    expand_arcs_kernel<<<grid,block,0,compute_st>>>(params);
}



// Reached final kernel
__global__ void reached_final_kernel(StateId *d_q, const int *d_q_token_from, const int *d_q_token_to, BaseFloat *final, float fst_zero, int *h_reached_final) {
    int q_offset = *d_q_token_from;
    int q_end = *d_q_token_to;

    for(int idx = q_offset + blockDim.x*blockIdx.x + threadIdx.x;
            idx < q_end;
            idx += blockDim.x*gridDim.x) {

       StateId state = d_q[idx];
       float final_val = final[state]; 

       if(final_val != fst_zero) {
            *h_reached_final = 1; // we could exit
       }
    }

}

  bool CudaDecoder::ReachedFinal() const {
      dim3 grid, block;
      block.x = 256;
      grid.x = DIV_ROUND_UP(*h_q_token_from_size, block.x);

      reached_final_kernel<<<grid,block,0,hipStreamPerThread>>>(d_allToken, d_q_token_from, d_q_token_to, fst_.final_d, StdWeight::Zero().Value(), h_reached_final);
      hipDeviceSynchronize(); //TODO...

      return *h_reached_final;
  }



// Used to find best costs.
// TODO Needs to be rewritten

#define FILL_COSTS_DIMX 256
__global__ void fill_costs_kernel(StateId *d_q, InfoToken *d_q_it, const int *d_q_token_from, const int *d_q_token_to,
int *d_costs, BaseFloat *d_final, bool final) {
    int q_offset = *d_q_token_from;
    int q_end = *d_q_token_to;

    for(int idx = q_offset + blockIdx.x*blockDim.x + threadIdx.x;
            idx < q_end;
            idx += blockDim.x*gridDim.x) {
        BaseFloat cost = d_q_it[idx].cost;
        
        if(final) {
            StateId state = d_q[idx];
            cost += d_final[state];
        }
        
        //printf("idx=%i, final=%i, cost=%f \n", idx, final, cost);

        d_costs[idx-q_offset] = floatToOrderedInt(cost);
    }

}


void CudaDecoder::GetBestCost(BaseFloat *min, int *arg, bool isfinal) const {
    dim3 grid, block;
    block.x = FILL_COSTS_DIMX;

    grid.x = DIV_ROUND_UP(*h_q_token_from_size, block.x);

    // TODO using lookup as float buffer for now - NEED TO CHANGE
    fill_costs_kernel<<<grid,block,0,hipStreamPerThread>>>(d_allToken, d_allTokenInfo,
    d_q_token_from, d_q_token_to, d_state_cost, fst_.final_d, isfinal);

    hipcub::KeyValuePair<int, int> *d_argmin;
    hipMalloc(&d_argmin, sizeof(hipcub::KeyValuePair<int, int>));
    
    void *d_temp_storage_amin = NULL;
    size_t temp_storage_amin_bytes = 0;

    int max_t = max_tokens;
    hipcub::DeviceReduce::ArgMin(d_temp_storage_amin, temp_storage_amin_bytes, d_state_cost, d_argmin, *h_q_token_from_size);
    hipMalloc(&d_temp_storage_amin, temp_storage_amin_bytes);

    hipcub::DeviceReduce::ArgMin(d_temp_storage_amin, temp_storage_amin_bytes, d_state_cost, d_argmin, *h_q_token_from_size);

    hipcub::KeyValuePair<int, int> h_argmin;

    hipMemcpy(&h_argmin, d_argmin, sizeof(hipcub::KeyValuePair<int, int>), hipMemcpyDeviceToHost);
   

    hipFree(d_temp_storage_amin);
    hipFree(d_argmin);

    //InitLookup(); // reset lookup

    //*min = orderedIntToFloat(h_argmin.value);
    *min = -10; // TODO switch back to real value once new kernel ready
    *arg = h_argmin.key;
}

  BaseFloat CudaDecoder::FinalRelativeCost() const {
    if(*h_q_token_from_size == 0)
        return FLT_MAX;

      BaseFloat best_cost;
      int arg_best;
      GetBestCost(&best_cost, &arg_best, false);


      BaseFloat best_cost_final;
      int arg_best_final;
      GetBestCost(&best_cost_final, &arg_best_final, true);

      return (best_cost_final - best_cost);
  }

// brutal - one thread, multiple global memory load. But avoids a massive memcpy D2H
// Will disappear with better memory management 
void __global__ get_best_path_kernel(int best_token_idx_in_all_tokens, StateId *d_all_tokens, InfoToken
*d_all_tokens_info, int *d_reversed_path, int *path_size) {

    int tok_idx = best_token_idx_in_all_tokens;
    int idx = 0;

    //printf("start from %i \n", tok_idx);

    printf("backtrack = ");
    while(tok_idx != INT_MIN) {
        //printf("%i -> ", tok_idx);
        int state = d_all_tokens[tok_idx];
        int arc_idx = d_all_tokens_info[tok_idx].arc_idx;
        //printf("state=%i, tok=%i, arc=%i \n", state, tok_idx, arc_idx);
        //printf("at %i, arc=%i, state=%s  \n", idx, arc_idx, state);
        d_reversed_path[idx++] = arc_idx;

        int old_tok_idx = tok_idx; 
        tok_idx = d_all_tokens_info[tok_idx].prev_token;
        if(old_tok_idx <= tok_idx) 
            printf("FAIL\n");
    }
    
    printf("\n");

    *path_size = idx;
}

// Outputs an FST corresponding to the single best path
  // through the lattice.
  bool CudaDecoder::GetBestPath(Lattice *fst_out, bool use_final_probs) const {
      nvtxRangePushA("GetBestPath");

      BaseFloat best_cost;
      int arg_best;
      GetBestCost(&best_cost, &arg_best, false);

      BaseFloat best_cost_final;
      int arg_best_final;
      GetBestCost(&best_cost_final, &arg_best_final, true);

      bool isfinal = ReachedFinal();

      int h_curr_token_offset;
      hipMemcpy(&h_curr_token_offset, d_q_token_from, sizeof(int), hipMemcpyDeviceToHost);

      int h_best_token_idx = isfinal ? arg_best_final : arg_best; 
      h_best_token_idx += h_curr_token_offset;
  
    printf("is final = %i \n", isfinal);
    printf("curr token off=%i \n", h_curr_token_offset);
    printf("best token idx=%i \n", h_best_token_idx);
    printf("final costs : %f  final = %f \n", best_cost, best_cost_final);
    printf("final costs idx : %i  final idx = %i \n", arg_best, arg_best_final);

    hipMemset(d_path_size, 0, sizeof(int));

    get_best_path_kernel<<<1,1,0,hipStreamPerThread>>>(h_best_token_idx, d_allToken, d_allTokenInfo, d_reversed_path, d_path_size);

    hipDeviceSynchronize();

    printf("flush \n");
    
    int h_path_size;
    hipMemcpy(&h_path_size, d_path_size, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_reversed_path, d_reversed_path, h_path_size * sizeof(int), hipMemcpyDeviceToHost);
    

    fst_out->DeleteStates();
     
     // We can assert first state equals to root
    
    StateId cur_state = fst_out->AddState();
    fst_out->SetStart(cur_state);

    // -1 for 0-indexing, -1 for ignoring starting arc
    for (int i = h_path_size-1-1; i >= 1; i--) {
      int arc_idx = h_reversed_path[i];
      LatticeArc arc(fst_.arc_ilabels_h[arc_idx], fst_.arc_olabels_h[arc_idx], LatticeWeight(fst_.arc_weights_h[arc_idx], 0), fst_.arc_nextstates_h[arc_idx]);

      arc.nextstate = fst_out->AddState();
      fst_out->AddArc(cur_state, arc);
      cur_state = arc.nextstate;
    }

    if (isfinal && use_final_probs)
      fst_out->SetFinal(cur_state,
          LatticeWeight(fst_.Final(fst_.arc_nextstates_h[h_reversed_path[0]]), 0.0));
    else
      fst_out->SetFinal(cur_state, LatticeWeight::One());

    fst::RemoveEpsLocal(fst_out);

    nvtxRangePop();
      return true;
  }


// Wrote for single CTA

/*

Persistent kernel

Used to avoid calling multiple "heavy lifting" kernels for the tail of non emitting
(lots of iterations with small number of arcs)

Code is greatly simplified because we can have only one CTA alive

Repeat until new queue empty:
    1) Computes degrees (cf ComputeDegrees) 
    2) Compute scan
    3) Expand arcs

1 and 2 are not done on the first iteration, because it's already done
(by corresponding kernels)

At the end, this kernel finalize the computation for current frame,
setting the queue [from,to[ to the complete curr_token queue
so that it's ready for next ProcessEmitting

We could optimize and speed up this kernel
It will only gives us a better latency for 1 stream, which is low enough
Instead, we let it compute while we use the GPU for other streams
This kernel only uses one block, and is a free rider on the GPU

*/


#define NONEM_LT_DIMX 1024
__launch_bounds__(NONEM_LT_DIMX, 1)
__global__ void process_nonem_longtail(unsigned int *d_arc_offsets, 
                                ExpandArcParams params) {

    typedef hipcub::BlockScan<int, NONEM_LT_DIMX> BlockScan;
    typedef hipcub::BlockReduce<float, NONEM_LT_DIMX> BlockReduce;

    __shared__ typename BlockScan::TempStorage temp_storage_scan;
    __shared__ typename BlockReduce::TempStorage temp_storage_reduce;

    __shared__ BaseFloat cutoff;
    
    __shared__ int total_narcs;

    __shared__ int new_q_end;

    int old_q_offset = *params.d_q_token_from;
    int new_q_offset = *params.d_q_token_to;

    if(threadIdx.x == 0) {
        new_q_end = *params.d_q_token_end;
        total_narcs = *params.d_q_token_from_narcs;
    }

    __syncthreads();

    int old_q_size = new_q_offset - old_q_offset;  // move to end
    
    cutoff = *params.d_cutoff;
    
    // We'll switch queue at the beg of the loop
    // Cleaner that way - we need the offsets ready for
    // the global updates at the very end of this kernel
    new_q_offset = old_q_offset;
   
    bool first = true;

    while(old_q_size > 0) {
        // Step 0 : move queues        
        old_q_offset = new_q_offset;
        new_q_offset = new_q_end;

        if(!first) {

            if(threadIdx.x == 0)  {
                total_narcs = 0;
            }

            __syncthreads();


            // Step 1 : compute_degrees
            for(int local_q_idx = threadIdx.x;
                    local_q_idx < old_q_size;
                    local_q_idx += blockDim.x) {

                int global_q_idx = old_q_offset + local_q_idx;

                StateId state = params.d_q[global_q_idx];
                BaseFloat cost = params.d_q_info[global_q_idx].cost;

                int degree = 0;
                if(cost < cutoff) {
                    BaseFloat best_cost = orderedIntToFloat(params.d_lookup[state]);

                    if(cost == best_cost) {
                        int start = d_arc_offsets[state];
                        int end = d_arc_offsets[state+1];
                        degree = end - start;
                        params.d_q_arc_offsets[local_q_idx] = start;
                    }
                }

                params.d_degrees_scan[local_q_idx] = degree;
            }

            __syncthreads();

            // Step 2 : Scan

            for(int block_off = 0;
                    block_off < old_q_size;
                    block_off += blockDim.x) {

                int local_q_idx = block_off + threadIdx.x;

                int degree = (local_q_idx < old_q_size) 
                    ? params.d_degrees_scan[local_q_idx]
                    : 0;
                int lscan;
                BlockScan(temp_storage_scan).ExclusiveSum(degree, lscan);
                int scan = lscan + total_narcs;

                if(local_q_idx < old_q_size)
                    params.d_degrees_scan[local_q_idx] = scan;

                if(threadIdx.x == (NONEM_LT_DIMX-1)) {
                    int total_in_block = lscan + degree;
                    total_narcs += total_in_block;
                }

                __syncthreads();
            }

        } else {
            first = false;    
        }

        //if(threadIdx.x == 0)
        //    printf("narcs=%i \n", total_narcs);

        // We already sync'ed

        // Step 3 : expand arcs

        for(int block_offset = 0;
                block_offset < total_narcs;
                block_offset += blockDim.x) {

            int th_idx = block_offset + threadIdx.x;
            bool valid_input = (th_idx < total_narcs);

            BaseFloat total_cost = FLT_MAX;
            int arc_idx;
            StateId arc_next_state;
            int q_idx;

            if(valid_input) {
                //we can do better than that
                int local_q_idx = binsearch_maxle(params.d_degrees_scan, th_idx, 0, old_q_size-1); 

                //printf("thx=%i, q_idx=%i, oldqsize=%i, oldqoff=%i \n", threadIdx.x, q_idx, old_q_size, old_q_offset);

                int lower_bound = params.d_degrees_scan[local_q_idx];
                int arc_offset_start = params.d_q_arc_offsets[local_q_idx];
                q_idx = old_q_offset + local_q_idx;

                arc_idx = arc_offset_start + (th_idx - lower_bound);

                arc_next_state = params.arc_nextstates[arc_idx];
                BaseFloat arc_weight = params.arc_weights[arc_idx];
                BaseFloat next_state_cost = orderedIntToFloat(params.d_lookup[arc_next_state]);
                BaseFloat old_tok_cost = params.d_q_info[q_idx].cost;

                total_cost = arc_weight + old_tok_cost;

                if(total_cost >= next_state_cost) {
                    total_cost = FLT_MAX;
                    valid_input = false; 
                } 
            }

            BaseFloat thread_cutoff = (total_cost < FLT_MAX) ? (total_cost + params.beam) : FLT_MAX;
            BaseFloat new_block_cutoff = BlockReduce(temp_storage_reduce).Reduce(thread_cutoff, hipcub::Min());

            if(threadIdx.x == 0) {
                if(new_block_cutoff < cutoff) {
                    cutoff = new_block_cutoff;
                }
            }

            __syncthreads();

            int has_successor = (total_cost < cutoff && valid_input) ? 1 : 0;

            if(has_successor) 
                atomicMin(&params.d_lookup[arc_next_state], floatToOrderedInt(total_cost));
            

            int new_q_idx_block;

            BlockScan(temp_storage_scan).ExclusiveSum(has_successor, new_q_idx_block);

            if(has_successor) {
                int new_q_index = new_q_end + new_q_idx_block;
                params.d_q[new_q_index] = arc_next_state;

                InfoToken new_tok_info;
                new_tok_info.cost = total_cost;
                new_tok_info.prev_token = q_idx;
                new_tok_info.arc_idx = arc_idx;

                params.d_q_info[new_q_index] = new_tok_info;
 
            }

            if(threadIdx.x == (NONEM_LT_DIMX - 1)) {
                int total_in_block = new_q_idx_block + has_successor; // exclusive sum
                new_q_end += total_in_block;
            }
        }

        __syncthreads(); // new_q_end

        old_q_size = new_q_end - new_q_offset; 

    }

    if(threadIdx.x == 0) {
        // Next step is ProcessEmitting of next frame, from is currToken_offset
        *params.d_q_token_from = *params.d_curr_token; 
        *params.d_q_token_to = new_q_end;
        *params.d_q_token_end = new_q_end;
        *params.d_cutoff = cutoff;

        *params.h_q_token_from_size = new_q_end - *params.d_q_token_from;
    }

}
  
void CudaDecoder::NonEmittingLongTail(unsigned int *d_arc_offsets, 
                                const ExpandArcParams &params) {

    dim3 grid,block;
    block.x = NONEM_LT_DIMX;
    grid.x = 1; // it is designed for the long tail
    process_nonem_longtail<<<grid,block,0,compute_st>>>(d_arc_offsets, params);
}

} // end namespace kaldi.
