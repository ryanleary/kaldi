#include "hip/hip_runtime.h"
// decoder/simple-decoder.cc

// Copyright 2009-2011 Microsoft Corporation
//           2012-2013 Johns Hopkins University (author: Daniel Povey)

// See ../../COPYING for clarification regarding multiple authors
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//  http://www.apache.org/licenses/LICENSE-2.0
//
// THIS CODE IS PROVIDED *AS IS* BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, EITHER EXPRESS OR IMPLIED, INCLUDING WITHOUT LIMITATION ANY IMPLIED
// WARRANTIES OR CONDITIONS OF TITLE, FITNESS FOR A PARTICULAR PURPOSE,
// MERCHANTABLITY OR NON-INFRINGEMENT.
// See the Apache 2 License for the specific language governing permissions and
// limitations under the License.

#include "decoder/cuda-decoder.h"
#include "fstext/remove-eps-local.h"
#include <algorithm>
#include <nvToolsExt.h>
#include <hip/hip_runtime_api.h>
#include <float.h>
#include <math.h>

#include <hipcub/hipcub.hpp>

#define MEMADVISE

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
        hipError_t e=hipGetLastError();                                 \
        if(e!=hipSuccess) {                                              \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
    }

#define DIV_ROUND_UP(a,b) ((a+b-1)/b)
namespace kaldi {

    /***************************************CudaFst Implementation*****************************************/
    HOST DEVICE inline float CudaFst::Final(StateId state) const {
#ifdef __CUDA_ARCH__
        return final_d[state];
#else
        return final_h[state];
#endif

    }
    void CudaFst::initialize(const fst::Fst<StdArc> &fst) {
        nvtxRangePushA("CudaFst constructor");
        bytes_cudaMalloc=0;
        //count states since Fst doesn't provide this functionality
        numStates=0;
        for( fst::StateIterator<fst::Fst<StdArc> > iter(fst); !iter.Done(); iter.Next()) {
            numStates++;
        }
        start=fst.Start();
        hipHostMalloc(&final_h,sizeof(float)*numStates);
        hipMalloc(&final_d,sizeof(float)*numStates);

        //allocate and initialize offset arrays
        e_offsets_h=(unsigned int *)malloc(sizeof(unsigned int)*(numStates+1));
        ne_offsets_h=(unsigned int *)malloc(sizeof(unsigned int)*(numStates+1));

        hipMalloc((void**)&e_offsets_d,sizeof(unsigned int)*(numStates+1)); bytes_cudaMalloc+=sizeof(unsigned int)*(numStates+1);
        hipMalloc((void**)&ne_offsets_d,sizeof(unsigned int)*(numStates+1)); bytes_cudaMalloc+=sizeof(unsigned int)*(numStates+1);

        memset(e_offsets_h,0,sizeof(unsigned int)*(numStates+1));
        memset(ne_offsets_h,0,sizeof(unsigned int)*(numStates+1));

        //iterate through states and arcs and count number of arcs per state
        e_count=0;
        ne_count=0;
        max_ilabel=0;

        for(int i=0;i<numStates;i++) {
            final_h[i]=fst.Final(i).Value();
            //count emmiting and non_emitting arcs
            for (fst::ArcIterator<fst::Fst<StdArc> > aiter(fst, i); !aiter.Done(); aiter.Next()) {
                StdArc arc = aiter.Value();
                int32 ilabel = arc.ilabel;
                int32 olabel = arc.olabel;

                if(ilabel>max_ilabel) {
                    max_ilabel=ilabel;
                }

                if(ilabel!=0) { //emitting
                    e_count++;
                } else { //non-emitting
                    ne_count++;
                }
            }
            ne_offsets_h[i+1]=ne_count;
            e_offsets_h[i+1]=e_count;
        }

        //offset ne_offsets by the number of emitting arcs
        for(int i=0;i<numStates+1;i++) {
            e_offsets_h[i]+=1;          //add dummy arc at the beginingg.
            ne_offsets_h[i]+=e_count+1;   //add dummy arc and put e_arcs before
        }

        arc_count=e_count+ne_count+1;

        hipMemcpy(final_d,final_h,sizeof(float)*numStates,hipMemcpyHostToDevice);

        hipMemcpy(e_offsets_d,e_offsets_h,sizeof(unsigned int)*(numStates+1),hipMemcpyHostToDevice);
        hipMemcpy(ne_offsets_d,ne_offsets_h,sizeof(unsigned int)*(numStates+1),hipMemcpyHostToDevice);


        //Allocate non-zero arrays
        hipHostMalloc(&arc_weights_h,arc_count*sizeof(BaseFloat));
        hipHostMalloc(&arc_nextstates_h,arc_count*sizeof(StateId));
        hipHostMalloc(&arc_ilabels_h,arc_count*sizeof(int32));
        hipHostMalloc(&arc_olabels_h,arc_count*sizeof(int32));

        hipMalloc((void**)&arc_weights_d,arc_count*sizeof(BaseFloat));
        hipMalloc((void**)&arc_nextstates_d,arc_count*sizeof(StateId));
        hipMalloc((void**)&arc_ilabels_d,arc_count*sizeof(int32)); 

        //now populate arc data
        int e_idx=1;          //save room for dummy arc (so start at 1)
        int ne_idx=e_count+1; //starts where e_offsets ends

        //create dummy arc
        arc_weights_h[0]=StdWeight::One().Value();
        arc_nextstates_h[0]=fst.Start();
        arc_ilabels_h[0]=0;
        arc_olabels_h[0]=0;

        for(int i=0;i<numStates;i++) {
            //count emiting and non_emitting arcs

            for (fst::ArcIterator<fst::Fst<StdArc> > aiter(fst, i); !aiter.Done(); aiter.Next()) {
                StdArc arc = aiter.Value();
                int idx;
                if(arc.ilabel!=0) { //emitting
                    idx=e_idx++;
                } else {
                    idx=ne_idx++;
                }
                arc_weights_h[idx]=arc.weight.Value();
                arc_nextstates_h[idx]=arc.nextstate;
                arc_ilabels_h[idx]=arc.ilabel;
                arc_olabels_h[idx]=arc.olabel;
            }
        }

        hipMemcpy(arc_weights_d,arc_weights_h,arc_count*sizeof(BaseFloat),hipMemcpyHostToDevice);
        hipMemcpy(arc_nextstates_d,arc_nextstates_h,arc_count*sizeof(StateId),hipMemcpyHostToDevice);
        hipMemcpy(arc_ilabels_d,arc_ilabels_h, arc_count*sizeof(int32),hipMemcpyHostToDevice);


        hipDeviceSynchronize();
        cudaCheckError();

        nvtxRangePop();
    }

    void CudaFst::finalize() {
        nvtxRangePushA("CudaFst destructor");
        printf("CudaFst::finalize()\n");
        hipHostFree(final_h);
        hipFree(final_d);
        free(e_offsets_h);
        free(ne_offsets_h);

        hipFree(e_offsets_d);
        hipFree(ne_offsets_d);

        hipHostFree(arc_weights_h);
        hipHostFree(arc_nextstates_h);
        hipHostFree(arc_ilabels_h);
        hipHostFree(arc_olabels_h);

        hipFree(arc_weights_d);
        hipFree(arc_nextstates_d);
        hipFree(arc_ilabels_d);
        nvtxRangePop();
    }

    /***************************************End CudaFst****************************************************/

    CudaDecoder::CudaDecoder(const CudaFst &fst, const CudaDecoderConfig &config): fst_(fst), beam_(config.beam),
    bytes_cudaMalloc(0), max_tokens(config.max_tokens) {
        printf("CudaDecoder2 Constructor\n");

        int max_token = config.max_tokens; // for CUB

        // Comments about variables are in the .h file

        hipStreamCreate(&compute_st);
        hipStreamCreate(&copy_st);

        hipEventCreate(&loglikelihood_evt);
        hipEventCreate(&q_token_from_narcs_evt);
        hipEventCreate(&can_write_to_main_q);

        int max_token_frame = 5000000; // move back to params
        int max_token_all_frames = 1000000000; // move back to params + use a pinned memory vector

        // we could use same pointer
        hipMalloc(&d_main_q_state, max_token_frame * sizeof(int));
        hipHostMalloc(&h_main_q_state, max_token_frame * sizeof(int));
        hipMalloc(&d_aux_q_state, max_token_frame * sizeof(int));

        hipMalloc(&d_main_q_cost, max_token_frame * sizeof(CostType));
        hipHostMalloc(&h_main_q_cost, max_token_frame * sizeof(CostType));
        hipMalloc(&d_aux_q_cost, max_token_frame * sizeof(CostType));

        hipMalloc(&d_main_q_info, max_token_frame * sizeof(InfoToken));
        hipMalloc(&d_aux_q_info, max_token_frame * sizeof(InfoToken));

        int *bufi4;
        hipMalloc(&bufi4, 6*sizeof(int));

        d_main_q_local_offset = &bufi4[0];
        d_aux_q_end = &bufi4[2];

        hipMalloc(&d_main_q_end_and_narcs_i2, sizeof(QEndAndNarcs));

        d_main_q_narcs = &d_main_q_end_and_narcs_i2->split.narcs;
        d_main_q_end = &d_main_q_end_and_narcs_i2->split.end;

        hipMalloc(&d_cutoff, sizeof(BaseFloat));

        hipHostMalloc(&h_all_tokens_info, max_token_all_frames * sizeof(InfoToken));

        hipHostMalloc(&h_main_q_end, sizeof(int));  
        hipHostMalloc(&h_main_q_narcs, sizeof(int));  
        hipHostMalloc(&h_main_q_local_offset, sizeof(int));  
        hipHostMalloc(&h_aux_q_end, sizeof(int));  

        // we could use same pointer
        hipMalloc(&d_degrees_scan, max_token_frame * sizeof(int));
        hipMalloc(&d_degrees_block_scan, (max_token_frame / 256 + 2)* sizeof(int)); // TODO remove hardcoded
        hipMalloc(&d_main_q_arc_offsets, max_token_frame * sizeof(int));

        hipMalloc(&loglikelihoods_d, sizeof(BaseFloat)*(fst_.max_ilabel+1));  
        hipMalloc(&next_loglikelihoods_d, sizeof(BaseFloat)*(fst_.max_ilabel+1));  
        hipHostMalloc(&loglikelihoods_h, sizeof(BaseFloat)*(fst_.max_ilabel+1));  

        hipMalloc(&d_state_cost,sizeof(CostType)*fst_.numStates);

        hipMalloc(&d_n_CTA_done, sizeof(int));

        cudaCheckError();
    }

    CudaDecoder::~CudaDecoder() {
        printf("CUDA DECODER DESTRUCTOR\n");
        // TODO
    }

    void CudaDecoder::InitDecoding() {
        printf("CUDA DECODER InitDecoding\n");

        InitLookup();

        StateId start_state = fst_.Start();
        KALDI_ASSERT(start_state != fst::kNoStateId);

        cudaCheckError();
        InfoToken it_init;
        it_init.prev_token = INT_MIN;
        it_init.arc_idx = -1;

        CostType cost = StdWeight::One().Value();

        // We'll call ProcessNonemitting just after,
        // which will move tokens from aux to main
        hipMemcpy(d_aux_q_state, &start_state, sizeof(StateId), hipMemcpyHostToDevice);
        hipMemcpy(d_aux_q_cost, &cost, sizeof(CostType), hipMemcpyHostToDevice);
        hipMemcpy(d_aux_q_info, &it_init, sizeof(InfoToken), hipMemcpyHostToDevice);

        // We simulate a regular execution for the first iteration
        hipMemcpy(&d_state_cost[start_state], &cost, sizeof(CostType), hipMemcpyHostToDevice);

        // Init state is in queue
        int one = 1;
        hipMemcpy(d_aux_q_end, &one, sizeof(int), hipMemcpyHostToDevice);
        *h_aux_q_end = 1;

        hipMemset(d_main_q_end, 0, sizeof(int));
        *h_main_q_end = 0;

        hipMemset(d_main_q_local_offset, 0, sizeof(int));
        main_q_global_offset = 0;

        CostType cutoff = FLT_MAX;
        hipMemcpy(d_cutoff, &cutoff, sizeof(CostType), hipMemcpyHostToDevice);

        hipMemset(d_n_CTA_done, 0, sizeof(int));

        cudaCheckError();

        num_frames_decoded_ = 0;

        printf("CUDA DECODER InitDecoding 1/2\n");
        ProcessNonemitting();

        int main_q_size = *h_main_q_end;
        hipMemcpy(h_all_tokens_info, d_main_q_info, main_q_size*sizeof(InfoToken), hipMemcpyDeviceToHost);

        printf("CUDA DECODER InitDecoding 2/2\n");
    }


    // Used to trigger the fire&forget version of atomicMin (only av for int/long)
    __device__ int floatToOrderedInt(float floatVal) {

        int intVal = __float_as_int( floatVal );

        return (intVal >= 0 ) ? intVal : intVal ^ 0x7FFFFFFF;

    }



    __device__ float orderedIntToFloat(int intVal) {

        return __int_as_float( (intVal >= 0) ? intVal : intVal ^ 0x7FFFFFFF );

    } 


    // Used before first frame
    __global__ void init_lookup_kernel(int *state_cost, int size) {
        for(int idx = blockIdx.x*blockDim.x + threadIdx.x;
                idx < size;
                idx += blockDim.x*gridDim.x) {
            state_cost[idx]  = floatToOrderedInt(FLT_MAX);
        }
    }

    void CudaDecoder::InitLookup() {
        int nstates = fst_.numStates;


        dim3 grid,block;
        block.x = 256;
        grid.x = DIV_ROUND_UP(nstates, block.x);

        init_lookup_kernel<<<grid,block>>>(d_state_cost, nstates);
    }

    typedef CudaDecoder::StateId StateId;
    typedef CudaDecoder::QEndAndNarcs QEndAndNarcs;
    typedef CudaDecoder::CostType CostType;

    // Used to reset lookup table between frames
    // Using the queue to reset only the values needed
    // Also takes care of resetting cutof
    // TODO rename to something like "ResetForNewFrame"
    __global__ void reset_lookup_kernel(StateId *d_main_q_state, const int *d_main_q_end, int *state_cost, CostType *d_cutoff) {
        int q_from_end = *d_main_q_end; 

        for(int idx = blockIdx.x*blockDim.x + threadIdx.x;
                idx < q_from_end;
                idx += blockDim.x*gridDim.x) {

            StateId state = d_main_q_state[idx];
            state_cost[state]  = floatToOrderedInt(FLT_MAX);
        }

        // Avoiding a kernel call just to reset the cutoff
        if(blockIdx.x == 0 && threadIdx.x == 0)
            *d_cutoff = FLT_MAX; 
    }

    void CudaDecoder::ResetLookup() {
        int size = *h_main_q_end;

        dim3 grid,block;
        block.x = 256;
        grid.x = DIV_ROUND_UP(size, block.x);

        reset_lookup_kernel<<<grid,block,0,compute_st>>>(d_main_q_state, d_main_q_end, d_state_cost, d_cutoff);
    }


    void CudaDecoder::AdvanceDecoding(DecodableInterface *decodable,
            int32 max_num_frames) {
        printf("AdvanceDecoding\n");

        KALDI_ASSERT(num_frames_decoded_ >= 0 &&
                "You must call InitDecoding() before AdvanceDecoding()");
        int32 num_frames_ready = decodable->NumFramesReady();

        // num_frames_ready must be >= num_frames_decoded, or else
        // the number of frames ready must have decreased (which doesn't
        // make sense) or the decodable object changed between calls
        // (which isn't allowed).
        KALDI_ASSERT(num_frames_ready >= num_frames_decoded_);

        int32 target_frames_decoded = num_frames_ready;
        if (max_num_frames >= 0)
            target_frames_decoded = std::min(target_frames_decoded,
                    num_frames_decoded_ + max_num_frames);

        ComputeLogLikelihoods(decodable);

        int prev_main_q_size = *h_main_q_end;
        while (num_frames_decoded_ < target_frames_decoded) {
            
            // Computing a new frame

            hipEventSynchronize(loglikelihood_evt);
            std::swap(next_loglikelihoods_d, loglikelihoods_d);
            num_frames_decoded_++; 
            ComputeLogLikelihoods(decodable);

            // Emitting 
            // we will not write in the main q in that step
            // (preprocess is in place)
            // we don't need can_write_to_main_q
            ProcessEmitting();
            // After process emitting we won't need the token
            // associated with the previous frame
            // the main q has been flushed, we update its offset
            main_q_global_offset += prev_main_q_size;
            
            // Non Emitting
            // we will write to the main q 
            // (preprocess is "contract and preprocess")
            hipEventSynchronize(can_write_to_main_q);
            ProcessNonemitting(); 
            
            prev_main_q_size = *h_main_q_end;
            
            // We are done with the current frame
            // We copy back its pruned tokens to the host
            // We only copy the "info" part (arc_idx + prev_token)
            // because we don't need anything else for the final backtrack
            hipMemcpyAsync(&h_all_tokens_info[main_q_global_offset], 
                            d_main_q_info, 
                            prev_main_q_size*sizeof(InfoToken),
                            hipMemcpyDeviceToHost, 
                            copy_st);
            hipEventRecord(can_write_to_main_q, copy_st);

        }   


        printf("AdvanceDecoding Done\n");
        nvtxRangePop();
    }


    void CudaDecoder::ComputeLogLikelihoods(DecodableInterface *decodable) {
        nvtxRangePushA("ComputeLogLikelihoods");

        int32 frame = num_frames_decoded_;

        decodable->ComputeLogLikelihoods(loglikelihoods_h,frame,fst_.max_ilabel+1);

        //copying in another stream to overlap transfer with compute
        hipMemcpyAsync(next_loglikelihoods_d, loglikelihoods_h, sizeof(BaseFloat)*(fst_.max_ilabel+1), hipMemcpyHostToDevice,
                copy_st);

        hipEventRecord(loglikelihood_evt, copy_st);

        nvtxRangePop();
    }


    // Below that value, we launch the persistent kernel for NonEmitting
#define NONEM_LT_MAX_NARCS 4096
    bool CudaDecoder::ProcessToken(unsigned int *d_arc_offsets,
            bool is_emitting) {

        if(is_emitting) {
            PreprocessInPlace(d_arc_offsets);
            hipEventRecord(q_token_from_narcs_evt, compute_st);
            ResetLookup();
            FinalizePreprocessInPlace();
        } else {
            ContractAndPreprocess(d_arc_offsets);
            hipEventRecord(q_token_from_narcs_evt, compute_st);
        }


        // We need h_q_token_from_narcs to be ready
        hipEventSynchronize(q_token_from_narcs_evt);
        int main_q_narcs = *h_main_q_narcs;

        ExpandArcParams params;

        params.d_main_q_state = d_main_q_state;
        params.d_main_q_cost = d_main_q_cost;
        params.d_main_q_info = d_main_q_info;

        params.d_main_q_local_offset = d_main_q_local_offset;
        params.main_q_global_offset = main_q_global_offset;

        params.d_main_q_end = d_main_q_end;
        params.d_main_q_narcs = d_main_q_narcs;

        params.h_main_q_end = h_main_q_end;
        params.h_main_q_narcs = h_main_q_narcs;

        params.d_aux_q_state = d_aux_q_state; 
        params.d_aux_q_cost = d_aux_q_cost; 
        params.d_aux_q_info = d_aux_q_info;
        params.d_aux_q_end = d_aux_q_end;

        params.h_aux_q_end = h_aux_q_end;

        params.d_degrees_scan = d_degrees_scan; 
        params.d_q_arc_offsets = d_main_q_arc_offsets;
        params.arc_ilabels = fst_.arc_ilabels_d;
        params.is_emitting = is_emitting;

        params.arc_weights = fst_.arc_weights_d; 
        params.arc_nextstates = fst_.arc_nextstates_d; 
        params.d_cutoff = d_cutoff;
        params.beam = beam_;
        params.d_loglikelihoods= loglikelihoods_d;
        params.d_lookup = d_state_cost;

        params.d_n_CTA_done = d_n_CTA_done;
    
        bool done = false;

        if(main_q_narcs) {
            if(!params.is_emitting 
                    && main_q_narcs < NONEM_LT_MAX_NARCS) { 
                NonEmittingLongTail(d_arc_offsets, params); 

                cudaCheckError();

                // Persistent kernel finishes the job
                done = true;
            }
            else {
                ExpandArcs(main_q_narcs, params);
            }

            hipStreamSynchronize(compute_st); 
        }

        cudaCheckError();
        return done;
    }


    void CudaDecoder::ProcessEmitting() {
        nvtxRangePushA("ProcessEmitting");

        // Using emitting arc offsets
        ProcessToken(fst_.e_offsets_d, true); 

        cudaCheckError();
        nvtxRangePop();
    }

    void CudaDecoder::ProcessNonemitting() {
        nvtxRangePushA("ProcessNonemitting");

        // While not done, call it
        // If remaining n_arcs < 4k, 
        // ProcessToken will call a persistent kernel
        while(!ProcessToken(fst_.ne_offsets_d, false));

        cudaCheckError();
        nvtxRangePop();
    }

    // TODO rename
    struct F2Sum {
        __device__ int2 operator()(const int2 &a, const int2 &b) const {
            int2 c;
            c.x = a.x + b.x;
            c.y = a.y + b.y;

            return c;
        }
    };

    typedef CudaDecoder::PreprocessParams PreprocessParams; // TODO move
    /*
       This kernel preprocess the necessary information for expand (scan of the outgoing degrees) 
       and explicitly prune the tokens

       It contracts (by pruning) the queue list:
       raw output from aux_q ----contract----> pruned output in main q

       This kernel is responsible for :

       1) Read a token from the aux queue (raw output from previous expand)

       2) Compute the outgoing degree of that token.next_state. For that :
       -> If that token is suboptimal (cutoff, best_cost), we prune it
       -> Otherwise, we set degree using CSR graph

       3) We move the non-pruned tokens into the main q. After a local prefix sum,
       we request a spot using the main_q_end_and_narcs counter. 
       main_q_end_and_narcs.split.end contains the number of tokens in the main q until now
       main_q_end_and_narcs.split.narcs contains the number of arcs in the main q until now

       We also do the degrees scan in one pass using the maind_q_end_and_narcs.split.narcs

       This kernel is used before ProcessNonEmitting
    */

#define COMPUTE_DEGREES_DIMX 256
    __global__ void contract_and_preprocess_kernel(PreprocessParams params) {


        typedef hipcub::BlockScan<int2, COMPUTE_DEGREES_DIMX> BlockScan;
        __shared__ typename BlockScan::TempStorage temp_storage;

        __shared__ QEndAndNarcs blk_local_offset_i2;

        const int aux_q_end = *params.d_aux_q_end;
        BaseFloat cutoff = *params.d_cutoff;

        for(int block_offset = blockDim.x*blockIdx.x;
                block_offset < aux_q_end;
                block_offset += gridDim.x*blockDim.x) {

            int aux_q_idx = block_offset + threadIdx.x;
            int degree = 0;
            int arc_start = -1;

            StateId state_idx;
            CostType cost;

            if(aux_q_idx < aux_q_end) {
                cost = params.d_aux_q_cost[aux_q_idx];
                state_idx = params.d_aux_q_state[aux_q_idx];

                if(cost < cutoff) {
                    BaseFloat best_cost = orderedIntToFloat(params.d_state_cost[state_idx]);

                    if(cost == best_cost) {
                        arc_start = params.d_arc_offsets[state_idx];
                        int arc_end = params.d_arc_offsets[state_idx+1];
                        degree = arc_end - arc_start;
                    }
                } 
            }

            int is_pruned = (arc_start == -1);
            int2 scan_i2;
            scan_i2.x =  is_pruned ? 0 : 1;
            scan_i2.y =  degree;

            int2 zero_i2;
            zero_i2.x = zero_i2.y = 0;

            BlockScan(temp_storage).ExclusiveScan(scan_i2, scan_i2, zero_i2, F2Sum());

            if(threadIdx.x == (COMPUTE_DEGREES_DIMX-1)) {
                // CUB Scan is exclusive
                QEndAndNarcs inclusive_scan;
                inclusive_scan.split.end = scan_i2.x + (is_pruned ? 0 : 1);
                inclusive_scan.split.narcs = scan_i2.y + degree;

                blk_local_offset_i2.both = atomicAdd(&params.d_main_q_end_and_narcs_i2->both, inclusive_scan.both);
            }

            __syncthreads(); // blk_local_offset + temp_storage

            if(!is_pruned) {
                // Moving non-pruned to the main q
                int main_q_idx = blk_local_offset_i2.split.end + scan_i2.x;

                InfoToken info = params.d_aux_q_info[aux_q_idx];

                params.d_main_q_state[main_q_idx] = state_idx;
                params.d_main_q_cost[main_q_idx] = cost;
                params.d_main_q_info[main_q_idx] = info;

                params.d_degrees_scan[main_q_idx] = blk_local_offset_i2.split.narcs + scan_i2.y;

                params.d_main_q_arc_offsets[main_q_idx] = arc_start;
            }


        }

        if(threadIdx.x == 0) {
            int old = atomicAdd(params.d_n_CTA_done, 1);
            bool is_last_CTA = (old == (gridDim.x -1));

            if(is_last_CTA) {
                __threadfence();

                // Avoid a mem copy
                *params.h_main_q_narcs = *params.d_main_q_narcs; // pinned memory
                *params.d_n_CTA_done = 0;
                *params.d_aux_q_end = 0; // we flushed the aux q

            }
        }

    }


/*
    This kernel is also a preprocessing kernel, but this time does it in place
    The tokens are already in the main q (they were placed here by a previous "contract and preprocess"). We implicitly
    prune the non-optimal ones (by setting the degree to 0), and we compute the degrees scan.

    Here we have to do the scan in two passes : the scan will be finished in "finalize_preprocess"

    This preprocess step is used in ProcessEmitting. Tokens were placed in main_q by
    the ProcessNonEmitting of the previous frame. We cannot renumber them (it would break
    the prev_token index). We preprocess in place, leaving things as they are in main_q

*/

#define COMPUTE_DEGREES_DIMX 256
    __global__ void preprocess_in_place_kernel(PreprocessParams params) {
    
        typedef hipcub::BlockScan<int, COMPUTE_DEGREES_DIMX> BlockScan;
        __shared__ typename BlockScan::TempStorage temp_storage;

        __shared__ int blk_scan_offset;
        __shared__ int is_last_CTA;


        int queue_offset = *params.d_main_q_local_offset;
        int queue_end = *params.d_main_q_end;
        int queue_size = queue_end - queue_offset;

        BaseFloat cutoff = *params.d_cutoff;

        for(int block_offset = blockDim.x*blockIdx.x;
                block_offset < queue_size;
                block_offset += gridDim.x*blockDim.x)
        {
            int idx = queue_offset + block_offset + threadIdx.x; 
            int degree = 0; 
            if(idx < queue_end) {
                StateId state_idx = params.d_main_q_state[idx]; 
                BaseFloat cost = params.d_main_q_cost[idx];

                if(cost < cutoff) {
                    BaseFloat best_cost = orderedIntToFloat(params.d_state_cost[state_idx]); 
                    if(cost == best_cost) {
                        int start = params.d_arc_offsets[state_idx]; 
                        int end = params.d_arc_offsets[state_idx+1]; 
                        degree  = end - start;
                        params.d_main_q_arc_offsets[idx] = start;
                    }
                }
            }

            int scan;
            BlockScan(temp_storage).ExclusiveSum(degree, scan);
            if(idx < queue_end) 
                params.d_degrees_scan[idx] = scan;


            if(threadIdx.x == (COMPUTE_DEGREES_DIMX-1))
                params.d_degrees_block_scan[block_offset/COMPUTE_DEGREES_DIMX] = (scan + degree); 

            if((block_offset + gridDim.x*blockDim.x) < queue_end)
                __syncthreads(); // we'll reuse temp_storage
        }

        if(threadIdx.x == 0) {
            int old = atomicAdd(params.d_n_CTA_done, 1); 
            blk_scan_offset = 0;
            is_last_CTA = (old == (gridDim.x -1));
        }

        // is_last_CTA + temp_storage reuse
        __syncthreads();
        
        if(is_last_CTA)
        {
            // The last block alive takes care of scan of block sums 
            __threadfence();

            if(threadIdx.x == 0) {
                *params.d_n_CTA_done = 0;
            }

            // following value can be different than gridDim.x 
            int total_blk_val = (queue_size + COMPUTE_DEGREES_DIMX -1) / COMPUTE_DEGREES_DIMX;

            for(int blk_idx_off = 0; blk_idx_off < total_blk_val; blk_idx_off += blockDim.x) {
                int blk_idx = blk_idx_off + threadIdx.x; 

                int blk_sum = (blk_idx < total_blk_val) ?  params.d_degrees_block_scan[blk_idx] : 0; 
                int blk_scan;
                BlockScan(temp_storage).ExclusiveSum(blk_sum, blk_scan);
                blk_scan += blk_scan_offset; 

                if(blk_idx < total_blk_val) {
                    params.d_degrees_block_scan[blk_idx] = blk_scan;
                }

                if(threadIdx.x == (COMPUTE_DEGREES_DIMX-1)) {
                    int total = blk_scan + blk_sum; 
                    blk_scan_offset = total;
                }

                __syncthreads();
                // blk_scan_offset + reuse temp_storage
            }

            if(threadIdx.x == 0)
            {
                *params.d_main_q_narcs = blk_scan_offset; 
                *params.h_main_q_narcs = blk_scan_offset; // pinned memory
            }
        }
    }



// TODO merge the two struct 
    void CudaDecoder::ContractAndPreprocess(unsigned int *d_arc_offsets) {
        dim3 grid,block;
        block.x = COMPUTE_DEGREES_DIMX;
        grid.x = DIV_ROUND_UP(*h_aux_q_end, block.x);

        PreprocessParams params;

        params.d_aux_q_state = d_aux_q_state; 
        params.d_aux_q_cost = d_aux_q_cost;
        params.d_aux_q_info = d_aux_q_info; 
        params.d_aux_q_end = d_aux_q_end;

        params.d_main_q_state = d_main_q_state; 
        params.d_main_q_cost = d_main_q_cost;
        params.d_main_q_info = d_main_q_info; 
        params.d_main_q_end_and_narcs_i2 = d_main_q_end_and_narcs_i2; 
        params.d_main_q_narcs = d_main_q_narcs;
        params.d_main_q_end = d_main_q_end;

        params.d_main_q_local_offset = d_main_q_local_offset;

        params.d_main_q_end = d_main_q_end;
        params.h_main_q_narcs = h_main_q_narcs;

        params.d_degrees_scan = d_degrees_scan; 
        params.d_arc_offsets = d_arc_offsets;
        params.d_main_q_arc_offsets = d_main_q_arc_offsets; // offsets, relative to the queue

        params.d_state_cost = d_state_cost; 
        params.d_cutoff = d_cutoff; 

        params.d_degrees_block_scan = d_degrees_block_scan; 

        params.d_n_CTA_done = d_n_CTA_done;

        contract_and_preprocess_kernel<<<grid,block,0,compute_st>>>(params);
    }


    void CudaDecoder::PreprocessInPlace(unsigned int *d_arc_offsets) {
        dim3 grid,block;
        block.x = COMPUTE_DEGREES_DIMX;
        int main_q_size = *h_main_q_end - *h_main_q_local_offset;

        grid.x = DIV_ROUND_UP(main_q_size, block.x);

        PreprocessParams params;

        params.d_aux_q_state = d_aux_q_state; 
        params.d_aux_q_cost = d_aux_q_cost;
        params.d_aux_q_info = d_aux_q_info; 
        params.d_aux_q_end = d_aux_q_end;

        params.d_main_q_state = d_main_q_state; 
        params.d_main_q_cost = d_main_q_cost;
        params.d_main_q_info = d_main_q_info; 
        params.d_main_q_end_and_narcs_i2 = d_main_q_end_and_narcs_i2; 
        params.d_main_q_end = d_main_q_end; 
        params.d_main_q_narcs = d_main_q_narcs; 


        params.d_main_q_local_offset = d_main_q_local_offset;

        params.d_degrees_scan = d_degrees_scan; 
        params.d_arc_offsets = d_arc_offsets;
        params.d_main_q_arc_offsets = d_main_q_arc_offsets; // offsets, relative to the queue

        params.d_state_cost = d_state_cost; 
        params.d_cutoff = d_cutoff; 

        params.d_degrees_block_scan = d_degrees_block_scan; 

        params.h_main_q_narcs = h_main_q_narcs; 
        params.d_n_CTA_done = d_n_CTA_done;


        preprocess_in_place_kernel<<<grid,block,0,compute_st>>>(params);
    }

    /*

       Part 2 of the scan for "PreprocessEmitting". For NonEmitting scan is already final

       Computes global prefix sum with block prefix sum and block offsets

       If we want to speed up expand, we can compute lower and upper bound to restrain 
       the binary search in expand
       This can be done on the fly here, and removes main bottleneck of expand
       Not done for now, because expand is fast enough

     */
    __global__ void finalize_degrees_scan_kernel(int *d_scan, int *d_blk_scan, const int *d_main_q_local_offset, const int
            *d_main_q_end) {

        int q_off = *d_main_q_local_offset;
        int q_end = *d_main_q_end;
        int q_size = q_end - q_off;

        for(int idx = q_off + blockDim.x*blockIdx.x + threadIdx.x;
                idx < q_size;
                idx += blockDim.x*gridDim.x) {

            int blk_idx = (idx - q_off) / COMPUTE_DEGREES_DIMX;
            int blk_scan_offset = d_blk_scan[blk_idx]; // we rely on L1 for this one, avoiding syncs

            d_scan[idx] += blk_scan_offset;
        }

    }

    void CudaDecoder::FinalizePreprocessInPlace() {
        dim3 grid,block;
        block.x = COMPUTE_DEGREES_DIMX;
        int main_q_size = *h_main_q_end - *h_main_q_local_offset;
        grid.x = DIV_ROUND_UP(main_q_size, block.x);

        finalize_degrees_scan_kernel<<<grid,block,0,compute_st>>>(d_degrees_scan, d_degrees_block_scan, d_main_q_local_offset,
                d_main_q_end); 
    }


    __forceinline__ __device__ int binsearch_maxle(const int *vec, const int val, int low, int high) {
        while(true) {
            if(low == high)
                return low; //we know it exists
            if((low + 1) == high)
                return (vec[high] <= val) ? high : low;

            int mid = low + (high- low) / 2;

            if(vec[mid] > val)
                high = mid-1;
            else
                low = mid;
        }
    }


    // Temporary used for cutoff - will be removed
    __device__ float fatomicMin(float *addr, float value)

    {

        float old = *addr, assumed;
        if(old <= value) return old;

        do
        {
            assumed = old;
            old = atomicCAS((unsigned int*)addr,
                    __float_as_int(assumed),
                    __float_as_int(value));

        } while(old!=assumed);

        return old;

    }

    typedef CudaDecoder::ExpandArcParams ExpandArcParams; // TODO move

#define EXPAND_ARCS_DIMX 256

    /*

       This kernel propagates arcs from the main q [main_q_local_offset, main_q_end[
       to the aux

       The main bottleneck is the first binary search. 
       If we want to remove it, preprocess it on the fly in preprocess

     */

    struct CostTInt {
        CostType cost;
        int i;
    };

    struct CISum {
        __device__ CostTInt operator()(const CostTInt &a, const CostTInt &b) const {
            CostTInt c;
            c.cost = fmin(a.cost, b.cost);
            c.i = a.i + b.i;

            return c;
        }
    };


    void __global__ expand_arcs_kernel(ExpandArcParams params) {
        typedef hipcub::BlockScan<CostTInt, EXPAND_ARCS_DIMX> BlockScan;

        __shared__ typename BlockScan::TempStorage temp_storage_scan;

        __shared__ int to_q_block_offset;
        __shared__ CostType blk_cutoff;

        const int total_narcs = *params.d_main_q_narcs;
        const int main_q_offset = *params.d_main_q_local_offset;
        const int main_q_end = *params.d_main_q_end;

        
        if(threadIdx.x == 0) {
            blk_cutoff = *params.d_cutoff;
        }

        __syncthreads();

        // Keeping the whole CTA alive, we'll have syncs
        for(int block_offset = blockDim.x*blockIdx.x;
                block_offset < total_narcs;
                block_offset += gridDim.x*blockDim.x) {

            int th_idx = block_offset + threadIdx.x;
            bool valid_input = (th_idx < total_narcs);

            BaseFloat total_cost = FLT_MAX;
            int arc_idx;
            StateId arc_next_state;
            int main_q_idx;

            if(valid_input) {
                //we can do better than that
                main_q_idx = binsearch_maxle(params.d_degrees_scan, th_idx, main_q_offset, main_q_end-1); 

                int lower_bound = params.d_degrees_scan[main_q_idx];
                int arc_offset_start = params.d_q_arc_offsets[main_q_idx];

                arc_idx = arc_offset_start + (block_offset + threadIdx.x - lower_bound);
                arc_next_state = params.arc_nextstates[arc_idx];

                total_cost = params.arc_weights[arc_idx];

                int arc_ilabel = params.is_emitting ? params.arc_ilabels[arc_idx] : 0;
                total_cost += (arc_ilabel != 0) ? -params.d_loglikelihoods[arc_ilabel] : 0.0; 
                total_cost += params.d_main_q_cost[main_q_idx];

                if(total_cost >= blk_cutoff)
                    valid_input = false;
                else {
                    // switch back to red, worst case is bad
                    BaseFloat next_state_cost = orderedIntToFloat(params.d_lookup[arc_next_state]);

                    if(total_cost >= next_state_cost)
                        valid_input = false;
                }
            }

                            int has_successor = valid_input ? 1 : 0;  // Need a spot in the new q
                            CostTInt ci;
                            ci.cost = valid_input ? (total_cost + params.beam) : FLT_MAX; // new cutoff candidate
                            ci.i = has_successor;

                            BlockScan(temp_storage_scan).InclusiveScan(ci, ci, CISum());

                            if(threadIdx.x == (EXPAND_ARCS_DIMX - 1)) {
                                int total_successors_in_block = ci.i;
                                to_q_block_offset = atomicAdd(params.d_aux_q_end, total_successors_in_block);

                                if(ci.cost < blk_cutoff) {
                                    CostType new_cutoff = fatomicMin(params.d_cutoff, ci.cost);
                                    blk_cutoff = fmin(ci.cost, new_cutoff);
                                }
                            }

                            __syncthreads(); // to_q_block_offset

                            ci.i -= has_successor; // we want the exclusive sum now
                            int to_q_index = to_q_block_offset + ci.i;


                            if(has_successor) {
                                params.d_aux_q_cost[to_q_index] = total_cost;
                                
                                atomicMin(&params.d_lookup[arc_next_state],
                                floatToOrderedInt(total_cost)
                                );

                                //printf("cost = %f, cutoff = %f, beam=%f \n", total_cost, blk_cutoff, params.beam);
                                if(total_cost < blk_cutoff) { // cutoff may have changed
                                    // We write the rest of the token only if necessary
                                    // if the cost is higher than cutoff, 
                                    // the token will be ignored anyway 

                                    params.d_aux_q_state[to_q_index] = arc_next_state;

                                    InfoToken new_tok_info;
                                    new_tok_info.prev_token = params.main_q_global_offset + main_q_idx;
                                    new_tok_info.arc_idx = arc_idx;
                            

                                    params.d_aux_q_info[to_q_index] = new_tok_info;

                                    /*
                                    printf("expand, adding %i (%i)  -> %i \n", new_tok_info.prev_token,
                                    params.main_q_global_offset, arc_next_state);
                                    */
                                }
                            }
        }


        // Last block alive sets h_aux_q_end (pinned memory)
        if(threadIdx.x == 0) {
            int old = atomicAdd(params.d_n_CTA_done, 1);
            if(old == (gridDim.x -1)) {
                __threadfence(); // we want last value of d_aux_q_end
                *params.h_aux_q_end = *params.d_aux_q_end;
                *params.d_n_CTA_done = 0;
                *params.d_main_q_narcs = 0;
                *params.h_main_q_narcs = 0;

                if(params.is_emitting) {
                    *params.d_main_q_local_offset = 0;
                    *params.d_main_q_end = 0;
                    *params.h_main_q_end = 0;
                } else {
                    *params.d_main_q_local_offset = main_q_end;
                }

            }
        }

    }

    void CudaDecoder::ExpandArcs(int nthreads, const ExpandArcParams &params) {
        dim3 grid,block;
        block.x = 256;
        grid.x = DIV_ROUND_UP(nthreads, block.x);

        expand_arcs_kernel<<<grid,block,0,compute_st>>>(params);
    }

    // Wrote for single CTA

    /*

       Persistent kernel

       Used to avoid calling multiple "heavy lifting" kernels for the tail of non emitting
       (lots of iterations with small number of arcs)

       Code is greatly simplified because we can have only one CTA alive

       Repeat until new queue empty:
       1) Computes degrees (cf ComputeDegrees) 
       2) Compute scan
       3) Expand arcs

       1 and 2 are not done on the first iteration, because it's already done
       (by corresponding kernels)

       At the end, this kernel finalize the computation for current frame,
       so that it's ready for next ProcessEmitting

       We could optimize and speed up this kernel
       It will only gives us a better latency for 1 stream, which is low enough
       Instead, we let it compute while we use the GPU for other streams
       This kernel only uses one block

     */


#define NONEM_LT_DIMX 1024
    __launch_bounds__(NONEM_LT_DIMX, 1)
        __global__ void process_nonem_longtail(unsigned int *d_arc_offsets, 
                ExpandArcParams params) {

            typedef hipcub::BlockScan<int, NONEM_LT_DIMX> BlockScan;
            typedef hipcub::BlockReduce<float, NONEM_LT_DIMX> BlockReduce;

            __shared__ typename BlockScan::TempStorage temp_storage_scan;
            __shared__ typename BlockReduce::TempStorage temp_storage_reduce;

            __shared__ BaseFloat cutoff;


            int old_q_offset = *params.d_main_q_local_offset;
            int new_q_offset = *params.d_main_q_end;
            int new_q_end = new_q_offset;

            int total_narcs = *params.d_main_q_narcs;
    
            int old_q_size = new_q_offset - old_q_offset;  // move to end

            cutoff = *params.d_cutoff;

            // We'll switch queue at the beg of the loop
            // Cleaner that way - we need the offsets ready for
            // the global updates at the very end of this kernel
            new_q_offset = old_q_offset;

            bool first = true;

            while(old_q_size > 0) {
                // Step 0 : move queues        
                old_q_offset = new_q_offset;
                new_q_offset = new_q_end;

                if(!first) {
                    __syncthreads(); // old_q_ready
                    total_narcs = 0;

                    // Step 1 : compute_degrees
                    // TODO fuse 1 and 2
                    for(int q_idx = old_q_offset + threadIdx.x;
                            q_idx < new_q_offset; // = old_q_end
                            q_idx += blockDim.x) {

                        StateId state = params.d_main_q_state[q_idx];
                        BaseFloat cost = params.d_main_q_cost[q_idx];

                        int degree = 0;
                        if(cost < cutoff) {
                            BaseFloat best_cost = orderedIntToFloat(params.d_lookup[state]);

                            if(cost == best_cost) {
                                int start = d_arc_offsets[state];
                                int end = d_arc_offsets[state+1];
                                degree = end - start;
                                params.d_q_arc_offsets[q_idx] = start;
                            }
                        }

                        params.d_degrees_scan[q_idx] = degree;
                    }

                    __syncthreads(); // will be removed

                    // Step 2 : Scan

                    for(int block_off = 0;
                            block_off < old_q_size;
                            block_off += blockDim.x) {

                        int q_idx = old_q_offset + block_off + threadIdx.x;

                        int degree = (q_idx < new_q_offset) 
                            ? params.d_degrees_scan[q_idx]
                            : 0;
                        int lscan;
                        int total_in_blk;
                        BlockScan(temp_storage_scan).ExclusiveSum(degree, lscan, total_in_blk);
                        int scan = lscan + total_narcs;
                        total_narcs += total_in_blk;

                        if(q_idx < new_q_offset)
                            params.d_degrees_scan[q_idx] = scan;

                         __syncthreads(); // reusing temp_storage_scan + degrees ready
                    }


                } else {
                    first = false;    
                }


                // We already sync'ed

                // Step 3 : expand arcs

                for(int block_offset = 0;
                        block_offset < total_narcs;
                        block_offset += blockDim.x) {

                    int th_idx = block_offset + threadIdx.x;
                    bool valid_input = (th_idx < total_narcs);

                    BaseFloat total_cost = FLT_MAX;
                    int arc_idx;
                    StateId arc_next_state;
                    int q_idx;

                    if(valid_input) {
                        //we can do better than that
                        q_idx = binsearch_maxle(params.d_degrees_scan, th_idx, old_q_offset, new_q_offset-1); 

                        int lower_bound = params.d_degrees_scan[q_idx];
                        int arc_offset_start = params.d_q_arc_offsets[q_idx];

                        arc_idx = arc_offset_start + (th_idx - lower_bound);

                        arc_next_state = params.arc_nextstates[arc_idx];
                        BaseFloat arc_weight = params.arc_weights[arc_idx];
                        BaseFloat next_state_cost = orderedIntToFloat(params.d_lookup[arc_next_state]);
                        BaseFloat old_tok_cost = params.d_main_q_cost[q_idx];

                        total_cost = arc_weight + old_tok_cost;

                        if(total_cost >= next_state_cost) {
                            total_cost = FLT_MAX;
                            valid_input = false; 
                        } 
                    }

                    BaseFloat thread_cutoff = (total_cost < FLT_MAX) ? (total_cost + params.beam) : FLT_MAX;
                    BaseFloat new_block_cutoff = BlockReduce(temp_storage_reduce).Reduce(thread_cutoff, hipcub::Min());

                    if(threadIdx.x == 0) {
                        if(new_block_cutoff < cutoff) {
                            cutoff = new_block_cutoff;
                        }
                    }

                    __syncthreads();

                    int has_successor = (total_cost < cutoff && valid_input) ? 1 : 0;

                    if(has_successor) 
                        atomicMin(&params.d_lookup[arc_next_state], floatToOrderedInt(total_cost));


                    int new_q_idx_block = has_successor;
                    int total_in_blk;
                    BlockScan(temp_storage_scan).ExclusiveSum(new_q_idx_block, new_q_idx_block, total_in_blk);

                    if(has_successor) {
                        int new_q_index = new_q_end + new_q_idx_block;
                        params.d_main_q_state[new_q_index] = arc_next_state;

                        params.d_main_q_cost[new_q_index] = total_cost;

                        InfoToken new_tok_info;
                        new_tok_info.prev_token = params.main_q_global_offset + q_idx;

                        new_tok_info.arc_idx = arc_idx;
                        params.d_main_q_info[new_q_index] = new_tok_info;
                        
                        //printf("new q index = %i (%i+%i) (tot=%i) \n", new_q_index, new_q_end, new_q_idx_block,
                        //total_in_blk);
                   }

                    new_q_end += total_in_blk;
                }

                old_q_size = new_q_end - new_q_offset; 
            }

            if(threadIdx.x == 0) {
                // Next step is ProcessEmitting of next frame, from is currToken_offset
                *params.d_main_q_end = new_q_end; 
                *params.h_main_q_end = new_q_end; 

                *params.d_main_q_local_offset = 0; 

                *params.d_cutoff = cutoff;
            }

        }

    void CudaDecoder::NonEmittingLongTail(unsigned int *d_arc_offsets, 
            const ExpandArcParams &params) {

        dim3 grid,block;
        block.x = NONEM_LT_DIMX;
        grid.x = 1; // it is designed for the long tail
        process_nonem_longtail<<<grid,block,0,compute_st>>>(d_arc_offsets, params);
    }


    /*
       GetBestCost, GetBestPath, IsFinal
       CPU only, called only at the end

     */


    void CudaDecoder::GetBestCost(BaseFloat *min, int *arg, bool isfinal) const {
        CostType best_cost = FLT_MAX; // switch to numeric limits std11
        int best_cost_idx;
        // we need main q end ready
        int main_q_size = *h_main_q_end;

        hipMemcpy(h_main_q_cost, d_main_q_cost, main_q_size * sizeof(CostType), hipMemcpyDeviceToHost);

        if(isfinal)
            hipMemcpy(h_main_q_state, d_main_q_state, main_q_size * sizeof(int), hipMemcpyDeviceToHost);

        // TODO add event main q ready once memcpy becomes async

        for(int i=0; i < main_q_size; ++i) {
            CostType cost = h_main_q_cost[i];

            if(isfinal) 
                cost += fst_.final_h[h_main_q_state[i]];

            if(cost < best_cost) {
                best_cost = cost;
                best_cost_idx = i;
            }
        }

        printf("global_offset=%i \n", main_q_global_offset);
        best_cost_idx += main_q_global_offset; 

        *min = best_cost;
        *arg = best_cost_idx;
    }


    bool CudaDecoder::ReachedFinal() const {
        int main_q_size = *h_main_q_end;
        hipMemcpy(h_main_q_state, d_main_q_state, main_q_size * sizeof(int), hipMemcpyDeviceToHost);


        for(int i=0; i < main_q_size; ++i) {
            if(fst_.final_h[h_main_q_state[i]] != StdWeight::Zero().Value())
                return true;
        }

        return false;
    }
    // Outputs an FST corresponding to the single best path
    // through the lattice.
    bool CudaDecoder::GetBestPath(Lattice *fst_out, bool use_final_probs) const {
        printf("Get best path \n");
        nvtxRangePushA("GetBestPath");

        bool isfinal = ReachedFinal();
        BaseFloat best_cost;
        int arg_best;
        GetBestCost(&best_cost, &arg_best, isfinal);


        printf("is final = %i \n", isfinal);
        printf("best cost : %f  with arg = %i \n", best_cost, arg_best);

        int token_idx = arg_best;
        std::vector<int> reversed_path;

        while(token_idx != INT_MIN) {
            int arc_idx = h_all_tokens_info[token_idx].arc_idx;
            reversed_path.push_back(arc_idx);
            token_idx = h_all_tokens_info[token_idx].prev_token;
        }


        fst_out->DeleteStates();

        // We can assert first state equals to root

        StateId cur_state = fst_out->AddState();
        fst_out->SetStart(cur_state);

        reversed_path.pop_back(); // dummy first arc

        for (int i = reversed_path.size()-1; i >= 1; i--) {
            int arc_idx = reversed_path[i];
            LatticeArc arc(fst_.arc_ilabels_h[arc_idx], fst_.arc_olabels_h[arc_idx], LatticeWeight(fst_.arc_weights_h[arc_idx], 0), fst_.arc_nextstates_h[arc_idx]);

            arc.nextstate = fst_out->AddState();
            fst_out->AddArc(cur_state, arc);
            cur_state = arc.nextstate;
        }

        if (isfinal && use_final_probs)
            fst_out->SetFinal(cur_state,
                    LatticeWeight(fst_.Final(fst_.arc_nextstates_h[reversed_path[0]]), 0.0));
        else
            fst_out->SetFinal(cur_state, LatticeWeight::One());

        fst::RemoveEpsLocal(fst_out);

        nvtxRangePop();
        return true;
    }





} // end namespace kaldi.
