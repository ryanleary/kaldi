#include "hip/hip_runtime.h"
// decoder/cuda-decoder.cu

// 2018 - Hugo Braun, Justin Luitjens, Ryan Leary

// See ../../COPYING for clarification regarding multiple authors
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//  http://www.apache.org/licenses/LICENSE-2.0
//
// THIS CODE IS PROVIDED *AS IS* BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, EITHER EXPRESS OR IMPLIED, INCLUDING WITHOUT LIMITATION ANY IMPLIED
// WARRANTIES OR CONDITIONS OF TITLE, FITNESS FOR A PARTICULAR PURPOSE,
// MERCHANTABLITY OR NON-INFRINGEMENT.
// See the Apache 2 License for the specific language governing permissions and
// limitations under the License.

#include "decoder/cuda-decoder.h"
#include <algorithm>
#include <nvToolsExt.h>
#include <hip/hip_runtime_api.h>
#include <float.h>
#include <algorithm>
#include <hipcub/hipcub.hpp>


#define MEMADVISE

#define KALDI_CUDA_DECODER_DIV_ROUND_UP(a,b) ((a+b-1)/b)

namespace kaldi {

    CudaDecoder::CudaDecoder(const CudaFst &fst, 
                             const CudaDecoderConfig &config,
                             int32 nlanes,
                             int32 nchannels): fst_(fst), 
                     default_beam_(config.default_beam),
                     max_tokens_(config.max_tokens), 
                     max_tokens_per_frame_(config.max_tokens_per_frame),
                     nlanes_(nlanes),
                     nchannels_(nchannels) {
        //
        // For a description of the class members, please refer to the cuda-decoder.h file
        //

        hipStreamCreate(&compute_st_);
        hipStreamCreate(&copy_st_); 

        hipEventCreate(&can_read_h_main_q_narcs_);
        hipEventCreate(&can_write_to_main_q_);
        hipEventCreate(&can_read_final_h_main_q_end_);
        hipEventCreate(&before_finalize_nonemitting_kernel_);

        KALDI_ASSERT(nlanes > 0);
        KALDI_ASSERT(nchannels > 0);

        ++n_channels_; // allocating init_channel_params at the same time

        hipHostMalloc(&h_lane_params, nlanes * sizeof(*h_lane_params));
        hipHostMalloc(&h_channel_params, nchannels * sizeof(*h_channels_params));
        hipMalloc(&d_lane_params, nlanes * sizeof(*d_lane_params));
        hipMalloc(&d_channel_params, nchannels * sizeof(*d_channels_params));

        // Allocating memory for all lanes
        // using intermediate size_t value because we're going reuse those sizes below,
        // but also to avoid overflowing int32 with byte counts in the future
        size_t one_aux_q_state_size = max_tokens_per_frame_ * sizeof(*d_all_aux_q_state_);
        size_t one_aux_q_cost_size =  max_tokens_per_frame_ * sizeof(*d_all_aux_q_cost_);
        size_t one_aux_q_info_size = max_tokens_per_frame_ * sizeof(*d_all_aux_q_info_);
        size_t one_main_q_info_size = max_tokens_per_frame_ * sizeof(*d_all_main_q_info_);
        size_t one_state_best_cost_size = fst_.num_states_*sizeof(*d_state_best_cost_);
        size_t one_main_q_degrees_block_sums_prefix_sum_size = (KALDI_CUDA_DECODER_DIV_ROUND_UP(max_tokens_per_frame_, KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX) + 1)
                                                                * sizeof(*d_main_q_degrees_block_sums_prefix_sum_);
        hipMalloc(&d_all_aux_q_state_, nlanes * one_aux_q_state_size);
        hipMalloc(&d_all_aux_q_cost_, nlanes * one_aux_q_cost_size);
        hipMalloc(&d_all_aux_q_info_, nlanes * one_aux_q_info_size);
        hipMalloc(&d_all_main_q_info_, nlanes * one_main_q_info_size);
        hipMalloc(&d_all_state_best_cost_, nlanes * one_state_best_cost_size);
        hipMalloc(&d_all_main_q_degrees_block_sums_prefix_sum_, nlanes * one_main_q_degrees_block_sums_prefix_sum_size_);


        // Allocating memory for all channels
        size_t one_main_q_state_size = max_tokens_per_frame_ * sizeof(*d_all_main_q_state_);
        size_t one_main_q_cost_size = max_tokens_per_frame_ * sizeof(*d_all_main_q_cost_);
        size_t one_main_q_arc_offsets_size = (max_tokens_per_frame_+1) * sizeof(*d_all_main_q_arc_offsets_);
        size_t one_loglikelihoods_size = (fst_.max_ilabel_+1)*sizeof(*d_loglikelihoods_);

        hipMalloc(&d_all_main_q_state_, nchannels * one_main_q_state_size);
        hipMalloc(&d_all_main_q_cost_, nchannels * one_main_q_cost_size);
        hipMalloc(&d_all_main_q_arc_offsets_, nchannels * one_main_q_arc_offsets_size);
        hipMalloc(&d_all_loglikelihoods_, nchannels * one_loglikelihoods_size);  
       
        // Setting lanes params
        for(int ilane=0; ilane<n_lanes_; ++ilane) {
            LaneParams params;
            params.main_q_end_and_narcs.split.ntokens = 0;
            params.main_q_end_and_narcs.split.narcs = 0;
            params.n_CTA_done_ = 0;
            params.aux_q_end_ = 0;
            params.q_overflow = 0;
            params.main_q_global_offset = 0;
            params.main_q_local_offset = 0;
            h_lane_params[ilane] = params;
        }

        // Setting channels params
        for(int ichannel=0; ichannel<n_channels_; ++ichannel) {
            ChannelParams params;
            // TODO init beam and min_cost (integer format)
            h_channel_params[ichannel] = params;
        }
        
        // Moving params to the device
        hipMemcpy(d_lane_params_, h_lane_params_, n_lanes_*sizeof(LaneParams), hipMemcpyHostToDevice);
        hipMemcpy(d_channel_params_, h_channel_params_, n_channels_*sizeof(ChannelParams), hipMemcpyHostToDevice)

        // Initialize host tokens memory pools
        for(int ichannel=0; ichannel<n_channels_; ++ichannel)
            h_all_tokens_info_.emplace_back(max_tokens_, copy_st_);

        // Using last one as init_channel_params
        init_channel_id_ = n_channels_-1;
        ComputeInitialChannel(init_channel_id);
        --n_channels_; // removing the init_channel_params from general list

        // infinite_cost : used as +INF for min_cost and d_state_cost
        // we will compute min_cost + beam during computation
        // if min_cost == FLT_MAX, we have an overflow
        // avoiding that by removing the beam from infinite
        // (2* the beam in case of rounding error)
        infinite_cost_ = FLT_MAX - 2*config.default_beam;

        // Setting Kernel Params
        // sent to kernels by copy

        // Making sure we'll be able to send it to the kernels
        KALDI_STATIC_ASSERT(sizeof(KernelsParams) < KALDI_CUDA_DECODER_MAX_KERNEL_ARGUMENTS_BYTE_SIZE);

        h_kernel_params_ = (KernelParams*)malloc(sizeof(KernelParams));
        h_kernel_params_->arc_ilabels = fst_.d_arc_ilabels_;
        h_kernel_params_->arc_weights = fst_.d_arc_weights_;
        h_kernel_params_->arc_nextstates = fst_.d_arc_nextstates_;
        h_kernel_params_->default_beam = default_beam_;
        h_kernel_params_->infinite_cost = infinite_cost_; 
        h_kernel_params_->q_capacity = max_tokens_per_frame_; 
        h_kernel_params_->init_channel_id = init_channel_id_; 

        if(KALDI_CUDA_DECODER_DEBUG_LEVEL > 0) {
            KALDI_LOG << "Running the decoder in debug level " << KALDI_CUDA_DECODER_DEBUG_LEVEL;
                     
            uint32_t debug_buffer_queue_size = max_tokens_per_frame_ + 1;
            hipHostMalloc(&h_debug_buf1_, std::max(fst_.num_states_, debug_buffer_queue_size) * sizeof(h_debug_buf1_));
            hipHostMalloc(&h_debug_buf2_, debug_buffer_queue_size * sizeof(h_debug_buf2_));
        }

        KALDI_DECODER_CUDA_CHECK_ERROR();
        num_frames_decoded_.resize(n_channels_);

        // Filling all best_state_cost with +INF
        dim3 grid,block;
        int32 nstates = fst_.NumStates();
        KALDI_ASSERT(nstates > 0);
        block.x = KALDI_CUDA_DECODER_KERNEL_GENERIC_DIMX;
        grid.x = KALDI_CUDA_DECODER_DIV_ROUND_UP(nstates, block.x);
        grid.z = n_lanes_;
        _init_state_best_cost_lookup_kernel<<<grid,block,0,compute_st_>>>(*kernel_params);

        // Making sure that everything is ready to use
        hipStreamSynchronize(compute_st_);
    }

    CudaDecoder::~CudaDecoder() {
        hipStreamDestroy(compute_st_);
        hipStreamDestroy(copy_st_);

        hipEventDestroy(can_read_h_main_q_narcs_);
        hipEventDestroy(can_write_to_main_q_);
        hipEventDestroy(can_read_final_h_main_q_end_);
        hipEventDestroy(before_finalize_nonemitting_kernel_);

        hipHostFree(h_lane_params);
        hipHostFree(h_channel_params);
        hipFree(d_lane_params);
        hipFree(d_channel_params);

        hipFree(d_all_aux_q_state_);
        hipFree(d_all_aux_q_cost_);
        hipFree(d_all_aux_q_info_);
        hipFree(d_all_main_q_info_);
        hipFree(d_all_state_best_cost_);
        hipFree(d_all_main_q_degrees_block_sums_prefix_sum_);

        hipFree(d_all_main_q_state_);
        hipFree(d_all_main_q_cost_);
        hipFree(d_all_main_q_arc_offsets_);
        hipFree(d_all_loglikelihoods_);

        hipHostFree(h_all_pinned_ints);
       
        if(KALDI_CUDA_DECODER_DEBUG_LEVEL > 0) {
            hipHostFree(h_debug_buf1_);
            hipHostFree(h_debug_buf2_);
        }
        free(h_kernel_params_);
        
        KALDI_DECODER_CUDA_CHECK_ERROR();

    }
    
    void CudaDecoder::ComputeInitialChannel() {
        // Lane used to compute init_channel_id_
        int32 lane_id = 0;

        // Filling the best state cost lookup table with +INF
        InitStateBestCostLookup(lane_id);

        // Adding the start state to the initial token queue
        StateId first_token_state;
        CostType first_token_cost;
        InfoToken first_token_info;

        first_token_state = fst_.Start();
        first_token_cost = StdWeight::One().Value();
        first_token_info.prev_token = INT_MIN;
        first_token_info.arc_idx = -1;

        KALDI_ASSERT(first_token_state != fst::kNoStateId);

        //
        // We add that initial token to the aux_q
        // it will be moved to the main_q during the ProcessNonemitting phase 
        // that will be called in a few lines
        //
        // Note : we launch copies in the compute stream here
        // It means that we want them to be in the main pipeline
        // compute_st_ is just a name - it's a generic CUDA stream
        //

        hipMemcpy(h_lane_params[lane_id].d_aux_q_state, &first_token_state, sizeof(StateId), hipMemcpyHostToDevice);
        hipMemcpy(h_lane_params[lane_id].d_aux_q_cost, &first_token_cost, sizeof(CostType), hipMemcpyHostToDevice);
        hipMemcpy(h_lane_params[lane_id].d_aux_q_info, &first_token_info, sizeof(InfoToken), hipMemcpyHostToDevice);

        // Updating the best state cost lookup table for the initial token state
        hipMemcpy(&h_lane_params[lane_id].d_state_best_cost[first_token_state], 
                        &first_token_cost, 
                        sizeof(IntegerCostType),
                        hipMemcpyHostToDevice);

        // We have one token is the aux_q
        int32 aux_q_end = 1;
        hipMemcpy(&d_lane_params[lane_id].aux_q_end, &aux_q_end, sizeof(*d_aux_q_end_), hipMemcpyHostToDevice);

        // Following kernels working channel_id
        h_kernel_params_->channel_to_compute[lane_id] = init_channel_id_;
        h_kernel_params_->nchannels = 1;

        // Initial ProcessNonEmitting
        PreprocessAndContract(aux_q_end);
        FinalizeProcessNonemitting(); 

        // Preparing for first frame + reverting back to init state (lookup table, etc.)
        int main_q_end;
        hipMemcpy(&main_q_end, &d_channel_params[init_channel_id_].frame_final_main_q_end, sizeof(int32), hipMemcpyDeviceToHost);
        PreprocessInPlace(main_q_end);
        ResetStateBestCostLookupAndFinalizePreprocessInPlace(main_q_end);

        // Saving init params on host
        hipMemcpy(h_channel_params[init_channel_id_], d_channel_params[init_channel_id_], sizeof(ChannelParams), hipMemcpyDeviceToHost);

        // Saving initial queue to host
        h_all_tokens_info_[init_channel_id_].CopyFromDevice(h_channel_params[init_channel_id_].d_main_q_info, main_q_size);

        // Waiting for copy to be done
        hipStreamSynchronize(copy_st_);

        KALDI_DECODER_CUDA_CHECK_ERROR();
    }

    void CudaDecoder::InitDecoding(const std::vector<ChannelId> &channels) {
        KALDI_ASSERT(channels.size() < n_lanes_);

        // Size of the initial main_q_size
        int init_main_q_size = h_channel_params_[init_channel_id_].final_frame_main_q_end;
        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_GENERIC_DIMX;
        grid.x = KALDI_CUDA_DECODER_DIV_ROUND_UP(init_main_q_size, block.x);
        grid.z = channels.size(); 

        // Getting *h_kernel_params ready to use
        SetChannelsInKernelParams(channels);

        // Initializing the main_q_end and everything else needed
        // to get the channels ready to compute new utterances
        init_decoding_on_device_kernel_<<<grid,block>>>(*h_kernel_params);

        // Tokens from initial main_q needed on host
        for(ChannelId channel_id : channels)
            h_all_tokens_info_[channel_id].Clone(h_all_tokens_info_[init_channel_id_]);
    }

    void CudaDecoder::AdvanceDecoding(DecodableInterface *decodable,
                                      const std::vector<ChannelId> &channels,
                                      int32 max_num_frames) {
        KALDI_ASSERT(num_frames_decoded_ >= 0 &&
                "You must call InitDecoding() before AdvanceDecoding()");
        
        int32 num_frames_ready = decodable->NumFramesReady();
        // num_frames_ready must be >= num_frames_decoded, or else
        // the number of frames ready must have decreased (which doesn't
        // make sense) or the decodable object changed between calls
        // (which isn't allowed).
        KALDI_ASSERT(num_frames_ready >= num_frames_decoded_);

        int32 target_frames_decoded = num_frames_ready;
        if (max_num_frames >= 0)
            target_frames_decoded = std::min(target_frames_decoded,
                    num_frames_decoded_ + max_num_frames);

        int32 n_lanes_used = channels.size();
        // We can process at most n_lanes_ channels at the same time
        KALDI_ASSERT(n_lanes_used < n_lanes_);

        // Setting up the  *kernel_params
        SetChannelsInKernelParams(channels);
        dim3 grid,block;
        block.x = 1;
        grid.x = 1;
        grid.z = n_lanes_used;
        // Getting the lanes ready to work with those channels  
        initialize_lanes_with_channels_<<<grid,block>>>(*kernel_params);

        // Loglikelihoods from the acoustic model
        // FIXME for now we duplicate the loglikelihoods 
        // to all channels for perf. measurement. 
        // We must decide which design to adopt
        ComputeLogLikelihoods(decodable);

        nvtxRangePushA("Decoding");
        
        int32 max_main_q_narcs = 0;
        // Looking for the channel with max numbers of arcs
        for(ChannelId channel_id : channels)
            max_main_q_narcs = std::max(max_main_q_narcs, h_channel_params_[channel_id].frame_final_main_q_narcs);

        while (num_frames_decoded_ < target_frames_decoded) {
            // Computing a new frame

            // ProcessEmitting 
            // 
            // Before executing ProcessEmitting, we have :
            // - The main_q contains tokens from the last frame
            // - The aux_q is empty
            //
            // ProcessEmitting will do the operation :
            //
            // read tokens from main_q ----FST---> create new tokens in the aux_q
            //
            // We will not write in the main q in that step
            // The input tokens are already in the main_q
            // (they were put there by the ProcessNonemittings 
            // from the previous frame)
            // We don't need can_write_to_main_q_
            // because we won't write to the main_q
            // The output tokens will go to aux_q

            // ProcessEmitting generates tokens associated with the new frame i
            // When we call ProcessEmitting, the main_q contains the tokens associated
            // with the previous frame (i-1). Using d_main_q_state and the emitting arcs from the FST graph,
            // we create a new tokens queue, which will be stored in the aux_q

            grid.x = KALDI_CUDA_DECODER_DIV_ROUND_UP(max_main_q_narcs, block.x);
            block.x = KALDI_CUDA_DECODER_KERNEL_GENERIC_DIMX;

            // Process emitting, expanding arcs
            _expand_arcs_kernel<<<grid,block,0,compute_st_>>>(*kernel_params_, true);

            // Post emitting phase. Resets the main_q.
            grid.x = 1; 
            block.x = 1;
            _post_expand_emitting<<<grid,block,0,compute_st_>>>(*kernel_params);

            // Updating the global_offsets on host
            for(ChannelId channel_id : channels) {
                h_channel_params_[channel_id].main_q_global_offset +=
                    h_channel_params_[channel_id].final_frame_main_q_end;
            }

            // Moving the lanes_params to host,
            // to have the aux_q_end values
            hipMemcpyAsync(h_lanes_params,     
                    d_lanes_params, 
                    n_lanes_*sizeof(LaneParams), 
                    hipMemcpyDeviceToHost,
                    compute_st_);

            hipStreamSynchronize(compute_st_);

            // Loglikelihoods from the acoustic model
            // We are done using loglikelihoods for current frame
            // Launching kernel for next frame now if there is one
            nvtxRangePop(); // Decoding
            if ((num_frames_decoded_+1) < target_frames_decoded) 
                ComputeLogLikelihoods(decodable);
            nvtxRangePushA("Decoding");

            // After ProcessEmitting we won't need the token
            // associated with the previous frame anymore
            // At the end of ProcessEmitting the main_q was flushed 
            // (by setting main_q_end == 0)
            // Tokens that were flushed at that step have been previously 
            // moved to the host memory 
            // We update the global offset of the main_q
            // the global offset takes into account all tokens that have been moved
            // to the host memory

            // ProcessNonemitting
            //
            // Processing non emitting arcs
            //
            // The operation is :
            //
            // PreprocessAndContract:
            // read input tokens from aux_q 
            //     ---contract (prune)--->
            // write non-pruned input tokens to main_q (append at the end of the queue)
            //
            // ExpandArc:
            // read input tokens from main_q 
            //     ---FST--->
            // create new tokens in the aux_q
            //
            // We then iterate those operations until no new tokens are created 
            //

            // We will write to main_q. We need it to be ready
            // for next kernels on compute_st_ 
            hipStreamWaitEvent(compute_st_, can_write_to_main_q_, 0);

            int32 max_aux_q_end = 0;
            bool finalize_nonemitting_was_executed = false;
            while(true) {
                for(LaneId lane_id=0; lane_id < n_lane_used; ++lane_id) {
                    int32 aux_q_end = h_lane_params[lane_id].aux_q_end;
                    max_aux_q_end = std::max(max_aux_q_end, aux_q_end);
                }

                grid.x = KALDI_CUDA_DECODER_DIV_ROUND_UP(max_aux_q_end, block.x);
                block.x = KALDI_CUDA_DECODER_KERNEL_GENERIC_DIMX;
                _preprocess_and_contract_kernel<<<grid,block,0,compute_st_>>>(*kernel_params);

                // Moving the lanes_params to host,
                // to have the main_q_narcs values
                hipMemcpyAsync(h_lanes_params,     
                        d_lanes_params, 
                        n_lanes_*sizeof(LaneParams), 
                        hipMemcpyDeviceToHost,
                        compute_st_);

                hipStreamSynchronize(compute_st_);

                for(LaneId lane_id=0; lane_id < n_lane_used; ++lane_id) {
                    int32 main_q_narcs = h_lane_params[lane_id].main_q_narcs;
                    max_main_q_narcs = std::max(max_aux_q_end, main_q_narcs);
                }
            
                // If we have only a few arcs, jumping to the one-CTA per channel persistent version
                if(max_main_q_narcs < KALDI_CUDA_DECODER_NONEM_LT_MAX_NARCS)
                    break;

                grid.x = KALDI_CUDA_DECODER_DIV_ROUND_UP(max_main_q_narcs, block.x);
                block.x = KALDI_CUDA_DECODER_KERNEL_GENERIC_DIMX;
                _expand_arcs_kernel<<<grid,block,0,compute_st_>>>(*kernel_params_, true);
                grid.x = 1; 
                block.x = 1;
                _post_expand_nonemitting<<<grid,block,0,compute_st_>>>(*kernel_params);

                // Moving the lanes_params to host,
                // to have the aux_q_end values
                hipMemcpyAsync(h_lanes_params,     
                        d_lanes_params, 
                        n_lanes_*sizeof(LaneParams), 
                        hipMemcpyDeviceToHost,
                        compute_st_);

                hipStreamSynchronize(compute_st_);

            }

            grid.x = 1;
            block.x = KALDI_CUDA_DECODER_KERNEL_NONEM_LT_DIMX;
            _finalize_process_non_emitting<<<grid,block,0,compute_st_>>>(*kernel_params);

            // No need to wait for the final main_q_end after FinalizeProcessNonEmitting,
            // it won't change much. Using the current value
            int32 max_main_q_end_estimate = 0;
            for(ChannelId channel_id : channels) {
                max_main_q_end_estimate = std::max(max_main_q_end_estimate,
                        h_lane_params_[channel_id].main_q_narcs);
            }

            // PreprocessInPlace for next ProcessEmitting
            // We do it here (and not at the beginning of the loop) to 
            // return the lane back to its original state after this frame computation
            // (preprocess in place is the last one to use the state_best_cost lookup)
            grid.x = KALDI_CUDA_DECODER_DIV_ROUND_UP(max_main_q_end_estimate, block.x);
            block.x = KALDI_CUDA_DECODER_KERNEL_GENERIC_DIMX;
            _preprocess_in_place_kernel<<<grid,block,0,compute_st_>>>(*kernel_params);
            // Resetting the lookup table for the next frame + FinalizePreprocessInPlace
            _finalize_frame_computation<<<grid,block>>>(*kernel_params);

            // Moving back to host the final (for this frame) values of :
            // - main_q_end
            // - main_q_narcs
            hipMemcpyAsync(h_lanes_params,     
                    d_lanes_params, 
                    n_lanes_*sizeof(LaneParams), 
                    hipMemcpyDeviceToHost,
                    compute_st_);

            hipStreamSynchronize(compute_st_);

            for(LaneId lane_id=0; lane_id < n_lane_used; ++lane_id) {
                int32 main_q_end = h_lane_params[lane_id].main_q_end;
                int32 main_q_narcs = h_lane_params[lane_id].main_q_narcs;
                ChannelId channel_id = channels[lane_id];
                h_channel_params[channel_id].final_frame_main_q_end = main_q_end;
                h_channel_params[channel_id].final_frame_main_q_narcs = main_q_narcs;
                // Computing for next iteration of current while loop
                max_main_q_narcs = std::max(max_main_q_narcs, main_q_narcs);
                // We are done with the current frame
                // We copy back its  tokens to the host
                // We only copy the "info" part (arc_idx + prev_token)
                // because we don't need anything else for the final backtrack
                // TODO buffer on device
                h_all_tokens_info_[channel_id].CopyFromDevice(h_lane_params[lane_id].d_main_q_info, main_q_end);
                num_frames_decoded_[channel_id]++; 
            }

            // We cannot write to the lanes.d_main_q_info 
            // until the copy is done
            hipEventRecord(can_write_to_main_q_, copy_st_);
            
            CheckOverflow();
            KALDI_DECODER_CUDA_CHECK_ERROR();
        }   
    
        nvtxRangePop();
    }


    void CudaDecoder::ComputeLogLikelihoods(DecodableInterface *decodable) {
        int32 frame = num_frames_decoded_;

        decodable->ComputeLogLikelihoods(d_loglikelihoods_,frame,fst_.max_ilabel_+1, compute_st_);
    }

    void CudaDecoder::CheckOverflow() {
            int32 q_overflow = *h_q_overflow_;
            if(q_overflow) {
                // An overflow was prevented in a kernel
                // The algorithm can still go on but quality of the result can be reduced
                // (less tokens were generated)
                KALDI_WARN << "Preventing overflow of the frame tokens. Pursuing "
                    << "execution but the quality of the output may be decreased. "
                    << "To prevent this from happening, please increase the parameter --max-tokens-per-frame"
                    << " and/or decrease --beam";

                *h_q_overflow_ = 0;
            }

    }


    // GetBestCost
    // CPU-only code
    // returns the minimum cost among all tokens cost in the current frame
    // also returns the index of one token with that min cost
    //
    // Only called at the end of the computation of one audio file
    // not optimized
    //
    void CudaDecoder::GetBestCost(bool isfinal, CostType *min, int32 *argmin) const {
        CostType best_cost = std::numeric_limits<CostType>::max();
        int32 min_cost_token_index;

        // we need h_main_q_end_ ready
        hipStreamSynchronize(compute_st_);

        // Copying the costs from current frame back to host memory
        // h_main_q_cost_ is never filled automatically 
        // when moving the tokens back to the host, we only move the { arc_idx, prev_token } part
        int32 main_q_size = *h_main_q_end_;
        hipMemcpyAsync(h_main_q_cost_, 
                        d_main_q_cost_, 
                        main_q_size * sizeof(*d_main_q_cost_), 
                        hipMemcpyDeviceToHost,
                        compute_st_);

        if(isfinal)
            hipMemcpyAsync(h_main_q_state_,     
                            d_main_q_state_, 
                            main_q_size * sizeof(*d_main_q_state_), 
                            hipMemcpyDeviceToHost,
                            compute_st_);

        // Waiting for data
        hipStreamSynchronize(compute_st_);


        // Finding best cost
        for(int32 i=0; i < main_q_size; ++i) {
            CostType cost = h_main_q_cost_[i];

            if(isfinal) 
                cost += fst_.h_final_[h_main_q_state_[i]];

            if(cost < best_cost) {
                best_cost = cost;
                min_cost_token_index = i;
            }
        }

        // The main_q always has a main_q_global_offset_
        min_cost_token_index += main_q_global_offset_; 

        // Saving result
        *min = best_cost;
        *argmin = min_cost_token_index;
    }


    //
    // ReachedFinal() returns true if the main_q contains a final state 
    // CPU-only code
    //
    // Only called at the end of the computation of one audio file
    // not optimized
    //
    bool CudaDecoder::ReachedFinal() const {
        // we need h_main_q_end_ ready
        hipStreamSynchronize(compute_st_);

        int32 main_q_size = *h_main_q_end_;
        
        // Copying the states from current frame back to host memory
        // h_main_q_state_ is never filled automatically 
        // when moving the tokens back to the host, we only move the { arc_idx, prev_token } part
        hipMemcpyAsync(h_main_q_state_,     
                d_main_q_state_, 
                main_q_size * sizeof(*d_main_q_state_), 
                hipMemcpyDeviceToHost,
                compute_st_);

        // Waiting for data
        hipStreamSynchronize(compute_st_);

        // Looking for a final state
        for(int32 i=0; i < main_q_size; ++i) {
            if(fst_.h_final_[h_main_q_state_[i]] != StdWeight::Zero().Value())
                return true;
        }

        return false;
    }



    //
    // GetBestPath is called at the end of the computation
    // It chooses the best token from the last frame, 
    // and backtracks all the path to the beginning (StartState)
    // from there
    // It then returns that path
    //
    bool CudaDecoder::GetBestPath(Lattice *fst_out, bool use_final_probs) const {
        nvtxRangePushA("GetBestPath");

        // We want the copy to host of the last tokens to be done
        hipEventSynchronize(can_write_to_main_q_);

        bool isfinal = ReachedFinal();

        // Finding the best token from the last frame
        // ie the token with min cost
        CostType best_cost;
        int32 token_with_best_cost;
        GetBestCost(isfinal, &best_cost, &token_with_best_cost);


        // Backtracking
        // Going all the way from the token with best cost
        // to the beginning (StartState)
        int32 token_idx = token_with_best_cost;
        std::vector<int32> reversed_path;

        // The first token was inserted at the beginning of the queue
        // it always has index 0
        // We backtrack until that first token
        while(token_idx != 0) {
            int32 arc_idx = h_all_tokens_info_.GetRawPointer()[token_idx].arc_idx;
            reversed_path.push_back(arc_idx);
            token_idx = h_all_tokens_info_.GetRawPointer()[token_idx].prev_token;
        }


        // Reset the fst_out
        fst_out->DeleteStates();

        // Building the output Lattice
        StateId cur_state = fst_out->AddState();
        fst_out->SetStart(cur_state);

        for (int32 i = reversed_path.size()-1; i >= 1; i--) {
            int32 arc_idx = reversed_path[i];

            LatticeArc arc(fst_.h_arc_ilabels_[arc_idx], 
                           fst_.h_arc_olabels_[arc_idx],
                           LatticeWeight(fst_.h_arc_weights_[arc_idx], 0), 
                           fst_.h_arc_nextstates_[arc_idx]);

            arc.nextstate = fst_out->AddState();
            fst_out->AddArc(cur_state, arc);
            cur_state = arc.nextstate;
        }

        // Adding final cost to final state
        if (isfinal && use_final_probs)
            fst_out->SetFinal(cur_state,
                    LatticeWeight(fst_.h_final_[fst_.h_arc_nextstates_[reversed_path[0]]], 0.0));
        else
            fst_out->SetFinal(cur_state, LatticeWeight::One());

        fst::RemoveEpsLocal(fst_out);

        nvtxRangePop();
        return true;
    }

    //
    // Debug functions
    // Called to verify that intermediate values are valid 
    //

    void CudaDecoder::DebugAssertsBeforeExpand(bool is_emitting) {
        hipStreamSynchronize(compute_st_);

        int32 main_q_end = *h_main_q_end_;
        int32 main_q_offset = *h_main_q_local_offset_;

        hipMemcpyAsync(h_main_q_state_,     
                d_main_q_state_,
                main_q_end * sizeof(*d_main_q_state_), 
                hipMemcpyDeviceToHost,
                compute_st_);

        unsigned int *h_arc_offsets = is_emitting ? fst_.h_e_offsets_ : fst_.h_ne_offsets_;

        int32 * h_prefix_sum = h_debug_buf1_;
        hipMemcpyAsync(h_prefix_sum,     
                d_main_q_degrees_prefix_sum_, 
                (main_q_end+1) * sizeof(*d_main_q_degrees_prefix_sum_), 
                hipMemcpyDeviceToHost,
                compute_st_);

        int32 * h_q_arc_offsets = h_debug_buf2_;
        hipMemcpyAsync(h_q_arc_offsets,     
                d_main_q_arc_offsets_,
                main_q_end * sizeof(*d_main_q_arc_offsets_), 
                hipMemcpyDeviceToHost,
                compute_st_);

        // Waiting for the copies
        hipStreamSynchronize(compute_st_);

        for(int32 i = main_q_offset; i < main_q_end; ++i) {
            int32 state = h_main_q_state_[i];
            KALDI_ASSERT(state >= 0);
            KALDI_ASSERT(state < fst_.num_states_);


            KALDI_ASSERT(h_prefix_sum[i] >= 0);
            KALDI_ASSERT(h_prefix_sum[i] <= h_prefix_sum[i+1]); 
            int32 degree_in_prefix_sum = h_prefix_sum[i+1] - h_prefix_sum[i];
            int32 degree_in_fst = h_arc_offsets[state+1] - h_arc_offsets[state];

            // Testing for degree == 0, which is possible in preprocessinplace
            // only possible if is_emitting, nonemitting uses contractandpreprocess
            if(is_emitting) {
                KALDI_ASSERT(degree_in_prefix_sum == 0 || degree_in_prefix_sum == degree_in_fst);
                // if degree == 0 arc_offsets may not be valid, but we won't use it
                KALDI_ASSERT(degree_in_prefix_sum == 0 || h_arc_offsets[state] == h_q_arc_offsets[i]); 
            } else {
                KALDI_ASSERT(degree_in_prefix_sum == degree_in_fst);
                KALDI_ASSERT(h_arc_offsets[state] == h_q_arc_offsets[i]); 
            }
        }
    }
    
    void CudaDecoder::DebugAssertsNewFrame() {
        hipStreamSynchronize(compute_st_);

        int32 float_inf_as_int = 2139095039; // FIXME use real infinite_cost_

        int32 nstates = fst_.num_states_;

        int *h_state_best_cost = h_debug_buf1_;
        hipMemcpyAsync(h_state_best_cost,     
                d_state_best_cost_,
                nstates * sizeof(*d_state_best_cost_), 
                hipMemcpyDeviceToHost,
                compute_st_);
        hipStreamSynchronize(compute_st_);

        for(int i=0; i<nstates; ++i)
            KALDI_ASSERT(h_state_best_cost[i] == float_inf_as_int);
    }

    void CudaDecoder::SetChannelsInKernelParams(const std::vector<ChannelId> &channels) {
        KALDI_ASSERT(channels.size() < n_lanes_);
        for(LaneId lane_id=0; lane_id<channels.size(); ++lane_id)
            h_kernel_params_->channel_to_compute[lane_id] = channels[lane_id];
        h_kernel_params_->nchannels = channels.size();
    }
} // end namespace kaldi.
