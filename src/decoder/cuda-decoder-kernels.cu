#include "hip/hip_runtime.h"
// decoder/cuda-decoder-kernels.cu

// See ../../COPYING for clarification regarding multiple authors
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//  http://www.apache.org/licenses/LICENSE-2.0
//
// THIS CODE IS PROVIDED *AS IS* BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, EITHER EXPRESS OR IMPLIED, INCLUDING WITHOUT LIMITATION ANY IMPLIED
// WARRANTIES OR CONDITIONS OF TITLE, FITNESS FOR A PARTICULAR PURPOSE,
// MERCHANTABLITY OR NON-INFRINGEMENT.
// See the Apache 2 License for the specific language governing permissions and
// limitations under the License.

#include <hipcub/hipcub.hpp>
#include "decoder/cuda-decoder.h"

#define DIV_ROUND_UP(a,b) ((a+b-1)/b)

namespace kaldi {

typedef CudaDecoder::StateId StateId;
typedef CudaDecoder::TokenAndArcCount TokenAndArcCount;
typedef CudaDecoder::TokenAndArcCountUnion TokenAndArcCountUnion;
typedef CudaDecoder::CostType CostType;
typedef CudaDecoder::PreprocessParams PreprocessParams; 
typedef CudaDecoder::ExpandArcParams ExpandArcParams; 

//
// Utils device function
//


    //
    // 1:1 Conversion float <---> sortable int
    // We convert floats to sortable ints in order
    // to use native atomics operation, which are 
    // way faster than looping over atomicCAS 
    //

    __device__ int32 floatToOrderedInt(float floatVal) {

        int32 intVal = __float_as_int( floatVal );

        return (intVal >= 0 ) ? intVal : intVal ^ 0x7FFFFFFF;
    }



    __device__ float orderedIntToFloat(int32 intVal) {

        return __int_as_float( (intVal >= 0) ? intVal : intVal ^ 0x7FFFFFFF );

    } 

    // Temporary used for cutoff - will be TODO removed
    __device__ float fatomicMin(float *addr, float value)

    {

        float old = *addr, assumed;
        if(old <= value) return old;

        do
        {
            assumed = old;
            old = atomicCAS((uint32_t*)addr,
                    __float_as_int(assumed),
                    __float_as_int(value));

        } while(old!=assumed); // TODO <

        return old;

    }

    //
    // Kernels
    //

    // For description of what each kernel is doing, please refer to cuda-decoder.h
    // and look for the corresponding wrapper
    // for instance, for a description of _init_lookup_kernel,
    // look for the description of CudaDecoder::InitStateCostLookup() in cuda-decoder.h

    // Used before first frame
    __global__ void _init_state_cost_lookup_kernel(int32 size, int32 *state_cost) {
        for(int32 idx = blockIdx.x*blockDim.x + threadIdx.x;
                idx < size;
                idx += blockDim.x*gridDim.x) {
            state_cost[idx]  = floatToOrderedInt(FLT_MAX);
        }
    }

    void CudaDecoder::InitStateCostLookup() {
        int32 nstates = fst_.numStates;
        KALDI_ASSERT(nstates > 0);

        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_KERNEL_INIT_LOOKUP_DIMX;
        grid.x = DIV_ROUND_UP(nstates, block.x);

        _init_state_cost_lookup_kernel<<<grid,block>>>(nstates, d_state_cost_);
    }

    // Used to reset lookup table between frames
    // Using the queue to reset only the values needed
    // Also takes care of resetting cutoff
    __global__ void _reset_state_cost_lookup_kernel(const StateId *d_main_q_state_, const int32 *d_main_q_end_, int32 *d_state_cost, CostType *d_cutoff) {
        int32 main_q_end = *d_main_q_end_; 

        for(int32 idx = blockIdx.x*blockDim.x + threadIdx.x;
                idx < main_q_end;
                idx += blockDim.x*gridDim.x) {
            // d_main_q_state_ contains the list of states that we've considered in the last frame
            // it corresponds to the list of indexes i such as d_state_cost[i] < +INF
            // faster than init_state_cost_lookup_kernel by a factor of ~10
            StateId state = d_main_q_state_[idx];
            d_state_cost[state]  = floatToOrderedInt(FLT_MAX);
        }

        if(blockIdx.x == 0 && threadIdx.x == 0)
            *d_cutoff = FLT_MAX; // we also reset the cutoff
    }

    void CudaDecoder::ResetStateCostLookup() {
        int32 size = *h_main_q_end_;

        KALDI_ASSERT(size > 0);

        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_KERNEL_INIT_LOOKUP_DIMX;
        grid.x = DIV_ROUND_UP(size, block.x);

        _reset_state_cost_lookup_kernel<<<grid,block,0,compute_st_>>>(d_main_q_state_, d_main_q_end_, d_state_cost_, d_cutoff);
    }


    // Sum operator for the TokenAndArcCount struct (2 ints) 
    // Used in preprocess_and_contract
    struct TokenAndArcCountSum {
        __device__ TokenAndArcCount operator()(const TokenAndArcCount &a, const TokenAndArcCount &b) const {
            TokenAndArcCount c;
            c.ntokens = a.ntokens + b.ntokens;
            c.narcs = a.narcs + b.narcs;

            return c;
        }
    };

    /*
       This kernel preprocess the necessary information for expand (scan of the outgoing degrees) 
       and explicitly prune the tokens

       The ExpandArc kernel writes the new raw token list in the aux_q. However, the cutoff 
       was progressively lowered during the computation, and some tokens now have a cost > cutoff.
       During the contract stage of this kernel, we remove such tokens. 
       We also remove duplicates, i.e. tokens pointing to the same state, but with token.cost > best_cost_for_that_state

       It contracts (by pruning) the queue list:
       raw output in aux_q ----contract----> pruned output in main q

       This kernel is responsible for :

       1) Read a token from the aux queue (raw output from previous expand)

       2) Compute the outgoing degree of that token.next_state. For that :
       -> If that token is suboptimal (cutoff, best_cost), we prune it
       -> Otherwise, we will move it to the main_q. We also read its arc degree in the FST graph 

       3) We move the non-pruned tokens into the main q. After a local prefix sum,
       we request a spot in the main_q for those tokens using the main_q_end_and_narcs counter. 
       main_q_end_and_narcs.split.end contains the number of tokens in the main q until now
       main_q_end_and_narcs.split.narcs contains the number of arcs in the main q until now

       We also compute the degrees prefix sum in one pass using the main_q_end_and_narcs.split.narcs

       This kernel is used before ProcessNonEmitting
    */

    // Important : pass the struct PreprocessParams by copy - passing it using a ref will not work (CPU -> GPU)
    __global__ void _preprocess_and_contract_kernel(PreprocessParams params) {
        
        // Prefix sum operator
        typedef hipcub::BlockScan<TokenAndArcCount, KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX> BlockScan;
        __shared__ typename BlockScan::TempStorage temp_storage;

        // This CUDA block (CTA) will count the number of tokens it has to move to the main_q
        // and store the result in nsurvival_tokens_in_CTA
        __shared__ int32 nsurvival_tokens_in_CTA;

        // We need to move the survival tokens to the main_q
        // 
        // main_q_global_block_offset has two purposes :
        // (1) to know where to store the survival tokens in the main_q
        // (2) to perform the prefix sum degrees of the survival degrees
        //
        // The reason why we store those two values together is because they are linked (see below)
        //
        // (1) We need a spot to store those tokens in the main_q 
        // We will ask the main_q counter where to store those tokens, the answer will be 
        // an offset of the main_q. We will store our tokens in positions :
        // d_main_q_state[main_q_global_block_offset.ntokens], d_main_q_state[main_q_global_block_offset.ntokens+1]...
        //
        // (2) main_q_global_block_offset.narcs contains the number of arcs in the main_q up until index main_q_global_block_offset.ntokens
        // ie the number of arcs going out of all states in d_main_q_state[0..main_q_global_block_offset.ntokens]
        // it is used to compute the global prefix sum of degrees in one pass
        //
        __shared__ TokenAndArcCountUnion main_q_global_block_offset;

        // Final cutoff from last ExpandArc execution
        const BaseFloat cutoff = *params.d_cutoff;

        const int32 aux_q_end = *params.d_aux_q_end;
        for(int32 block_offset = blockDim.x*blockIdx.x;
                block_offset < aux_q_end;
                block_offset += gridDim.x*blockDim.x) {

            int32 aux_q_idx = block_offset + threadIdx.x;
            int32 degree = 0;
            int32 arc_start = -1;

            StateId token_state;
            CostType token_cost;

            if(aux_q_idx < aux_q_end) {
                // Cost and state associated with the token
                token_cost = params.d_aux_q_cost[aux_q_idx];
                token_state = params.d_aux_q_state[aux_q_idx];

                // Best cost for that token_state
                // We know we have a token associated with token_state in the queue with the cost best_state_cost
                BaseFloat best_state_cost = orderedIntToFloat(params.d_state_cost[token_state]);

                // Cutoff may have decreased since the creation of the token
                if(token_cost < cutoff) {
                    
                    // We can have duplicates, ie token associated with the same states
                    // If this token is not the best candidate, get rid of it
                    if(token_cost == best_state_cost) {
                        arc_start = params.d_arc_offsets[token_state];
                        int32 arc_end = params.d_arc_offsets[token_state+1];
                        degree = arc_end - arc_start;
                    }
                }

                // the d_state_cost lookup table is reset to +INF for all states between frame
                // for perf. reason we only reset states that are in d_main_q_state
                // however if best_state_cost >= cutoff, all tokens associated with token_state 
                // will be pruned, and that state will not be in d_main_q_state
                // we need to reset the lookup table now

                if (best_state_cost >= cutoff)
                    params.d_state_cost[token_state] = floatToOrderedInt(FLT_MAX);

            }

            int32 is_pruned = (arc_start == -1);


            TokenAndArcCount block_prefix_sum_token_arc_count;

            // We now know which tokens will be moved to the main_q, the remaining will be pruned
            // we now compute a prefix sum inside the CUDA block to determine the local indexes of the survival tokens
            // the first survival token will have a index of 0, the second 1, ...
            block_prefix_sum_token_arc_count.ntokens =  is_pruned ? 0 : 1;
            
            // We also need to compute the prefix sum of the degrees
            // we start by doing a local prefix sum inside the CUDA block
            block_prefix_sum_token_arc_count.narcs =  degree;

            TokenAndArcCount zero_struct;
            zero_struct.ntokens = zero_struct.narcs = 0;

            // Computing the prefix sum (exclusive)
            BlockScan(temp_storage).ExclusiveScan(block_prefix_sum_token_arc_count, 
                                                    block_prefix_sum_token_arc_count, 
                                                    zero_struct,
                                                    TokenAndArcCountSum());

            
            TokenAndArcCountUnion token_and_arc_count_block_sum;
            if(threadIdx.x == (KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX-1)) {
                // This conditional branch is entered by the last thread
                // because it is the last, the prefix_sum of that thread contains the sum of all elts

                // We also add the value from this thread - the prefix sum is exclusive
                token_and_arc_count_block_sum.split.ntokens = block_prefix_sum_token_arc_count.ntokens + (is_pruned ? 0 : 1);
                token_and_arc_count_block_sum.split.narcs = block_prefix_sum_token_arc_count.narcs + degree;

                nsurvival_tokens_in_CTA = token_and_arc_count_block_sum.split.ntokens;
                
                // Doing two things at the same time :
                // requesting a spot in the main_q to store the survival tokens from this CTA 
                // (we need space for token_and_arc_count_block_sum.split.ntokens tokens)
                // informing the main_q that our survival tokens contain token_arc_count_block_sum.split.narcs arcs
                //
                // We then store the return value, which is the global offset on where to store those tokens,
                // and the total number of arcs up until that global offset
                main_q_global_block_offset.both = atomicAdd(&params.d_main_q_end_and_narcs_i2->both, token_and_arc_count_block_sum.both);
            }

            // Syncing for three reasons :
            // - Broadcasting main_q_global_block_offset
            // - Broadcasting nsurvival_tokens_in_CTA
            // - We may reuse temp_storage (cf CUB doc)
            __syncthreads(); 

            // Checking if we are overflowing the main_q
            if((main_q_global_block_offset.split.ntokens + nsurvival_tokens_in_CTA) >= params.q_capacity) {
                if(threadIdx.x == (KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX-1)) {
                    // We are overflowing the main_q
                    // We first revert what this CTA has done, ie revert the previous atomicAdd
                    // because all CTAs will revert, we know we will have a valid state after completion of this kernel
                    atomicAdd(&params.d_main_q_end_and_narcs_i2->both, -token_and_arc_count_block_sum.both); // revert

                    // Setting the flag. It will print a warning to stderr
                    *params.h_q_overflow = 1;
                }

                // We abort computation, we no longer have space in the main_q.
                // We still jump to finalize_kernel, to do what's needed before completion
                goto finalize_kernel;
            }

            // If we are executing the following lines it means that we are not overflowing the queue
            // We then continue what we were doing

            if(!is_pruned) {
                // This thread is in charge of a survival token
                // we will move it to the main_q, at index main_q_idx

                // Note : we could remove the branch divergence here 

                int32 main_q_idx = main_q_global_block_offset.split.ntokens + block_prefix_sum_token_arc_count.ntokens;

                InfoToken token_info = params.d_aux_q_info[aux_q_idx];

                // Moving the token to the main q
                params.d_main_q_state[main_q_idx] = token_state;
                params.d_main_q_cost[main_q_idx] = token_cost;
                params.d_main_q_info[main_q_idx] = token_info;

                // Saving the global prefix sum
                // = (narcs until now in the main queue) + (narcs until this thread in the CTA)
                params.d_main_q_degrees_prefix_sum[main_q_idx] = main_q_global_block_offset.split.narcs 
                                                                 + block_prefix_sum_token_arc_count.narcs;

                // Saving the CSR arc offset for that token's state
                // it will be used by the expand kernel, and avoid doing a new random memory access in the expand kernel
                params.d_main_q_arc_offsets[main_q_idx] = arc_start;
            }
        }

        finalize_kernel:

        // Avoiding races 
        // We will write d_aux_q_end
        // And some threads may be still reading it 
        // At the beg of this kernel
        __syncthreads();
        
        if(threadIdx.x == 0) {
            // Declaring the CTA as done
            int32 old = atomicAdd(params.d_n_CTA_done, 1);

            // If we're the last CTA to exit, detect it
            bool is_last_CTA = (old == (gridDim.x -1));

            if(is_last_CTA) {
                __threadfence();

                // We added things to the main_q
                // d_main_q_end was modified
                // we update h_main_q_end to keep it consistent
                // the h_* pointers are in the pinned host memory, we can access them from the device
                *params.h_main_q_end = *params.d_main_q_end;
                *params.h_main_q_narcs = *params.d_main_q_narcs;

                // We moved what we had to move from the aux q to the main q
                // We now empty the aux q 
                *params.d_aux_q_end = 0;
                *params.h_aux_q_end = 0; 

                // Reset the counter for next time
                *params.d_n_CTA_done = 0;
            }
        }

    }


    void CudaDecoder::PreprocessAndContract(const PreprocessParams &params) {
        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX;
        grid.x = DIV_ROUND_UP(*h_aux_q_end_, block.x);

        KALDI_ASSERT(grid.x > 0);

        _preprocess_and_contract_kernel<<<grid,block,0,compute_st_>>>(params);
    }



/*
    This kernel is also a preprocessing kernel, but this time does it in place
    The tokens are already in the main q (they were placed here by a previous "contract and preprocess").
    We avoid performing the next phase on non-optimal ones by setting the degree to 0 and
    computing a degrees scan.

    Here we have to do the scan in two passes : the scan will be finished in "finalize_preprocess"

    This preprocess step is used in ProcessEmitting. Tokens were placed in main_q by
    the ProcessNonEmitting of the previous frame. We cannot renumber them (it would break
    the prev_token index). We preprocess in place, leaving things as they are in main_q

*/

    __global__ void _preprocess_in_place_kernel(PreprocessParams params) {
    
        typedef hipcub::BlockScan<int32, KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX> BlockScan;
        __shared__ typename BlockScan::TempStorage temp_storage;

        __shared__ int32 is_last_CTA;

        int32 queue_offset = *params.d_main_q_local_offset;
        int32 queue_end = *params.d_main_q_end;
        int32 queue_size = queue_end - queue_offset;

        BaseFloat cutoff = *params.d_cutoff;

        for(int32 block_offset = blockDim.x*blockIdx.x;
                block_offset < queue_size;
                block_offset += gridDim.x*blockDim.x)
        {
            int32 idx = queue_offset + block_offset + threadIdx.x; 
            int32 degree = 0; 
            if(idx < queue_end) {
                StateId state_idx = params.d_main_q_state[idx]; 
                BaseFloat cost = params.d_main_q_cost[idx];

                if(cost < cutoff) {
                    BaseFloat best_cost = orderedIntToFloat(params.d_state_cost[state_idx]); 
                    if(cost == best_cost) {
                        int32 start = params.d_arc_offsets[state_idx]; 
                        int32 end = params.d_arc_offsets[state_idx+1]; 
                        degree  = end - start;
                        params.d_main_q_arc_offsets[idx] = start;
                    }
                }
            }

            int32 scan;
            BlockScan(temp_storage).ExclusiveSum(degree, scan);
            if(idx < queue_end) 
                params.d_main_q_degrees_prefix_sum[idx] = scan;


            if(threadIdx.x == (KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX-1))
                params.d_main_q_degrees_block_prefix_sum[block_offset/KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX] = (scan + degree); 

            __syncthreads(); // we'll reuse temp_storage
        }

        if(threadIdx.x == 0) {
            int32 old = atomicAdd(params.d_n_CTA_done, 1); 
            is_last_CTA = (old == (gridDim.x -1));
        }

        // is_last_CTA + temp_storage reuse
        __syncthreads();
        
        if(is_last_CTA)
        {
            // The last block alive takes care of scan of block sums 
            __threadfence();

            if(threadIdx.x == 0) {
                *params.d_n_CTA_done = 0;
            }

            // following value can be different than gridDim.x 
            int32 total_blk_val = (queue_size + KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX -1) / KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX;
            int32 scan_offset = 0;

            for(int32 blk_idx_off = 0; blk_idx_off < total_blk_val; blk_idx_off += blockDim.x) {
                int32 blk_idx = blk_idx_off + threadIdx.x; 

                int32 blk_sum = (blk_idx < total_blk_val) ?  params.d_main_q_degrees_block_prefix_sum[blk_idx] : 0; 
                int32 blk_scan, iteration_total;
                BlockScan(temp_storage).ExclusiveSum(blk_sum, blk_scan, iteration_total);
                blk_scan += scan_offset;
                scan_offset += iteration_total;

                if(blk_idx < total_blk_val) {
                    params.d_main_q_degrees_block_prefix_sum[blk_idx] = blk_scan;
                }

                // temp storage
                __syncthreads();
            }

            if(threadIdx.x == 0)
            {
                *params.d_main_q_narcs = scan_offset; 
                *params.h_main_q_narcs = scan_offset; // pinned memory
            }
        }
    }


    void CudaDecoder::PreprocessInPlace(const PreprocessParams &params) {
        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX;
        int32 main_q_size = *h_main_q_end_ - *h_main_q_local_offset_;

        grid.x = DIV_ROUND_UP(main_q_size, block.x);

        // If the main_q is empty, we will not be able to continue
        KALDI_ASSERT(grid.x > 0);

        _preprocess_in_place_kernel<<<grid,block,0,compute_st_>>>(params);
    }



    /*

       Part 2 of the scan for "PreprocessEmitting". For NonEmitting scan is already final

       Computes global prefix sum with block prefix sum and block offsets

       If we want to speed up expand, we can compute lower and upper bound to restrain 
       the binary search in expand
       This can be done on the fly here, and removes main bottleneck of expand
       Not done for now, because expand is fast enough

     */
    __global__ void _finalize_degrees_scan_kernel(int32 *d_scan, int32 *d_blk_scan, const int32 *d_main_q_local_offset_, const int32
            *d_main_q_end_) {

        int32 q_off = *d_main_q_local_offset_;
        int32 q_end = *d_main_q_end_;
        int32 q_size = q_end - q_off;

        for(int32 idx = q_off + blockDim.x*blockIdx.x + threadIdx.x;
                idx < q_size;
                idx += blockDim.x*gridDim.x) {

            int32 blk_idx = (idx - q_off) / KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX;
            int32 blk_scan_offset = d_blk_scan[blk_idx]; // we rely on L1 for this one, avoiding syncs

            d_scan[idx] += blk_scan_offset;
        }

    }

    void CudaDecoder::FinalizePreprocessInPlace() {
        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX;
        int32 main_q_size = *h_main_q_end_ - *h_main_q_local_offset_;
        grid.x = DIV_ROUND_UP(main_q_size, block.x);

        // If the main_q is empty, we will not be able to continue
        KALDI_ASSERT(grid.x > 0);

        _finalize_degrees_scan_kernel<<<grid,block,0,compute_st_>>>(d_main_q_degrees_prefix_sum_, d_main_q_degrees_block_prefix_sum_, d_main_q_local_offset_,
                d_main_q_end_); 
    }




    /*
       This kernel propagates arcs from the main q [main_q_local_offset, main_q_end[
       to the aux

       The main bottleneck is the first binary search. 
       If we want to remove it, preprocess it on the fly in preprocess

     */

    struct CostTInt {
        CostType cost;
        int32 i;
    };

    struct CISum {
        __device__ CostTInt operator()(const CostTInt &a, const CostTInt &b) const {
            CostTInt c;
            c.cost = fmin(a.cost, b.cost);
            c.i = a.i + b.i;

            return c;
        }
    };


__device__ __inline__ CostType GetCutoffCandidate(const CostType current_cutoff,
                                const CostType min_cost,
                                const CostType default_beam,
                                const int32 q_size,
                                const int32 q_capacity) {
                                 

    // Doing something simple for now
    // We have to keep beam large enough,
    // the final cutoff will be used for the final
    // prune. If it is too small, we won't keep enough tokens

   CostType beam = default_beam;

   if(q_size >= q_capacity/2) 
       beam /= 2;

    return fmin(current_cutoff, min_cost + beam);
}

    __forceinline__ __device__ int32 binsearch_maxle(const int32 *vec, const int32 val, int32 low, int32 high) {
        while(true) {
            if(low == high)
                return low; //we know it exists
            if((low + 1) == high)
                return (vec[high] <= val) ? high : low;

            int32 mid = low + (high- low) / 2;

            if(vec[mid] > val)
                high = mid-1;
            else
                low = mid;
        }
    }


    void __global__ _expand_arcs_kernel(ExpandArcParams params) {
        typedef hipcub::BlockScan<CostTInt, KALDI_CUDA_DECODER_KERNEL_EXPAND_ARCS_DIMX> BlockScan;

        __shared__ typename BlockScan::TempStorage temp_storage_scan;

        __shared__ int32 to_q_block_offset;
        __shared__ CostType blk_cutoff;

        const int32 total_narcs = *params.d_main_q_narcs;
        const int32 main_q_offset = *params.d_main_q_local_offset;
        const int32 main_q_end = *params.d_main_q_end;

        
        if(threadIdx.x == 0) {
            blk_cutoff = *params.d_cutoff;
        }

        __syncthreads();

        // Keeping the whole CTA alive, we'll have syncs
        for(int32 block_offset = blockDim.x*blockIdx.x;
                block_offset < total_narcs;
                block_offset += gridDim.x*blockDim.x) {

            int32 th_idx = block_offset + threadIdx.x;
            bool valid_input = (th_idx < total_narcs);

            BaseFloat total_cost = FLT_MAX;
            int32 arc_idx;
            StateId arc_next_state;
            int32 main_q_idx;

            if(valid_input) {
                //we can do better than that
                main_q_idx = binsearch_maxle(params.d_main_q_degrees_prefix_sum, th_idx, main_q_offset, main_q_end-1); 

                int32 lower_bound = params.d_main_q_degrees_prefix_sum[main_q_idx];
                int32 arc_offset_start = params.d_q_arc_offsets[main_q_idx];

                arc_idx = arc_offset_start + (block_offset + threadIdx.x - lower_bound);
                arc_next_state = params.arc_nextstates[arc_idx];

                total_cost = params.arc_weights[arc_idx];

                int32 arc_ilabel = params.is_emitting ? params.arc_ilabels[arc_idx] : 0;
                total_cost += (arc_ilabel != 0) ? -params.d_loglikelihoods[arc_ilabel] : 0.0; 
                total_cost += params.d_main_q_cost[main_q_idx];

                if(total_cost >= blk_cutoff)
                    valid_input = false;
                else {
                    // switch back to red, worst case is bad
                    BaseFloat next_state_cost = orderedIntToFloat(params.d_lookup[arc_next_state]);

                    if(total_cost >= next_state_cost)
                        valid_input = false;
                }
            }

                            int32 has_successor = valid_input ? 1 : 0;  // Need a spot in the new q
                            CostTInt ci;
                            ci.cost = valid_input ? total_cost : FLT_MAX; 
                            ci.i = has_successor;

                            BlockScan(temp_storage_scan).InclusiveScan(ci, ci, CISum());

                            if(threadIdx.x == (KALDI_CUDA_DECODER_KERNEL_EXPAND_ARCS_DIMX - 1)) {
                                int32 total_successors_in_block = ci.i;
                                to_q_block_offset = atomicAdd(params.d_aux_q_end, total_successors_in_block);
                                if((to_q_block_offset + total_successors_in_block) >= params.q_capacity) {
                                    to_q_block_offset = params.q_capacity; // used to broadcast the info

                                }
                                /*
                                
                                GetCutoffCandidate takes int32o account the current value of 
                                d_aux_q_end and compares it with its maximum capacity.
                                If necessary it progressively cuts down the beam 
                                (reducing the cutoff) to only keep the best candidates
                                and avoiding an overflow

                                */
                                CostType cutoff_candidate = GetCutoffCandidate(blk_cutoff,
                                                                  ci.cost,
                                                                  params.beam,
                                                                  to_q_block_offset + total_successors_in_block,
                                                                  params.q_capacity);

                                blk_cutoff = (cutoff_candidate < blk_cutoff) 
                                             ? fmin(fatomicMin(params.d_cutoff, cutoff_candidate), cutoff_candidate)
                                             : fmin(*params.d_cutoff, blk_cutoff);
                            }

                            __syncthreads(); // to_q_block_offset


                            // aux_q is full. UpdateCutoff should prevent this from happening
                            if(to_q_block_offset == params.q_capacity) {
                                if(threadIdx.x == (KALDI_CUDA_DECODER_KERNEL_EXPAND_ARCS_DIMX - 1)) {
                                    // Revert
                                    int32 total_successors_in_block = ci.i;
                                    atomicAdd(params.d_aux_q_end, -total_successors_in_block); 
                                    *params.h_q_overflow = 1; 
                                }

                                goto finalize_kernel; // keeping things clean before aborting
                            }

                            ci.i -= has_successor; // we want the exclusive sum now
                            int32 to_q_index = to_q_block_offset + ci.i;

                            if(has_successor) {
                                params.d_aux_q_cost[to_q_index] = total_cost;
                                params.d_aux_q_state[to_q_index] = arc_next_state;
                                
                                atomicMin(&params.d_lookup[arc_next_state],
                                floatToOrderedInt(total_cost)
                                );

                                //print32f("cost = %f, cutoff = %f, beam=%f \n", total_cost, blk_cutoff, params.beam);
                                if(total_cost < blk_cutoff) { // cutoff may have changed
                                    // We write the rest of the token only if necessary
                                    // if the cost is higher than cutoff, 
                                    // the token will be ignored anyway 


                                    InfoToken new_tok_info;
                                    new_tok_info.prev_token = params.main_q_global_offset + main_q_idx;
                                    new_tok_info.arc_idx = arc_idx;
                            

                                    params.d_aux_q_info[to_q_index] = new_tok_info;

                                    /*
                                    print32f("expand, adding %i (%i)  -> %i \n", new_tok_info.prev_token,
                                    params.main_q_global_offset, arc_next_state);
                                    */
                                }
                            }
        }

        finalize_kernel:

        __syncthreads(); // avoiding races on d_main_q_narcs for instance

        // Last block alive sets h_aux_q_end_ (pinned memory)
        if(threadIdx.x == 0) {
            int32 old = atomicAdd(params.d_n_CTA_done, 1);
            if(old == (gridDim.x -1)) {
                __threadfence(); // we want last value of d_aux_q_end
                *params.h_aux_q_end = *params.d_aux_q_end;
                *params.d_n_CTA_done = 0;
                *params.d_main_q_narcs = 0;
                *params.h_main_q_narcs = 0;

                if(params.is_emitting) {
                    *params.d_main_q_local_offset = 0; // not needed
                    *params.h_main_q_local_offset = 0; // not needed
                    *params.d_main_q_end = 0;
                    *params.h_main_q_end = 0;
                } else {
                    *params.d_main_q_local_offset = main_q_end;
                    *params.h_main_q_local_offset = main_q_end;
                }

            }
        }

    }

    void CudaDecoder::ExpandArcs(const ExpandArcParams &params, int32 nthreads) {
        dim3 grid,block;
        block.x = 256;
        grid.x = DIV_ROUND_UP(nthreads, block.x);

        // It's possible to have zero threads and still be valid
        if(grid.x > 0)
            _expand_arcs_kernel<<<grid,block,0,compute_st_>>>(params);
    }


    // Wrote for single CTA

    /*

       Persistent kernel

       Used to avoid calling multiple "heavy lifting" kernels for the tail of non emitting
       (lots of iterations with small number of arcs)

       Code is greatly simplified because we can have only one CTA alive

       Repeat until new queue empty:
       1) Computes degrees (cf ComputeDegrees) 
       2) Compute scan
       3) Expand arcs

       1 and 2 are not done on the first iteration, because it's already done
       (by corresponding kernels)

       At the end, this kernel finalize the computation for current frame,
       so that it's ready for next ProcessEmitting

       We could optimize and speed up this kernel
       It will only gives us a better latency for 1 stream, which is low enough
       Instead, we let it compute while we use the GPU for other streams
       This kernel only uses one block

     */


    __launch_bounds__(KALDI_CUDA_DECODER_KERNEL_NONEM_LT_DIMX, 1)
        __global__ void _process_nonem_longtail(const uint32_t *d_arc_offsets, 
                ExpandArcParams params) {

            typedef hipcub::BlockScan<int32, KALDI_CUDA_DECODER_KERNEL_NONEM_LT_DIMX> BlockScan;
            typedef hipcub::BlockReduce<float, KALDI_CUDA_DECODER_KERNEL_NONEM_LT_DIMX> BlockReduce;

            __shared__ typename BlockScan::TempStorage temp_storage_scan;
            __shared__ typename BlockReduce::TempStorage temp_storage_reduce;

            __shared__ BaseFloat cutoff;


            int32 old_q_offset = *params.d_main_q_local_offset;
            int32 new_q_offset = *params.d_main_q_end;
            int32 new_q_end = new_q_offset;

            int32 total_narcs = *params.d_main_q_narcs;
    
            int32 old_q_size = new_q_offset - old_q_offset;  // move to end

            cutoff = *params.d_cutoff;

            // We'll switch queue at the beg of the loop
            // Cleaner that way - we need the offsets ready for
            // the global updates at the very end of this kernel
            new_q_offset = old_q_offset;

            bool first = true;

            while(old_q_size > 0) {
                // Step 0 : move queues        
                old_q_offset = new_q_offset;
                new_q_offset = new_q_end;

                if(!first) {
                    __syncthreads(); // old_q_ready
                    total_narcs = 0;

                    // Step 1 : compute_degrees
                    // TODO fuse 1 and 2
                    for(int32 q_idx = old_q_offset + threadIdx.x;
                            q_idx < new_q_offset; // = old_q_end
                            q_idx += blockDim.x) {

                        StateId state = params.d_main_q_state[q_idx];
                        BaseFloat cost = params.d_main_q_cost[q_idx];

                        int32 degree = 0;
                        if(cost < cutoff) {
                            BaseFloat best_cost = orderedIntToFloat(params.d_lookup[state]);

                            if(cost == best_cost) {
                                int32 start = d_arc_offsets[state];
                                int32 end = d_arc_offsets[state+1];
                                degree = end - start;
                                params.d_q_arc_offsets[q_idx] = start;
                            }
                        }

                        params.d_main_q_degrees_prefix_sum[q_idx] = degree;
                    }

                    __syncthreads(); // will be removed

                    // Step 2 : Scan

                    for(int32 block_off = 0;
                            block_off < old_q_size;
                            block_off += blockDim.x) {

                        int32 q_idx = old_q_offset + block_off + threadIdx.x;

                        int32 degree = (q_idx < new_q_offset) 
                            ? params.d_main_q_degrees_prefix_sum[q_idx]
                            : 0;
                        int32 lscan;
                        int32 total_in_blk;
                        BlockScan(temp_storage_scan).ExclusiveSum(degree, lscan, total_in_blk);
                        int32 scan = lscan + total_narcs;
                        total_narcs += total_in_blk;

                        if(q_idx < new_q_offset)
                            params.d_main_q_degrees_prefix_sum[q_idx] = scan;

                         __syncthreads(); // reusing temp_storage_scan + degrees ready
                    }


                } else {
                    first = false;    
                }


                // We already sync'ed

                // Step 3 : expand arcs

                for(int32 block_offset = 0;
                        block_offset < total_narcs;
                        block_offset += blockDim.x) {

                    int32 th_idx = block_offset + threadIdx.x;
                    bool valid_input = (th_idx < total_narcs);

                    BaseFloat total_cost = FLT_MAX;
                    int32 arc_idx;
                    StateId arc_next_state;
                    int32 q_idx;

                    if(valid_input) {
                        //we can do better than that
                        q_idx = binsearch_maxle(params.d_main_q_degrees_prefix_sum, th_idx, old_q_offset, new_q_offset-1); 

                        int32 lower_bound = params.d_main_q_degrees_prefix_sum[q_idx];
                        int32 arc_offset_start = params.d_q_arc_offsets[q_idx];

                        arc_idx = arc_offset_start + (th_idx - lower_bound);

                        arc_next_state = params.arc_nextstates[arc_idx];
                        BaseFloat arc_weight = params.arc_weights[arc_idx];
                        BaseFloat next_state_cost = orderedIntToFloat(params.d_lookup[arc_next_state]);
                        BaseFloat old_tok_cost = params.d_main_q_cost[q_idx];

                        total_cost = arc_weight + old_tok_cost;

                        if(total_cost >= next_state_cost) {
                            total_cost = FLT_MAX;
                            valid_input = false; 
                        } 
                    }

                    BaseFloat min_cost = BlockReduce(temp_storage_reduce).Reduce(total_cost, hipcub::Min());

                    if(threadIdx.x == 0) {
                        cutoff = GetCutoffCandidate(cutoff,
                                min_cost,
                                params.beam,
                                new_q_end,
                                params.q_capacity);
                    }

                    __syncthreads();

                    int32 has_successor = (total_cost < cutoff && valid_input) ? 1 : 0;

                    if(has_successor) 
                        atomicMin(&params.d_lookup[arc_next_state], floatToOrderedInt(total_cost));

                    int32 new_q_idx_block = has_successor;
                    int32 total_in_blk;
                    BlockScan(temp_storage_scan).ExclusiveSum(new_q_idx_block, new_q_idx_block, total_in_blk);

                    if((new_q_end + total_in_blk) >= params.q_capacity) {
                        *params.h_q_overflow = 1;
                        
                        goto finalize_kernel; // keeping things clean before aborting
                    }

                    if(has_successor) {
                        int32 new_q_index = new_q_end + new_q_idx_block;
                        params.d_main_q_state[new_q_index] = arc_next_state;

                        params.d_main_q_cost[new_q_index] = total_cost;

                        InfoToken new_tok_info;
                        new_tok_info.prev_token = params.main_q_global_offset + q_idx;

                        new_tok_info.arc_idx = arc_idx;
                        params.d_main_q_info[new_q_index] = new_tok_info;
                        
                        //print32f("new q index = %i (%i+%i) (tot=%i) \n", new_q_index, new_q_end, new_q_idx_block,
                        //total_in_blk);
                   }

                    new_q_end += total_in_blk;
                }

                old_q_size = new_q_end - new_q_offset; 
            }

            finalize_kernel:

            if(threadIdx.x == 0) {
                // Next step is ProcessEmitting of next frame, from is currToken_offset
                *params.d_main_q_end = new_q_end; 
                *params.d_main_q_narcs = 0;

                *params.h_main_q_end = new_q_end; 
                *params.h_main_q_narcs = 0; 

                *params.d_main_q_local_offset = 0; 
                *params.h_main_q_local_offset = 0; 

                *params.d_cutoff = cutoff;
            }

        }

    void CudaDecoder::NonEmittingLongTail(const uint32_t *d_arc_offsets, 
            const ExpandArcParams &params) {

        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_KERNEL_NONEM_LT_DIMX;
        grid.x = 1; // it is designed for the long tail
        _process_nonem_longtail<<<grid,block,0,compute_st_>>>(d_arc_offsets, params);
    }


} // end namespace kaldi
