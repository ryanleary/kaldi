#include "hip/hip_runtime.h"
// decoder/cuda-decoder-kernels.cu

// 2018 - Hugo Braun, Justin Luitjens, Ryan Leary

// See ../../COPYING for clarification regarding multiple authors
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//  http://www.apache.org/licenses/LICENSE-2.0
//
// THIS CODE IS PROVIDED *AS IS* BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, EITHER EXPRESS OR IMPLIED, INCLUDING WITHOUT LIMITATION ANY IMPLIED
// WARRANTIES OR CONDITIONS OF TITLE, FITNESS FOR A PARTICULAR PURPOSE,
// MERCHANTABLITY OR NON-INFRINGEMENT.
// See the Apache 2 License for the specific language governing permissions and
// limitations under the License.

#include <hipcub/hipcub.hpp>
#include "decoder/cuda-decoder.h"

#define KALDI_CUDA_DECODER_DIV_ROUND_UP(a,b) ((a+b-1)/b)

namespace kaldi {
	typedef CudaDecoder::StateId StateId;
	typedef CudaDecoder::TokenAndArcCount TokenAndArcCount;
	typedef CudaDecoder::TokenAndArcCountUnion TokenAndArcCountUnion;
	typedef CudaDecoder::CostType CostType;
	typedef CudaDecoder::MinCostAndBeamIntegers MinCostAndBeamIntegers;
	typedef CudaDecoder::MinCostAndBeam MinCostAndBeam;
	typedef CudaDecoder::IntegerCostType IntegerCostType;
	typedef CudaDecoder::PreprocessParams PreprocessParams; 
	typedef CudaDecoder::ExpandArcParams ExpandArcParams; 

	// In AdvanceDecoding,
	// the lane lane_id will compute the channel
	// with channel_id = channel_to_compute[lane_id]


	//
	// Utils device function
	//


	//
	// 1:1 Conversion float <---> sortable int
	// We convert floats to sortable ints in order
	// to use native atomics operation, which are 
	// way faster than looping over atomicCAS 
	//

	__device__ int32 floatToOrderedInt(float floatVal) {
		int32 intVal = __float_as_int( floatVal );

		return (intVal >= 0 ) ? intVal : intVal ^ 0x7FFFFFFF;
	}



	__device__ float orderedIntToFloat(int32 intVal) {
		return __int_as_float( (intVal >= 0) ? intVal : intVal ^ 0x7FFFFFFF );
	} 

	// Kernels

	// Used to initialize the lane lookup tables in CudaDecoder's constructor
	__global__ void _init_state_best_cost_lookup_kernel(KernelParams params) {
		const int nlanes = params.nlanes;
		KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
			const int num_states = params.num_states;
			KALDI_CUDA_DECODER_1D_KERNEL_LOOP(istate, num_states) {
				// Reset lookup table
				lane_params.d_state_best_cost.lane(ilane)[istate] = INT_MAX; 
			}
		}
	}

	/*
	   This kernel preprocess the necessary information for expand (scan of the outgoing degrees) 
	   and explicitly prune the tokens

	   The ExpandArc kernel writes the new raw token list in the aux_q. However, the cutoff 
	   was progressively lowered during the computation, and some tokens now have a cost > cutoff.
	   During the contract stage of this kernel, we remove such tokens. 
	   We also remove duplicates, i.e. tokens pointing to the same state, but with token.cost > best_cost_for_that_state

	   It contracts (by pruning) the queue list:
	   raw output in aux_q ----contract----> pruned output in main q

	   This kernel is responsible for :

	   1) Read a token from the aux queue (raw output from previous expand)

	   2) Compute the outgoing degree of that token.next_state. For that :
	   -> If that token is suboptimal (cutoff, best_cost), we prune it
	   -> Otherwise, we will move it to the main_q. We also read its arc degree in the FST graph 

	   3) We move the non-pruned tokens into the main q. After a local prefix sum,
	   we request a spot in the main_q for those tokens using the main_q_end_and_narcs counter. 
	   main_q_end_and_narcs.split.end contains the number of tokens in the main q until now
	   main_q_end_and_narcs.split.narcs contains the number of arcs in the main q until now

	   We also compute the degrees prefix sum in one pass using the main_q_end_and_narcs.split.narcs

	   This kernel is used before ProcessNonEmitting
	 */

	// Important : pass the struct PreprocessParams by copy - passing it using a ref will not work (CPU -> GPU)
	__global__ void _preprocess_and_contract_kernel(KernelParams params) {
		typedef hipcub::BlockScan<int2, KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX> BlockScan;
		__shared__ typename BlockScan::TempStorage sh_temp_storage;

		// This CUDA block (CTA) will count the number of tokens it has to move to the main_q
		// and store the result in sh_nsurvival_tokens_in_CTA
		__shared__ int32 sh_nsurvival_tokens_in_CTA;

		// We need to move the survival tokens to the main_q
		// 
		// sh_main_q_global_block_offset has two purposes :
		// (1) to know where to store the survival tokens in the main_q
		// (2) to perform the prefix sum degrees of the survival degrees
		//
		// The reason why we store those two values together is because they are linked (see below)
		//
		// (1) We need a spot to store those tokens in the main_q 
		// We will ask the main_q counter where to store those tokens, the answer will be 
		// an offset of the main_q. We will store our tokens in positions :
		// d_main_q_state[sh_main_q_global_block_offset.ntokens], d_main_q_state[sh_main_q_global_block_offset.ntokens+1]...
		//
		// (2) sh_main_q_global_block_offset.narcs contains the number of arcs in the main_q up until index sh_main_q_global_block_offset.ntokens
		// ie the number of arcs going out of all states in d_main_q_state[0..sh_main_q_global_block_offset.ntokens]
		// it is used to compute the global prefix sum of degrees in one pass
		//
		__shared__ int2 sh_main_q_global_block_offset;

		const int nlanes = params.nchannels_to_compute;
		KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
			// The condition of the for loop is the same for all threads in the CUDA block
			// we want to keep all threads alive at the same time for now
			// otherwise __syncthreads() would fail
			const LaneCounters *lane_counters = params.d_lane_counters.channel(ilane);
			const post_expand_aux_q_end = lane_counters->post_expand_aux_q_end;
			KALDI_CUDA_DECODER_1D_BLOCK_OFFSET_KERNEL_LOOP(block_offset, thread_idx, post_expand_aux_q_end) {
				const int32 aux_q_idx = block_offset + thread_idx;
				const ChannelId ichannel = kernel_params.channel_to_compute[ilane];
				int32 degree = 0;
				int32 arc_start = -1;
				StateId token_state;
				IntegerCostType token_int_cost;
				// if aux_q_idx is a valid index in the main_q
				if(aux_q_idx < aux_q_end) {
					// Cost and state associated with the token
					const int2 both = params.d_aux_q_state_and_cost.lane(ilane)[aux_q_idx]
					token_state = both.x;
					token_int_cost = both.y;

					// Best cost for that token_state
					// We know we have a token associated with token_state in the queue with the cost state_best_cost
					const IntegerCostType state_best_int_cost = params.d_state_best_cost.lane(ilane)[token_state];
					// Final cutoff from last ExpandArc execution
					const IntegerCostType int_cutoff = lane_counters->int_cutoff;
					// Cutoff may have decreased since the creation of the token
					if(token_int_cost < int_cutoff) {
						// We can have duplicates, ie token associated with the same states
						// If this token is not the best candidate, get rid of it
						if(token_int_cost == state_best_int_cost) {
							arc_start = params.d_arc_offsets[token_state];
							const int32 arc_end = params.d_arc_offsets[token_state+1];
							degree = arc_end - arc_start;
						}
					}
					// the d_state_best_cost lookup table is reset to +INF for all states between frame
					// for perf. reason we only reset states that are in d_main_q_state
					// however if state_best_cost >= cutoff, all tokens associated with token_state 
					// will be pruned, and that state will not be in d_main_q_state
					// we need to reset the lookup table now
					if (state_best_int_cost >= int_cutoff)
						params.d_state_best_int_cost.lane(ilane)[token_state] = INT_MAX; 
				}

				int32 is_pruned = (arc_start == -1);
				// We now know which tokens will be moved to the main_q, the remaining will be pruned
				// we now compute a prefix sum inside the CUDA block to determine the local indexes of the survival tokens
				// the first survival token will have a index of 0, the second 1, ...
				// We also need to compute the prefix sum of the degrees
				// we start by doing a local prefix sum inside the CUDA block
				int2 block_prefix_sum_narc_and_end = {(is_pruned ? 0 : 1), degree};
				const int2 zero2 = {0,0};

				// Computing the prefix sum (exclusive)
				BlockScan(sh_temp_storage).ExclusiveScan(block_prefix_sum_token_arc_count, 
						block_prefix_sum_token_arc_count, 
						zero2,
						SumSum());

				int2 token_and_arc_count_block_sum;
				if(IS_LAST_1D_THREAD()) {
					// This conditional branch is entered by the last thread
					// because it is the last, the prefix_sum of that thread contains the sum of all elts

					// We also add the value from this thread - the prefix sum is exclusive
					token_and_arc_count_block_sum.split.ntokens = block_prefix_sum_token_arc_count.ntokens + (is_pruned ? 0 : 1);
					token_and_arc_count_block_sum.split.narcs = block_prefix_sum_token_arc_count.narcs + degree;

					sh_nsurvival_tokens_in_CTA = token_and_arc_count_block_sum.split.ntokens;

					// Doing two things at the same time :
					// requesting a spot in the main_q to store the survival tokens from this CTA 
					// (we need space for token_and_arc_count_block_sum.split.ntokens tokens)
					// informing the main_q that our survival tokens contain token_arc_count_block_sum.split.narcs arcs
					//
					// We then store the return value, which is the global offset on where to store those tokens,
					// and the total number of arcs up until that global offset
					sh_main_q_global_block_offset.both = atomicAdd(&lane_counters->main_q_end_and_narcs_i2.both, token_and_arc_count_block_sum.both);
				}

				// Syncing for three reasons :
				// - Broadcasting sh_main_q_global_block_offset
				// - Broadcasting sh_nsurvival_tokens_in_CTA
				// - We may reuse sh_temp_storage (cf CUB doc)
				__syncthreads(); 

				// Checking if we are overflowing the main_q
				if((sh_main_q_global_block_offset.split.ntokens + sh_nsurvival_tokens_in_CTA) >= kernel_params.q_capacity) {
					// TODO move above
					if(threadIdx.x == (KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX-1)) {
						// We are overflowing the main_q
						// We first revert what this CTA has done, ie revert the previous atomicAdd
						// because all CTAs will revert, we know we will have a valid state after completion of this kernel
						atomicAdd(&lane_counters->main_q_end_and_narcs_i2.both, -token_and_arc_count_block_sum.both); // revert
						// Setting the flag. It will print a warning to stderr
						lane_counters->q_overflow = 1;
					}

					// We abort computation, we no longer have space in the main_q.
					// We still jump to finalize_kernel, to do what's needed before completion
					goto finalize_kernel;
				}

				// If we are executing the following lines it means that we are not overflowing the queue
				// We then continue what we were doing
				// Note : we could remove the branch divergence here 
				if(!is_pruned) {
					// This thread is in charge of a survival token
					// we will move it to the main_q, at index main_q_idx
					const int32 main_q_idx = sh_main_q_global_block_offset.split.ntokens + block_prefix_sum_token_arc_count.ntokens;
					// Moving the token to the main q
					params.d_main_q_state_and_cost.channel(ichannel)[main_q_idx] = {token_state, token_int_cost};
					params.d_main_q_info.lane(ilane)[main_q_idx] = params.d_aux_q_info.lane(ilane)[aux_q_idx];
					// Saving the global prefix sum
					// = (narcs until now in the main queue) + (narcs until this thread in the CTA)
					params.d_main_q_degrees_prefix_sum.channel(ichannel)[main_q_idx] = sh_main_q_global_block_offset.split.narcs 
						+ block_prefix_sum_token_arc_count.narcs;
					// Saving the CSR arc offset for that token's state
					// it will be used by the expand kernel, and avoid doing a new random memory access in the expand kernel
					params.d_main_q_arc_offsets.channel(ichannel)[main_q_idx] = arc_start;
				}
			}

		}
finalize_kernel:
		// Nothing to do
	}


	/*
	   PreprocessInPlace
	   This kernel is also a preprocessing kernel, but this time does it in place
	   ie it will not move tokens from the aux_q to the main_q
	   It will do the preprocess operation directly on the main_q
	   The tokens are already in the main q (they were placed here by a previous "contract and preprocess").

	   We cannot prune non-optimal tokens, because the tokens are already in the main_q (we cannot prune 
	   the main_q - it would break the prev_token indexes). To avoid doing unnecessary computation 
	   in the expand kernel, we simulate the pruning by setting non-optimal token's degree to 0
	   We then rely on the 1 thread = 1 arc exact load balacing of expand to ignore that token

	   Please note that even if 0 threads will perform work on an ignored token in expand (degree = 0),
	   it is not exactly the same as pruning it : the main_q accesses will not be perfectly coalesced
	   in expand, because some "dead" tokens exist between living ones

	   For the preprocess stage we have to compute the prefix sum of the tokens arc degrees
	   Here we have to do the prefix sum in two passes : first local prefix sums inside CUDA block,
	   then in a second kernel (finalize_preprocess_in_place), we add the necessary block offsets to end up 
	   with the global prefix sum

	   This preprocess step is used in ProcessEmitting. Tokens were placed in main_q by
	   the ProcessNonEmitting of the previous frame. We cannot renumber them (it would break
	   the prev_token index). We preprocess in place, leaving things as they are in main_q

	 */

	__global__ void _preprocess_in_place_kernel(KernelParams params) {
		// Operator for the prefix sum inside the CUDA block
		typedef hipcub::BlockScan<int32, KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX> BlockScan;
		__shared__ typename BlockScan::TempStorage sh_temp_storage;

		// All threads in the last CUDA block (CTA) alive will have work to do at the end
		// this bool will be needed to broadcast the information from thread0 to all threads in the last CTA 
		__shared__ bool sh_is_last_CTA;

		const int nlanes = params.nchannels_to_compute;
		KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
			const LaneCounters *lane_counters = params.d_lane_counters.channel(ilane);
			const int32 main_q_end = lane_counters->main_q_end;

			// The condition of the for loop is the same for all threads in the CUDA block
			// we want to keep all threads alive at the same time for now
			// otherwise __syncthreads() would fail
			KALDI_CUDA_DECODER_1D_BLOCK_OFFSET_KERNEL_LOOP(block_offset, thread_idx, main_q_end) {
				// Position of considered token in the main_q
				const int32 main_q_idx = block_offset + thread_idx; 
				const ChannelId ichannel = params.channel_to_compute[ilane];

				// Total number of arcs from that token's state
				int32 degree = 0; 
				if(main_q_idx < main_q_end) {
					StateId token_state = params.d_main_q_state.channel(ichannel)[main_q_idx]; 
					CostType token_cost = params.d_main_q_cost.channel(ichannel)[main_q_idx];

					// Final cutoff from last ExpandArc execution
					// The cutoff can have decreased since moving tokens to the main_q
					// min_cost cannot be lower than before (we only did non-emitting phases since then)
					// but the adaptive beam may have lowered the beam
					const CostType cutoff = lane_counters->cutoff;

					if(token_cost < cutoff) {
						// Best cost for that token_state
						// We know we have a token associated with token_state in the queue with the cost state_best_cost
						const CostType state_best_cost = orderedIntToFloat(params.d_state_best_cost.lane(ilane)[token_state]); 

						// We can have duplicates, ie token associated with the same states
						// If this token is not the best candidate, get rid of it
						if(token_cost == state_best_cost) {
							const int32 start = params.d_arc_offsets[token_state]; 
							const int32 end = params.d_arc_offsets[token_state+1]; 
							degree  = end - start;

							// Saving the start offset for the expand kernel
							// avoid a new random memory access
							params.d_main_q_arc_offsets.channel(ichannel)[main_q_idx] = start;
						}
					}
				}

				int32 degree_local_prefix_sum;

				// Computing a local prefix sum inside that CUDA block
				// A second kernel will take care of adding the necessary offset to those local prefix sums
				BlockScan(sh_temp_storage).ExclusiveSum(degree, degree_local_prefix_sum);

				if(main_q_idx < main_q_end) {
					// This is not the final global prefix sum
					// A second kernel will add the necessary offset
					params.d_main_q_degrees_prefix_sum.channel(ichannel)[main_q_idx] = degree_local_prefix_sum; 
				}

				if(threadIdx.x == (KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX-1)) {
					// Saving the local sum of degrees of that CUDA block
					// That's necessary to compute the global offset of that CUDA block,
					// and that offset is what we need to transform the local prefix sum into a global prefix sum

					const int local_sum_index = block_offset/KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX;
					const int local_sum = degree_local_prefix_sum + degree; // the prefix sum was exclusive, adding missing value
					params.d_main_q_degrees_block_sums_prefix_sum.lane(ilane)[local_sum_index] = local_sum; 
				}


				// Synchronization because: 
				// - we may need to reuse sh_temp_storage if the for loop iterates (cf CUB's doc)
				// TODO replace with double buffering
				__syncthreads(); 
			}
		}
	}


	//
	// Helper functions/data structure for the ExpandArc kernel
	//

	// 
	// We'll use the same BlockScan to compute two things :
	//     1) The prefix sum of indexes
	//     1) The minimum cost overall all costs in the CUDA Block 
	//
	// We use a + for the prefix sum, and a min for the min
	//

	struct MinPlus {
		__device__ int2 operator()(const int2 &a, const int2 &b) const {
			int2 c;
			c.x = fmin(a.x, b.x);
			c.y = a.y + b.y;
			return c;
		}
	};

	//
	// GetAdaptiveBeam is used by ExpandArc and FinalizeProcessNonemitting
	//
	// Given the fact that the token queues are too small to store 
	// all possible tokens in the worst case scenario (where we could generate "nstates" tokens),
	// we need to tighten the beam if we notice that we are at risk of overflowing either the aux_q
	// or the main_q
	//

	__device__ __forceinline__ CostType GetAdaptiveBeam(const CostType default_beam,
			const int32 q_size,
			const int32 q_capacity) {

		// Doing something simple for now
		// We have to keep beam large enough,
		// the final cutoff will be used for the final
		// prune. If it is too small, we won't keep enough tokens

		CostType beam = default_beam;

		// TODO do something better 
		if(q_size >= q_capacity/2) 
			beam /= 2;

		return beam;
	}

	__device__ __forceinline__ int32 binsearch_maxle(const int32 *vec, const int32 val, int32 low, int32 high) {
		while(true) {
			if(low == high)
				return low; //we know it exists
			if((low + 1) == high)
				return (vec[high] <= val) ? high : low;

			int32 mid = low + (high- low) / 2;

			if(vec[mid] > val)
				high = mid-1;
			else
				low = mid;
		}
	}


	//
	// ExpandArc kernel
	// This kernel does the actual work of traversing arcs 
	//
	// Pseudo code :
	// for all token tok in main_q[main_q_offset...end]:
	//      u = tok.next_state
	//      for all arc a(u->v) in the FST:
	//          v_cost = tok.cost + a.cost + accoustic_cost
	// 
	//          if v_cost < cutoff and v_cost < best_state_cost[v]
	//              generate token associated to v, add to aux_q
	//              update best_state_cost[v]
	//              if necessary update cutoff
	//
	// For more information please refer to http://kaldi-asr.org/doc/decoders.html
	//
	// ExpandArc rely on some preprocessed data to be able to function 
	// for instance, it needs the prefix sum of the arc degree of all token.state in the
	// main_q
	// We need to call a Preprocess kernel before ExpandArc
	//
	// ExpandArc is used for both emitting and nonemitting phases
	// Differences between emitting and nonemitting :
	//      1) params.d_q_arc_offset contains offsets to either emitting or nonemitting arcs. 
	//         It is transparent for this kernel. The differentiation was done in the Preprocess kernel,
	//         which is responsible for filling the params.d_q_arc_offset array
	//      2) Computation of the acoustic cost. If nonemitting, it is equal to 0. If emitting, we need
	//         to use values from the acoustic model (through the d_loglikelihoods array)
	//
	//
	//
	// Note : ExpandArc is not the only kernel able to traverse arcs. 
	// FinalizeProcessNonemitting contains a simplified version of expand for only one CUDA block
	//
	void __global__ _expand_arcs_kernel(ExpandArcParams params) {
		// BlockScan that we will use to compute token indexes in the output queue, 
		// and to find the min cost in the block
		typedef hipcub::BlockScan<int2, KALDI_CUDA_DECODER_KERNEL_EXPAND_ARCS_DIMX> BlockScan;
		__shared__ typename BlockScan::TempStorage sh_temp_storage_scan;

		// This kernel writes the new token to the output queue aux_q
		// We will request a spot to store all the new tokens created by threads in this CUDA block
		// sh_aux_q_index_block_offset indicates where to store them in the aux_q
		// tokens created in this CUDA block will be store in :
		// aux_q[sh_aux_q_index_block_offset], aux_q[sh_aux_q_index_block_offset + 1], ...
		__shared__ int32 sh_aux_q_index_block_offset;

		const int nlanes = params.nchannels_to_compute;
		KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
			const LaneCounters *lane_counters = params.d_lane_counters.channel(ilane);
			const int32 main_q_offset = lane_counters->main_q_offset;
			const int32 main_q_end = lane_counters->main_q_end_and_narcs.end;
			const int32 total_narcs = lane_counters->main_q_end_and_narcs.narcs;
			KALDI_CUDA_DECODER_1D_BLOCK_OFFSET_KERNEL_LOOP(block_offset, thread_idx, total_narcs) {
				// Position of considered token in the main_q
				const ChannelId ichannel = params.channel_to_compute[ilane];

				//
				// Important : this thread is not responsible for a token in the input queue main_q
				// but for an arc, going out of a token in the main_q
				// The main_q contains in total total_narcs
				// and this thread will compute the main_q_arc_index-th arc of the main_q
				// For instance, first thread in the grid with threadIdx.x == 0 and blockIdx.x == 0 
				// will process the first arc of the token in main_q[main_q_offset + 0] 
				// (if that token has at least one arc)
				//
				// This insure a perfect one thread = one arc load balancing
				// but we have work to do to know exactly which arc is the main_q_arc_index-th arc
				// (what's its source ? its destination ? its arc_idx the FST CSR ?)
				//
				int32 main_q_arc_index = block_offset + thread_idx;

				// We'll need those variables later in the kernel
				// we declare them outside of the "valid_input" scope
				// to be able to access them later
				int32 main_q_idx;
				int32 arc_idx;
				StateId arc_next_state;
				IntegerCostType int_total_cost = INT_MAX;
				if(main_q_arc_index < total_narcs) {
					// Current thread must take care of main_q_arc_index-th arc
					// we need to now what's the source of that arc
					// ie which token.state in main_q does it start from ? 
					// We use a binary search in the prefix sum of the token's degree to get that information
					// 
					// Example : main_q contains 3 tokens
					// - First token is associated to a state which has 3 outgoing arc
					// - Second token is associated to a state which has 0 outgoing arc
					// - Third token is associated to a state which has 2 outgoing arc
					//
					// We store the degrees in an array :
					// [3, 0, 2]
					//
					// We then compute the exclusive prefix sum of that array :
					// [0, 3, 3, 5]
					//
					// In total, we have 5 arcs in the main_q. ExpandArc will use 5 threads.
					//
					// Let's say we are the fifth thread in ExpandArc. 
					// we have threadIdx.x == 4, and blockIdx.x == 0
					// it gives us main_q_arc_index == 4
					// From there we have no idea what we're supposed to do next, we need to have information about the
					// arc that we're supposed to traverse
					//
					// To do that, we look for the maximum index maxle_i in the prefix sum array such prefix_sum[i] <= 4
					//
					// [0, 3, 3, 5]
					//         /\
					//         here
					// maxle_i = 2
					// it means that our source token is at index 2 in the main_q
					// and we are computing the arc at index (main_q_arc_index - prefix_sum[maxle_i]) of that token 
					// ie the arc at index (4-3) = 1, the second arc of the second token in main_q

					// Searching for the source of the arc that we will process (main_q_arc_index)
					// we could preprocess the search in the preprocess kernels - for now this kernel is fast enough
					const int32 *degrees_prefix_sum = params.d_main_q_degrees_prefix_sum.channel(ichannel);
					main_q_idx = binsearch_maxle(degrees_prefix_sum, main_q_arc_index, main_q_offset, main_q_end-1); 

					// state_first_arc_idx_in_main_q
					// d_main_q_degrees_prefix_sum contains the prefix sum of the 
					// degrees of all tokens in the main_q
					// d_main_q_degrees_prefix_sum[main_q_idx] contains the number of arc
					// in the main_q until that token
					const int32 state_first_arc_idx_in_main_q = degrees_prefix_sum[main_q_idx];

					// arc_offset_start is the offset in the CSR, to find the arcs 
					// related to the state main_q_state_[main_q_idx]
					// it was set by the preprocess kernel
					const int32 arc_offset_start = params.d_q_arc_offsets.channel(ichannel)[main_q_idx];

					// local_arc_index is the arc index for that state
					// if local_arc_index == 2, we will process the second arc
					// of state main_q_state_[main_q_idx]
					const int32 local_arc_index = main_q_arc_index - state_first_arc_idx_in_main_q;

					// corresponding arc_idx in the FST
					arc_idx = arc_offset_start + local_arc_index; 

					// Destination of that arc
					arc_next_state = params.arc_nextstates[arc_idx];

					// Building the total cost incrementally 
					// we'll add the acoustic cost and the old token's cost
					const CostType arc_fixed_cost = params.arc_weights[arc_idx];

					const int32 arc_ilabel = params.is_emitting ? params.arc_ilabels[arc_idx] : 0;

					const CostType acoustic_cost = (arc_ilabel != 0) ? -params.d_loglikelihoods.channel(ichannel)[arc_ilabel] : 0.0; 
					const CostType prev_token_cost  = orderedIntToFloat(params.d_main_q_cost.channel(ichannel)[main_q_idx]);

					int_total_cost = floatToOrderedInt(prev_token_cost + arc_fixed_cost + acoustic_cost);

					// If the total_cost is too large compared to our cutoff (beam search)
					// then let's drop it
					const IntegerCostType int_cutoff = lane_counters->int_cutoff;
					if(int_total_cost >= int_cutoff)
						int_total_cost = INT_MAX;
					else {
						// We need to check if we already have a token going to that next_state,
						// and if that token has a lower cost that we have
						// params.d_state_best_cost[state] contains the best cost for that state in the current frame
						const IntegerCostType next_state_best_int_cost = params.d_state_best_cost.lane(ilane)[arc_next_state];

						// If that token is the best for that state, drop it
						if(int_total_cost >= next_state_best_int_cost)
							int_total_cost = INT_MAX;
					}
				}

				//
				// If int_total_cost < INT_MAX, it means that : 
				// - this thread had a valid input (main_q_arc_index < total_narcs)
				// - the total_cost of the generated token is < cutoff
				// - the generated token is the best candidate for that next_state
				// We will then add that new token in the output queue, aux_q
				// We need to know where to put that token in the aux_q
				// we'll first compute its index inside the CUDA block
				// the first valid output token in the CUDA block will have index 0, 
				// the second index 1... We compute that using a prefix sum
				//
				// We also need to find the overall min cost in the CUDA block
				// a prefix sum is a scan operation, and a min a reduce operation
				// we can perform a reduce operation using a scan (using the last value)
				// we compute the prefix sum and the min in one scan, using the data 
				// struct CostTypeAndInt
				//
				const int32 has_successor = (int_total_cost < INT_MAX) ? 1 : 0; 

				// Updating the best_state_cost lookup table with our new best cost
				if(has_successor)
					atomicMin(&params.d_state_best_cost.channel(ichannel)[arc_next_state],
						int_total_cost);

				int2 int_cost_and_index = {int_total_cost, has_successor};
				BlockScan(sh_temp_storage_scan).InclusiveScan(cost_and_index, cost_and_index, MinPlus());
				if(IS_LAST_1D_THREAD())
					// We can find a lower global_min_cost only in the emitting stage
					if(params.is_emitting) { // TODO templatize
						IntegerCostType global_int_min_cost = lane_counters->int_min_cost;
						IntegerCostType local_int_min_cost = int_cost_and_index.x;
						// if we found a lower min_cost, update the global value
						if(local_int_min_cost < global_int_min_cost) {
							atomicMin(&lane_counters->int_min_cost, global_int_min_cost);
							const CostType beam = orderedIntToFloat(lane_counters->int_beam);
							IntegerCostType new_int_cutoff = floatToOrderedInt(orderedIntToFloat(local_int_min_cost) + beam);
							atomicMin(&lane_counters->int_cutoff, new_int_cutoff);
						}
					}
					// We are in a divergent branch
					// This is the last thread. The last value of the inclusive scan is the total
					const int32 total_successors_in_block = cost_and_index.y;
					// Requesting a spot of size total_successors_in_block in the aux_q
					const int aux_q_index_block_offset = atomicAdd(&lane_counters->aux_q_end, total_successors_in_block);
					// All threads will need this value
					// Saving in shared memory
					sh_aux_q_index_block_offset = aux_q_index_block_offset;
					//
					// Here we detect an overflow of the aux_q
					// we detect it before actually using the aux_q
					// We try to prevent an overflow from happening using an adaptive beam (cf GetAdaptiveBeam)
					//
					if((sh_aux_q_index_block_offset + total_successors_in_block) >= params.q_capacity) {
						// sh_aux_q_index_block_offset is in shared memory
						// its value is currently invalid (overflow)
						// we set it to a special value and use it as a flag to broadcast
						// the fact that we have an overflow and that all threads should exit
						sh_aux_q_index_block_offset = params.q_capacity;
						// We revert the last operation. All threads that detected the overflow 
						// will revert what they've done. It means that at the end of the kernel,
						// we'll be back to the last valid state 
						// We'll be able to continue computation, but quality of the output
						// may be lower (we weren't able to save all tokens)
						atomicAdd(&lane_counters->d_aux_q_end, -total_successors_in_block); 
						// Setting the flag for the host. It will be used to print a warning to stderr
						lane_counters->q_overflow = 1; 
						// We do not jump to finalize_kernel now, because only threadIdx.x == 0 
						// is executing this
						// We wait until the end of the divergent branch
					} else {
						// If we are not overflowing the queue, let's check if we need to 
						// tighten the beam. If the occupancy of the aux_q gets too high,
						// the adaptive beam will reduce the beam
						CostType new_beam = GetAdaptiveBeam(params.default_beam, 
								aux_q_index_block_offset,
								params.q_capacity);
						if(new_beam < params.default_beam
							&& new_beam < orderedIntToFloat(lane_counters->int_beam)) 
							atomicMin(&lane_counters->int_beam, floatToOrderedInt(new_beam));
					}
				}

				// Sync'ing for two reasons :
				// - Broadcasting sh_aux_q_index_block_offset
				// - reusing sh_temp_storage (cf CUB's doc)
				__syncthreads(); 
				// The only case where we can have that condition met,
				// is if we detected an overflow if the previous lines
				// we need to finalize our work and quit 
				// Now all threads are executing this code. We can jump
				// to finalize_kernel
				if(sh_aux_q_index_block_offset == params.q_capacity) 
					goto finalize_kernel; // keeping things clean before aborting
				//
				// If we're executing the following lines it means everything
				// is valid and we are not overflowing the aux_q
				//
				cost_and_index.y -= has_successor; // we want the exclusive sum now
				const int32 aux_q_block_index = cost_and_index.y;
				const int32 aux_q_index = sh_aux_q_index_block_offset + aux_q_block_index;
				if(has_successor) {
					// We save the new token to the aux_q
					params.d_aux_q_state_int_cost[aux_q_index] = {arc_next_state, int_cost};
					// Index of the parent token
					// the parent is the token used as input 
					// that parent is at index main_q_idx in the GPU memory
					// However, the main_q is emptied before processing a new frame
					// we need to add the offset related to the previous frames index
					// we add params.main_q_global_offset
					const int32 prev_token = lane_counters->main_q_global_offset + main_q_idx;
					params.d_aux_q_prev_and_arc[aux_q_index] = {prev_token, arc_idx};
				}
			}
		}
		finalize_kernel:
	}

	// Called when channels will start decoding a new utterance
	// do everything that's needed to do on the device to start decoding a new utterance with those channels
	__global__ init_decoding_on_device_kernel_(KernelParams params) {
		const int init_channel_id = params.init_channel_id;
		const ChannelCounters *init_channel_counters = params.d_channels_counters.channel(init_channel_id);
		const int init_main_q_end = init_channel_counters->final_frame_main_q_end;
		const int nlanes = params.nlanes;
		KALDI_CUDA_2D_KERNEL_LOOP(idx, init_main_q_end, ilane, nlanes) { 
			ChannelId channel_id = kernel_params.channel_to_compute[ilane];
			params.d_main_q_state_and_cost.channel(channel_id)[idx] = params.d_main_q_state_and_cost.channel(init_channel_id)[idx];
			params.d_main_q_degrees_prefix_sum.channel(channel_id)[idx] = params.d_main_q_degrees_prefix_sum.channel(init_channel_id)[idx];
			params.d_main_q_arc_offset.channel(channel_id)[idx] = params.d_main_q_arc_offset.channel(init_channel_id)[idx];
			if(idx == 0) {
				ChannelCounters *channel_counters = params.d_channels_counters.channel(channel_id);
				channel_counters->final_frame_main_q_end  = main_q_end;
				channel_counters->final_frame_main_q_narcs = init_params.final_frame_main_q_narcs;
				channel_counters->global_min_cost_and_beam.min_cost = kernel_params.infinite_cost;
				channel_counters->global_min_cost_and_beam.beam = kernel_params.default_beam;
			}
		}
	}

	__global__ initialize_lanes_with_channels_(KernelParams kernel_params) {
		LaneParams &lane_params = kernel_params.d_lane_params[blockIdx.z];
		ChannelId channel_id = kernel_params.channel_to_compute[blockIdx.z];
		ChannelParams &channel_params = kernel_params.d_channel_params[channel_id];

		// Getting the lane ready for that channel
		// TODO save beam and everything
		lane_params.main_q_end = channel_params.final_frame_main_q_end;
		lane_params.main_q_narcs = channel_params.final_frame_main_q_narcs;
	}

	__global__ void _finalize_frame_computation(KernelParams params) {
		const int nlanes = params.nchannels_to_compute;
		KALDI_CUDA_DECODER_BATCH_KERNEL_LOOP(ilane, nlanes) {
			const ChannelId ichannel = kernel_params.channel_to_compute[ilane];
			const int32 main_q_end = params.d_lane_counters.lane(ilane)->main_q_end;
			const ChannelCounters *channel_counters = params.d_channels_counters.channel(ichannel);
			const LaneCounters *lane_counters = params.d_lane_counters.channel(ilane);

			KALDI_CUDA_DECODER_1D_KERNEL_LOOP(main_q_idx, main_q_end) {
				// One thread takes care of the counters
				if(main_q_idx == 0) {
					// Reset the min_cost for next frame computation
					channel_counters->global_min_cost_and_beam.min_cost = floatToOrderedInt(params.infinite_cost); 
					// Resetting the beam back to default between frames
					const CostType previous_beam = lane_counters->global_min_cost_and_beam.beam;
					const CostType beam = fmin(params.default_beam, previous_beam * KALDI_CUDA_DECODER_ADAPTIVE_BEAM_RECOVER_RATE);
					channel_counters->global_min_cost_and_beam.beam = floatToOrderedInt(beam); 
					const int32 main_q_narcs = lane_counters->main_q_narcs;
					// Saving main_q_{end,narcs} - the current lane will be used for another channel
					channel_params.final_frame_main_q_end = main_q_end;
					channel_params.final_frame_main_q_narcs = main_q_narcs;
				}
				StateId state = params.d_main_q_state.channel(ichannel)[main_q_idx];

				int32 local_sum_idx = main_q_idx / KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX;
				int32 local_sum_offset = params.d_local_sums_prefix_sum.lane(ilane)[local_sum_idx];
				params.d_main_q_degrees_prefix_sum.channel(ichannel)[main_q_idx] += local_sum_offset;

				// d_main_q_state_ contains the list of states that we've considered in the last frame
				// it corresponds to the list of indexes i such as d_state_best_cost[i] < +INF
				// we just reset those states between frames
				params.d_state_best_cost.lane(ilane)[state] = floatToOrderedInt(infinite_cost);
			}
		}
	}

	__global__ post_expand_emitting_(KernelParams kernel_params) {
		LaneParams &lane_params = kernel_params.d_lane_params[blockIdx.z];
		ChannelId channel_id = kernel_params.channel_to_compute[blockIdx.z];
		ChannelParams &channel_params = kernel_params.d_channel_params[channel_id];
		// main_q_end contains the tokens from the previous frame
		// after emitting, we won't use them anymore to create new tokens
		// we reset the main_q, making space for tokens from this current frame
		lane_params.main_q_end = 0;
		lane_params.main_q_narcs = 0;
		lane_params.pre_expand_main_q_end = 0;
		channel_params.main_q_global_offset += channel_params.final_frame_main_q_end;
	}

	__global__ post_expand_non_emitting_(KernelParams kernel_params) {
		LaneParams &lane_params = kernel_params.d_lane_params[blockIdx.z];
		ChannelId channel_id = kernel_params.channel_to_compute[blockIdx.z];
		ChannelParams &channel_params = kernel_params.d_channel_params[channel_id];
		// Resetting narcs, we are done processing those arcs
		lane_params.main_q_narcs = 0;
		// Done processing tokens [offset, end[. Moving the offset
		lane_params.main_q_local_offset += lane_params.pre_expand_main_q_end;
		lane_params.pre_expand_main_q_end = lane_params.main_q_end;
		lane_params.post_expand_aux_q_end = aux_q_end;
		aux_q_end = 0;
	}
	/*

	   FinalizeProcessNonemitting
	   Meta-kernel (merging preprocess and expand) but only works with 1 CUDA block

	   Used to avoid calling multiple "heavy lifting" kernels for the tail of non emitting
	   (lots of iterations with small number of arcs)

	   Code is greatly simplified because we can have only one CTA alive

	   Repeat until new queue empty:
	   1) Preprocess 
	   2) Expand arcs

	   The preprocess stage is not done on the first iteration, because it was
	   already done by the ProcessAndContract kernel. We always call ProcessAndContract
	   before calling FinalizeProcessNonemitting 

	   At the end, this kernel finalize the computation for current frame,
	   so that it's ready for next ProcessEmitting

	   TODO This kernel could be easily optimized  

Note : For a detailed description on how the Preprocess and Expand operation work,
please refer to the PreprocessInPlace and ExpandArc kernel implemention. The algorithm are 
described there. In this kernel, we compute simplified version of preprocess and expand, because
we do not need inter-block communication (we launch only one CUDA block)

Important : in ExpandArc, the input is the main_q, the ouput is the aux_q. We then call PreprocessAndContract
that move the tokens from the aux_q to the main_q.
Here we directly output the tokens in the main_q. It helps use simplify the code, and we are not generating a lot
of tokens anyway (so the pruning stage of PreprocessAndContract is less critical)

	 */


	__launch_bounds__(KALDI_CUDA_DECODER_KERNEL_NONEM_LT_DIMX, 1)
		__global__ void _finalize_process_non_emitting(const uint32_t *d_arc_offsets, 
				ExpandArcParams params) {

			// Used to compute the index in the output queue
			typedef hipcub::BlockScan<TokenAndArcCount, KALDI_CUDA_DECODER_KERNEL_NONEM_LT_DIMX> BlockScanTokenAndArcCount;
			__shared__ typename BlockScanTokenAndArcCount::TempStorage sh_temp_storage_scan_token_arc;

			typedef hipcub::BlockScan<int, KALDI_CUDA_DECODER_KERNEL_NONEM_LT_DIMX> BlockScanInt;
			__shared__ typename BlockScanInt::TempStorage sh_temp_storage_scan_int;

			int32 total_narcs = *params.d_main_q_narcs;

			int32 main_q_offset = *params.d_main_q_local_offset;
			int32 main_q_end = *params.d_main_q_end;

			// aux_q is empty when this kernel is called
			int32 aux_q_end = 0;

			MinCostAndBeamIntegers global_min_cost_and_beam = *params.d_global_min_cost_and_beam;
			CostType global_min_cost = orderedIntToFloat(global_min_cost_and_beam.min_cost);
			CostType beam = orderedIntToFloat(global_min_cost_and_beam.beam);

			while(total_narcs > 0) {

				// Step 1 : ExpandArcs

				for(int32 main_q_arc_index_block_offset = 0;
						main_q_arc_index_block_offset < total_narcs;
						main_q_arc_index_block_offset += blockDim.x) {

					int32 main_q_arc_index = main_q_arc_index_block_offset + threadIdx.x;

					// For details on how this code works, please refer to ExpandArc's comments
					CostType total_cost = FLT_MAX;
					int32 arc_idx;
					StateId arc_next_state;
					int32 main_q_idx;

					if(main_q_arc_index < total_narcs) {
						main_q_idx = binsearch_maxle(params.d_main_q_degrees_prefix_sum, 
								main_q_arc_index, 
								main_q_offset,
								main_q_end-1); 

						int32 state_first_arc_idx_in_main_q = params.d_main_q_degrees_prefix_sum[main_q_idx];
						int32 arc_offset_start = params.d_q_arc_offsets[main_q_idx];

						arc_idx = arc_offset_start + (main_q_arc_index - state_first_arc_idx_in_main_q);

						arc_next_state = params.arc_nextstates[arc_idx];
						CostType arc_weight = params.arc_weights[arc_idx];
						CostType next_state_cost = orderedIntToFloat(params.d_state_best_cost[arc_next_state]);
						CostType old_tok_cost = params.d_main_q_cost[main_q_idx];

						total_cost = arc_weight + old_tok_cost;

						CostType cutoff = global_min_cost + beam;
						if(total_cost >= cutoff || total_cost >= next_state_cost) {
							total_cost = FLT_MAX;
						} 
					}

					int32 has_successor = (total_cost < FLT_MAX) ? 1 : 0;

					if(has_successor) {
						//TODO _block
						atomicMin(&params.d_state_best_cost[arc_next_state], floatToOrderedInt(total_cost)); 
					}

					int32 local_aux_q_idx;
					int32 total_ntokens_to_aux_q;
					BlockScanInt(sh_temp_storage_scan_int).ExclusiveSum(has_successor, 
							local_aux_q_idx,
							total_ntokens_to_aux_q);

					// Checking if we are not overflowing the aux_q
					if((aux_q_end + total_ntokens_to_aux_q) >= params.q_capacity) {
						*params.h_q_overflow = 1;

						goto finalize_kernel;
					}


					if(has_successor) {
						int32 aux_q_idx = aux_q_end + local_aux_q_idx;
						params.d_aux_q_state[aux_q_idx] = arc_next_state;
						params.d_aux_q_cost[aux_q_idx] = total_cost;

						InfoToken new_tok_info;
						new_tok_info.prev_token = params.main_q_global_offset + main_q_idx;

						new_tok_info.arc_idx = arc_idx;
						params.d_aux_q_info[aux_q_idx] = new_tok_info;
					}

					aux_q_end += total_ntokens_to_aux_q;

					// Getting new beam using aux_q_end
					beam = GetAdaptiveBeam(params.default_beam, 
							aux_q_end,
							params.q_capacity);


					// reusing sh_temp_storage_scan_int
					__syncthreads();
				}

				// Step 2 : PreprocessAndContract
				// Sync : reusing some data pointers, like d_main_q_prefix_sum

				// Reset for new iteration
				total_narcs = 0;
				main_q_offset = main_q_end;

				for(int32 block_off = 0;
						block_off < aux_q_end;
						block_off += blockDim.x) {

					int32 aux_q_idx = block_off + threadIdx.x;

					int32 degree = 0;
					int32 start = -1;

					StateId token_state;
					CostType token_cost;

					if(aux_q_idx < aux_q_end) {
						token_state = params.d_aux_q_state[aux_q_idx];
						token_cost = params.d_aux_q_cost[aux_q_idx];

						// beam may have changed since generation
						CostType cutoff = global_min_cost + beam;
						if(token_cost < cutoff) {
							CostType best_cost = orderedIntToFloat(params.d_state_best_cost[token_state]);

							if(token_cost == best_cost) {
								start = d_arc_offsets[token_state];
								int32 end = d_arc_offsets[token_state+1];
								degree = end - start;
							}
						}
					}

					bool has_valid_nonpruned_token = (start != -1);

					TokenAndArcCount token_and_arc_count;
					token_and_arc_count.ntokens = has_valid_nonpruned_token ? 1 : 0;
					token_and_arc_count.narcs   = degree;
					TokenAndArcCount scan_aggregate;

					TokenAndArcCount zero_struct;
					zero_struct.ntokens = zero_struct.narcs = 0;

					BlockScanTokenAndArcCount(sh_temp_storage_scan_token_arc).ExclusiveScan(token_and_arc_count, 
							token_and_arc_count,
							zero_struct,
							TokenAndArcCountSum(),
							scan_aggregate);

					// Checking if we are not overflowing the main_q
					int32 total_ntokens_to_main_q = scan_aggregate.ntokens;
					if((main_q_end + total_ntokens_to_main_q) >= params.q_capacity) {
						*params.h_q_overflow = 1;

						goto finalize_kernel;
					}

					int32 degree_this_iteration_prefix_sum = token_and_arc_count.narcs;
					int32 degree_sum_for_this_iteration = scan_aggregate.narcs;

					int32 degree_prefix_sum = total_narcs + degree_this_iteration_prefix_sum;
					total_narcs += degree_sum_for_this_iteration;

					if(has_valid_nonpruned_token) {
						int32 local_main_q_idx = token_and_arc_count.ntokens;
						int32 main_q_idx = main_q_end + local_main_q_idx;

						params.d_q_arc_offsets[main_q_idx] = start;
						params.d_main_q_degrees_prefix_sum[main_q_idx] = degree_prefix_sum;
						params.d_main_q_state[main_q_idx] = token_state;
						params.d_main_q_cost[main_q_idx] = token_cost;

						InfoToken info_token = params.d_aux_q_info[aux_q_idx];
						params.d_main_q_info[main_q_idx] = info_token;
					}

					main_q_end += total_ntokens_to_main_q; 

					__syncthreads(); // reusing sh_temp_storage_scan
				}

				aux_q_end = 0; // aux_q is now considered empty
			}

finalize_kernel:

			if(threadIdx.x == 0) {
				// Next step is ProcessEmitting of next frame, from is currToken_offset
				*params.d_main_q_end = main_q_end; 
				*params.d_main_q_local_offset = 0; 

				// TODO update global_offset
				// No need to update the cutoff - maybe the beam
				//*params.d_cutoff = floatToOrderedInt(sh_cutoff);
			}

		}

	void CudaDecoder::FinalizeProcessNonemitting() {
		dim3 grid,block;
		block.x = KALDI_CUDA_DECODER_KERNEL_NONEM_LT_DIMX;
		grid.x = 1; // this kernel is designed for one CTA 

		expand_params_.main_q_global_offset = main_q_global_offset_;
		expand_params_.is_emitting = false;

	}


} // end namespace kaldi
