#include "hip/hip_runtime.h"
// decoder/cuda-decoder-kernels.cu

// See ../../COPYING for clarification regarding multiple authors
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//  http://www.apache.org/licenses/LICENSE-2.0
//
// THIS CODE IS PROVIDED *AS IS* BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, EITHER EXPRESS OR IMPLIED, INCLUDING WITHOUT LIMITATION ANY IMPLIED
// WARRANTIES OR CONDITIONS OF TITLE, FITNESS FOR A PARTICULAR PURPOSE,
// MERCHANTABLITY OR NON-INFRINGEMENT.
// See the Apache 2 License for the specific language governing permissions and
// limitations under the License.

#include <hipcub/hipcub.hpp>
#include "decoder/cuda-decoder.h"

#define DIV_ROUND_UP(a,b) ((a+b-1)/b)

namespace kaldi {

typedef CudaDecoder::StateId StateId;
typedef CudaDecoder::TokenAndArcCount TokenAndArcCount;
typedef CudaDecoder::TokenAndArcCountUnion TokenAndArcCountUnion;
typedef CudaDecoder::CostType CostType;
typedef CudaDecoder::PreprocessParams PreprocessParams; 
typedef CudaDecoder::ExpandArcParams ExpandArcParams; 

//
// Utils device function
//


    //
    // 1:1 Conversion float <---> sortable int
    // We convert floats to sortable ints in order
    // to use native atomics operation, which are 
    // way faster than looping over atomicCAS 
    //

    __device__ int32 floatToOrderedInt(float floatVal) {

        int32 intVal = __float_as_int( floatVal );

        return (intVal >= 0 ) ? intVal : intVal ^ 0x7FFFFFFF;
    }



    __device__ float orderedIntToFloat(int32 intVal) {

        return __int_as_float( (intVal >= 0) ? intVal : intVal ^ 0x7FFFFFFF );

    } 

    // Temporary used for cutoff - will be TODO removed
    __device__ float fatomicMin(float *addr, float value)

    {

        float old = *addr, assumed;
        if(old <= value) return old;

        do
        {
            assumed = old;
            old = atomicCAS((uint32_t*)addr,
                    __float_as_int(assumed),
                    __float_as_int(value));

        } while(old!=assumed); // TODO <

        return old;

    }

    //
    // Kernels
    //

    // For description of what each kernel is doing, please refer to cuda-decoder.h
    // and look for the corresponding wrapper
    // for instance, for a description of _init_lookup_kernel,
    // look for the description of CudaDecoder::InitStateCostLookup() in cuda-decoder.h

    // Used before first frame
    __global__ void _init_state_cost_lookup_kernel(int32 size, int32 *state_cost) {
        for(int32 idx = blockIdx.x*blockDim.x + threadIdx.x;
                idx < size;
                idx += blockDim.x*gridDim.x) {
            state_cost[idx]  = floatToOrderedInt(FLT_MAX);
        }
    }

    void CudaDecoder::InitStateCostLookup() {
        int32 nstates = fst_.numStates;
        KALDI_ASSERT(nstates > 0);

        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_KERNEL_INIT_LOOKUP_DIMX;
        grid.x = DIV_ROUND_UP(nstates, block.x);

        _init_state_cost_lookup_kernel<<<grid,block>>>(nstates, d_state_best_cost_);
    }

    // Used to reset lookup table between frames
    // Using the queue to reset only the values needed
    // Also takes care of resetting cutoff
    __global__ void _reset_state_cost_lookup_kernel(const StateId *d_main_q_state_, const int32 *d_main_q_end_, int32 *d_state_best_cost, CostType *d_cutoff) {
        int32 main_q_end = *d_main_q_end_; 

        for(int32 idx = blockIdx.x*blockDim.x + threadIdx.x;
                idx < main_q_end;
                idx += blockDim.x*gridDim.x) {
            // d_main_q_state_ contains the list of states that we've considered in the last frame
            // it corresponds to the list of indexes i such as d_state_best_cost[i] < +INF
            // faster than init_state_cost_lookup_kernel by a factor of ~10
            StateId state = d_main_q_state_[idx];
            d_state_best_cost[state]  = floatToOrderedInt(FLT_MAX);
        }

        if(blockIdx.x == 0 && threadIdx.x == 0)
            *d_cutoff = FLT_MAX; // we also reset the cutoff
    }

    void CudaDecoder::ResetStateCostLookup() {
        int32 size = *h_main_q_end_;

        KALDI_ASSERT(size > 0);

        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_KERNEL_INIT_LOOKUP_DIMX;
        grid.x = DIV_ROUND_UP(size, block.x);

        _reset_state_cost_lookup_kernel<<<grid,block,0,compute_st_>>>(d_main_q_state_, d_main_q_end_, d_state_best_cost_, d_cutoff);
    }


    // Sum operator for the TokenAndArcCount struct (2 ints) 
    // Used in preprocess_and_contract
    struct TokenAndArcCountSum {
        __device__ TokenAndArcCount operator()(const TokenAndArcCount &a, const TokenAndArcCount &b) const {
            TokenAndArcCount c;
            c.ntokens = a.ntokens + b.ntokens;
            c.narcs = a.narcs + b.narcs;

            return c;
        }
    };

    /*
       This kernel preprocess the necessary information for expand (scan of the outgoing degrees) 
       and explicitly prune the tokens

       The ExpandArc kernel writes the new raw token list in the aux_q. However, the cutoff 
       was progressively lowered during the computation, and some tokens now have a cost > cutoff.
       During the contract stage of this kernel, we remove such tokens. 
       We also remove duplicates, i.e. tokens pointing to the same state, but with token.cost > best_cost_for_that_state

       It contracts (by pruning) the queue list:
       raw output in aux_q ----contract----> pruned output in main q

       This kernel is responsible for :

       1) Read a token from the aux queue (raw output from previous expand)

       2) Compute the outgoing degree of that token.next_state. For that :
       -> If that token is suboptimal (cutoff, best_cost), we prune it
       -> Otherwise, we will move it to the main_q. We also read its arc degree in the FST graph 

       3) We move the non-pruned tokens into the main q. After a local prefix sum,
       we request a spot in the main_q for those tokens using the main_q_end_and_narcs counter. 
       main_q_end_and_narcs.split.end contains the number of tokens in the main q until now
       main_q_end_and_narcs.split.narcs contains the number of arcs in the main q until now

       We also compute the degrees prefix sum in one pass using the main_q_end_and_narcs.split.narcs

       This kernel is used before ProcessNonEmitting
    */

    // Important : pass the struct PreprocessParams by copy - passing it using a ref will not work (CPU -> GPU)
    __global__ void _preprocess_and_contract_kernel(PreprocessParams params) {
        
        // Prefix sum operator
        typedef hipcub::BlockScan<TokenAndArcCount, KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX> BlockScan;
        __shared__ typename BlockScan::TempStorage temp_storage;

        // This CUDA block (CTA) will count the number of tokens it has to move to the main_q
        // and store the result in nsurvival_tokens_in_CTA
        __shared__ int32 nsurvival_tokens_in_CTA;

        // We need to move the survival tokens to the main_q
        // 
        // main_q_global_block_offset has two purposes :
        // (1) to know where to store the survival tokens in the main_q
        // (2) to perform the prefix sum degrees of the survival degrees
        //
        // The reason why we store those two values together is because they are linked (see below)
        //
        // (1) We need a spot to store those tokens in the main_q 
        // We will ask the main_q counter where to store those tokens, the answer will be 
        // an offset of the main_q. We will store our tokens in positions :
        // d_main_q_state[main_q_global_block_offset.ntokens], d_main_q_state[main_q_global_block_offset.ntokens+1]...
        //
        // (2) main_q_global_block_offset.narcs contains the number of arcs in the main_q up until index main_q_global_block_offset.ntokens
        // ie the number of arcs going out of all states in d_main_q_state[0..main_q_global_block_offset.ntokens]
        // it is used to compute the global prefix sum of degrees in one pass
        //
        __shared__ TokenAndArcCountUnion main_q_global_block_offset;

        // Final cutoff from last ExpandArc execution
        const BaseFloat cutoff = *params.d_cutoff;

        const int32 aux_q_end = *params.d_aux_q_end;

        // The condition of the for loop is the same for all threads in the CUDA block
        // we want to keep all threads alive at the same time for now
        // otherwise __syncthreads() would fail
        for(int32 block_offset = blockDim.x*blockIdx.x;
                block_offset < aux_q_end;
                block_offset += gridDim.x*blockDim.x) {

            int32 aux_q_idx = block_offset + threadIdx.x;
            int32 degree = 0;
            int32 arc_start = -1;

            StateId token_state;
            CostType token_cost;

            // if aux_q_idx is a valid index in the main_q
            if(aux_q_idx < aux_q_end) {
                // Cost and state associated with the token
                token_cost = params.d_aux_q_cost[aux_q_idx];
                token_state = params.d_aux_q_state[aux_q_idx];

                // Best cost for that token_state
                // We know we have a token associated with token_state in the queue with the cost state_best_cost
                BaseFloat state_best_cost = orderedIntToFloat(params.d_state_best_cost[token_state]);

                // Cutoff may have decreased since the creation of the token
                if(token_cost < cutoff) {
                    
                    // We can have duplicates, ie token associated with the same states
                    // If this token is not the best candidate, get rid of it
                    if(token_cost == state_best_cost) {
                        arc_start = params.d_arc_offsets[token_state];
                        int32 arc_end = params.d_arc_offsets[token_state+1];
                        degree = arc_end - arc_start;
                    }
                }

                // the d_state_best_cost lookup table is reset to +INF for all states between frame
                // for perf. reason we only reset states that are in d_main_q_state
                // however if state_best_cost >= cutoff, all tokens associated with token_state 
                // will be pruned, and that state will not be in d_main_q_state
                // we need to reset the lookup table now

                if (state_best_cost >= cutoff)
                    params.d_state_best_cost[token_state] = floatToOrderedInt(FLT_MAX);

            }

            int32 is_pruned = (arc_start == -1);


            TokenAndArcCount block_prefix_sum_token_arc_count;

            // We now know which tokens will be moved to the main_q, the remaining will be pruned
            // we now compute a prefix sum inside the CUDA block to determine the local indexes of the survival tokens
            // the first survival token will have a index of 0, the second 1, ...
            block_prefix_sum_token_arc_count.ntokens =  is_pruned ? 0 : 1;
            
            // We also need to compute the prefix sum of the degrees
            // we start by doing a local prefix sum inside the CUDA block
            block_prefix_sum_token_arc_count.narcs =  degree;

            TokenAndArcCount zero_struct;
            zero_struct.ntokens = zero_struct.narcs = 0;

            // Computing the prefix sum (exclusive)
            BlockScan(temp_storage).ExclusiveScan(block_prefix_sum_token_arc_count, 
                                                    block_prefix_sum_token_arc_count, 
                                                    zero_struct,
                                                    TokenAndArcCountSum());

            
            TokenAndArcCountUnion token_and_arc_count_block_sum;
            if(threadIdx.x == (KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX-1)) {
                // This conditional branch is entered by the last thread
                // because it is the last, the prefix_sum of that thread contains the sum of all elts

                // We also add the value from this thread - the prefix sum is exclusive
                token_and_arc_count_block_sum.split.ntokens = block_prefix_sum_token_arc_count.ntokens + (is_pruned ? 0 : 1);
                token_and_arc_count_block_sum.split.narcs = block_prefix_sum_token_arc_count.narcs + degree;

                nsurvival_tokens_in_CTA = token_and_arc_count_block_sum.split.ntokens;
                
                // Doing two things at the same time :
                // requesting a spot in the main_q to store the survival tokens from this CTA 
                // (we need space for token_and_arc_count_block_sum.split.ntokens tokens)
                // informing the main_q that our survival tokens contain token_arc_count_block_sum.split.narcs arcs
                //
                // We then store the return value, which is the global offset on where to store those tokens,
                // and the total number of arcs up until that global offset
                main_q_global_block_offset.both = atomicAdd(&params.d_main_q_end_and_narcs_i2->both, token_and_arc_count_block_sum.both);
            }

            // Syncing for three reasons :
            // - Broadcasting main_q_global_block_offset
            // - Broadcasting nsurvival_tokens_in_CTA
            // - We may reuse temp_storage (cf CUB doc)
            __syncthreads(); 

            // Checking if we are overflowing the main_q
            if((main_q_global_block_offset.split.ntokens + nsurvival_tokens_in_CTA) >= params.q_capacity) {
                if(threadIdx.x == (KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX-1)) {
                    // We are overflowing the main_q
                    // We first revert what this CTA has done, ie revert the previous atomicAdd
                    // because all CTAs will revert, we know we will have a valid state after completion of this kernel
                    atomicAdd(&params.d_main_q_end_and_narcs_i2->both, -token_and_arc_count_block_sum.both); // revert

                    // Setting the flag. It will print a warning to stderr
                    *params.h_q_overflow = 1;
                }

                // We abort computation, we no longer have space in the main_q.
                // We still jump to finalize_kernel, to do what's needed before completion
                goto finalize_kernel;
            }

            // If we are executing the following lines it means that we are not overflowing the queue
            // We then continue what we were doing

            if(!is_pruned) {
                // This thread is in charge of a survival token
                // we will move it to the main_q, at index main_q_idx

                // Note : we could remove the branch divergence here 

                int32 main_q_idx = main_q_global_block_offset.split.ntokens + block_prefix_sum_token_arc_count.ntokens;

                InfoToken token_info = params.d_aux_q_info[aux_q_idx];

                // Moving the token to the main q
                params.d_main_q_state[main_q_idx] = token_state;
                params.d_main_q_cost[main_q_idx] = token_cost;
                params.d_main_q_info[main_q_idx] = token_info;

                // Saving the global prefix sum
                // = (narcs until now in the main queue) + (narcs until this thread in the CTA)
                params.d_main_q_degrees_prefix_sum[main_q_idx] = main_q_global_block_offset.split.narcs 
                                                                 + block_prefix_sum_token_arc_count.narcs;

                // Saving the CSR arc offset for that token's state
                // it will be used by the expand kernel, and avoid doing a new random memory access in the expand kernel
                params.d_main_q_arc_offsets[main_q_idx] = arc_start;
            }
        }

        finalize_kernel:

        // Avoiding races 
        // We will write d_aux_q_end
        // And some threads may be still reading it 
        // At the beg of this kernel
        __syncthreads();
        
        if(threadIdx.x == 0) {
            // Declaring the CTA as done
            int32 old = atomicAdd(params.d_n_CTA_done, 1);

            // If we're the last CTA to exit, detect it
            bool is_last_CTA = (old == (gridDim.x -1));

            if(is_last_CTA) {
                __threadfence();

                // We added things to the main_q
                // d_main_q_end was modified
                // we update h_main_q_end to keep it consistent
                // the h_* pointers are in the pinned host memory, we can access them from the device
                *params.h_main_q_end = *params.d_main_q_end;
                *params.h_main_q_narcs = *params.d_main_q_narcs;

                // We moved what we had to move from the aux q to the main q
                // We now empty the aux q 
                *params.d_aux_q_end = 0;
                *params.h_aux_q_end = 0; 

                // Reset the counter for next time
                *params.d_n_CTA_done = 0;
            }
        }

    }


    void CudaDecoder::PreprocessAndContract(const PreprocessParams &params) {
        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX;
        grid.x = DIV_ROUND_UP(*h_aux_q_end_, block.x);

        KALDI_ASSERT(grid.x > 0);

        _preprocess_and_contract_kernel<<<grid,block,0,compute_st_>>>(params);
    }



/*
    PreprocessInPlace
    This kernel is also a preprocessing kernel, but this time does it in place
    ie it will not move tokens from the aux_q to the main_q
    It will do the preprocess operation directly on the main_q
    The tokens are already in the main q (they were placed here by a previous "contract and preprocess").

    We cannot prune non-optimal tokens, because the tokens are already in the main_q (we cannot prune 
    the main_q - it would break the prev_token indexes). To avoid doing unnecessary computation 
    in the expand kernel, we simulate the pruning by setting non-optimal token's degree to 0
    We then rely on the 1 thread = 1 arc exact load balacing of expand to ignore that token

    Please note that even if 0 threads will perform work on an ignored token in expand (degree = 0),
    it is not exactly the same as pruning it : the main_q accesses will not be perfectly coalesced
    in expand, because some "dead" tokens exist between living ones

    For the preprocess stage we have to compute the prefix sum of the tokens arc degrees
    Here we have to do the prefix sum in two passes : first local prefix sums inside CUDA block,
    then in a second kernel (finalize_preprocess_in_place), we add the necessary block offsets to end up 
    with the global prefix sum

    This preprocess step is used in ProcessEmitting. Tokens were placed in main_q by
    the ProcessNonEmitting of the previous frame. We cannot renumber them (it would break
    the prev_token index). We preprocess in place, leaving things as they are in main_q

*/

    __global__ void _preprocess_in_place_kernel(PreprocessParams params) {
   
        // Operator for the prefix sum inside the CUDA block
        typedef hipcub::BlockScan<int32, KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX> BlockScan;
        __shared__ typename BlockScan::TempStorage temp_storage;


        // All threads in the last CUDA block (CTA) alive will have work to do at the end
        // this bool will be needed to broadcast the information from thread0 to all threads in the last CTA 
        __shared__ bool is_last_CTA;

        const int32 main_q_offset = *params.d_main_q_local_offset; // TODO ASSERT offset == 0
        const int32 main_q_end = *params.d_main_q_end;
        const int32 main_q_size = main_q_end - main_q_offset;

        // Final cutoff from the expand kernel
        const BaseFloat cutoff = *params.d_cutoff;

        // The condition of the for loop is the same for all threads in the CUDA block
        // we want to keep all threads alive at the same time for now
        // otherwise __syncthreads() would fail
        for(int32 block_offset = blockDim.x*blockIdx.x;
                block_offset < main_q_size;
                block_offset += gridDim.x*blockDim.x) {

            // Position of considered token in the main_q
            int32 main_q_idx = main_q_offset + block_offset + threadIdx.x; 

            // Total number of arcs from that token's state
            int32 degree = 0; 

            if(main_q_idx < main_q_end) {
                StateId token_state = params.d_main_q_state[main_q_idx]; 
                BaseFloat token_cost = params.d_main_q_cost[main_q_idx];

                // the cutoff may have decreased since the creation of that token
                if(token_cost < cutoff) {

                    // Best cost for that token_state
                    // We know we have a token associated with token_state in the queue with the cost state_best_cost
                    BaseFloat state_best_cost = orderedIntToFloat(params.d_state_best_cost[token_state]); 
                    
                    // We can have duplicates, ie token associated with the same states
                    // If this token is not the best candidate, get rid of it
                    if(token_cost == state_best_cost) {
                        int32 start = params.d_arc_offsets[token_state]; 
                        int32 end = params.d_arc_offsets[token_state+1]; 
                        degree  = end - start;
                        
                        // Saving the start offset for the expand kernel
                        // avoid a new random memory access
                        params.d_main_q_arc_offsets[main_q_idx] = start;
                    }
                }
            }

            int32 degree_local_prefix_sum;

            // Computing a local prefix sum inside that CUDA block
            // A second kernel will take care of adding the necessary offset to those local prefix sums
            BlockScan(temp_storage).ExclusiveSum(degree, degree_local_prefix_sum);

            if(main_q_idx < main_q_end) {
                // This is not the final global prefix sum
                // A second kernel will add the necessary offset
                params.d_main_q_degrees_prefix_sum[main_q_idx] = degree_local_prefix_sum; 
            }

            if(threadIdx.x == (KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX-1)) {
                // Saving the local sum of degrees of that CUDA block
                // That's necessary to compute the global offset of that CUDA block,
                // and that offset is what we need to transform the local prefix sum into a global prefix sum

                int local_sum_index = block_offset/KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX;
                int local_sum = degree_local_prefix_sum + degree; // the prefix sum was exclusive, adding missing value
                params.d_main_q_degrees_block_sums_prefix_sum[local_sum_index] = local_sum; 
            }


            // Synchronization for two reasons :
            // - we may need to reuse temp_storage if the for loop iterates (cf CUB's doc)
            // - we need all threads to be done before considering the CTA as done (see below)
            __syncthreads(); 

        }

        //
        // The last CUDA block alive will compute the prefix sum of the block degrees sum
        // We need that prefix sum, because it represents the offsets that each CUDA block has in the global prefix sum
        // we will then add those offsets in finalize_preprocess_in_place

        if(threadIdx.x == 0) {
            // We indicate that this CTA is done
            int32 old = atomicAdd(params.d_n_CTA_done, 1); 
            
            // If we're the last CTA to exit, detect it
            is_last_CTA = (old == (gridDim.x -1));
        }

        // Synchronization for two reasons :
        // - Broadcasting is_last_CTA
        // - reusing temp_storage (cf CUB's doc)
        __syncthreads();
        
        if(is_last_CTA)
        {
            //
            // Our goal here is to compute the prefix sum of the previous local sums
            // What we call local sum is what contains the local_sum variables in the previous lines
            // it is the sum of degrees inside a given CUDA block, at a given for loop iteration
            // all local sums are stored in params.d_main_q_degrees_block_sums_prefix_sum
            // we want to do the prefix sum of that array
            //
            // Once this is done, params.d_main_q_degrees_block_sums_prefix_sum[i] will contain the 
            // offset that we need to add to the local prefix sum #i to convert it to a global
            // prefix sum
            // Right now we are only computing the offsets ; adding them to the local prefix sums will be 
            // done in FinalizePreprocessInPlace
            //

            //
            // We are the last CTA alive
            // which means that all local sums have been written to params.d_main_q_degrees_block_sums_prefix_sum
            // We can now do the prefix sum of that array   
            //

            // Making sure that we see changes from other CTAs 
            __threadfence();

            //
            // How many local sums values do we have ?
            // Please note that this number can be different from gridDim.x
            // We may have applied a upper limit on gridDim.x, and in that case
            // gridDim.x < number_of_local_sums
            //

            int32 number_of_local_sums = DIV_ROUND_UP(main_q_size, KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX);

            // We may iterate the following for loop multiple times
            // on iteration > 0, we will have to consider the offset from previous iterations
            int32 prefix_sum_of_local_sums_offset = 0;

            // local_sum_index is an index in the array d_main_q_degrees_block_prefix
            // 
            // The condition inside the loop is common to all threads in the CTA
            // we want to keep all threads active, we will use syncthreads()
            for(int32 local_sum_index_offset = 0; 
                      local_sum_index_offset < number_of_local_sums; 
                      local_sum_index_offset += blockDim.x) {

                int32 local_sum_index = local_sum_index_offset + threadIdx.x; 

                int32 local_sum = (local_sum_index < number_of_local_sums) 
                                ? params.d_main_q_degrees_block_sums_prefix_sum[local_sum_index] 
                                : 0; // neutral element

                int32 prefix_sum_of_local_sums, total_sum_of_local_sums_for_this_iteration;

                BlockScan(temp_storage).ExclusiveSum(local_sum, prefix_sum_of_local_sums, total_sum_of_local_sums_for_this_iteration);

                prefix_sum_of_local_sums += prefix_sum_of_local_sums_offset;
                prefix_sum_of_local_sums_offset += total_sum_of_local_sums_for_this_iteration;

                if(local_sum_index < number_of_local_sums) {
                    params.d_main_q_degrees_block_sums_prefix_sum[local_sum_index] = prefix_sum_of_local_sums;
                }

                // Sync'ing to be able to reuse temp_storage (cf CUB's doc)
                __syncthreads();
            }

            if(threadIdx.x == 0)
            {
                // Final offset is the overall total
                int total_sum_of_local_sums = prefix_sum_of_local_sums_offset;
                *params.d_main_q_narcs = total_sum_of_local_sums; 
                // h_main_q_narcs is in pinned memory, we can write to it from the device
                *params.h_main_q_narcs = total_sum_of_local_sums; 
                // reset for next time
                *params.d_n_CTA_done = 0;
            }
        }
    }


    void CudaDecoder::PreprocessInPlace(const PreprocessParams &params) {
        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX;
        int32 main_q_size = *h_main_q_end_ - *h_main_q_local_offset_;

        grid.x = DIV_ROUND_UP(main_q_size, block.x);

        KALDI_ASSERT(grid.x > 0);

        _preprocess_in_place_kernel<<<grid,block,0,compute_st_>>>(params);
    }



    /*

       Part 2 of the prefix sum for "PreprocessInPlace" 
       
       For PreprocessAndContract we were able to do the global prefix sum of degrees in one pass, so we should not call
       this kernel

       Our final goal is to have the prefix sum of the degrees of the token's state of the main_q
       and store that prefix sum in d_main_q_degrees_prefix_sum

       In PreprocessInPlace we've computed two things :
       
       - "local prefix sums" of the degree. Each CUDA block has computed the local prefix sum of its degrees. We've
       stored each of the local prefix sums in d_main_q_degrees_prefix_sum
       - the prefix sum of the local sums (local sum = sum of all degrees in a CUDA block). This gives us the offset
       to add to each local prefix sum to end up with a global prefix sum

       Note : If we want to speed up expand, we can compute lower and upper bound to restrain 
       the binary search in expand
       This can be done on the fly here, and removes main bottleneck of expand

       TODO merge with ResetStateCostLookup
     */

    // d_main_q_degrees_prefix is both an input and an output
    __global__ void _finalize_degrees_scan_kernel(const int32 *d_local_sums_prefix_sum, 
                                                  const int32 *d_main_q_local_offset, 
                                                  const int32 *d_main_q_end, 
                                                  int32 *d_main_q_degrees_prefix_sum) {

        const int32 main_q_offset = *d_main_q_local_offset;
        const int32 main_q_end = *d_main_q_end;

        for(int32 main_q_idx = main_q_offset + blockDim.x*blockIdx.x + threadIdx.x;
                  main_q_idx < main_q_end;
                  main_q_idx += blockDim.x*gridDim.x) {

            int32 local_sum_idx = (main_q_idx - main_q_offset) / KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX;
            int32 local_sum_offset = d_local_sums_prefix_sum[local_sum_idx]; // we rely on the caches for this one

            d_main_q_degrees_prefix_sum[main_q_idx] += local_sum_offset;
        }

    }

    void CudaDecoder::FinalizePreprocessInPlace() {
        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX;

        // Should be called only at Emitting phase
        // during that phase, the main_q_offset must be zero
        KALDI_ASSERT(*h_main_q_local_offset_ == 0);

        // TODO remove code related to offset in that code
        int32 main_q_size = *h_main_q_end_ - *h_main_q_local_offset_;
        grid.x = DIV_ROUND_UP(main_q_size, block.x);

        // If the main_q is empty, we will not be able to continue
        KALDI_ASSERT(grid.x > 0);

        _finalize_degrees_scan_kernel<<<grid,block,0,compute_st_>>>(d_main_q_degrees_block_sums_prefix_sum_, 
                                                            d_main_q_local_offset_,
                                                            d_main_q_end_, 
                                                            d_main_q_degrees_prefix_sum_);
    }



   //
   // Helper functions/data structure for the ExpandArc kernel
   //


   // We'll need to do a BlockScan on both an int and a cost
   // data struct and its associated binary operation

    struct CostTypeAndInt {
        CostType cost;
        int32 i;
    };

    // 
    // We'll use the same BlockScan to compute two things :
    //     1) The prefix sum of indexes
    //     1) The minimum cost overall all costs in the CUDA Block 
    //
    // We use a + for the prefix sum, and a min for the min
    //

    struct MinCostPlusInt {
        __device__ CostTypeAndInt operator()(const CostTypeAndInt &a, const CostTypeAndInt &b) const {
            CostTypeAndInt c;
            c.cost = fmin(a.cost, b.cost);
            c.i = a.i + b.i;
            return c;
        }
    };

    //
    // GetCutoffCandidate is used by ExpandArc and NonEmittingLongTail
    // It computes a candidate for a new cutoff. It will not necessarily be the new cutoff,
    // we'll apply an atomicMin(d_cutoff, cutoff_candidate)
    //
    // The cutoff represents the upper limit of acceptable token cost 
    // with min_cost = minimum token.cost for all token of the current frame,
    // we will not consider token with cost > (min_cost + beam), beam being a parameter
    //
    // However, given the fact that the output token queue (aux_q) is too small to store 
    // all possible tokens in the worst case scenario (where we could generate "nstates" tokens),
    // we need to tighten the beam if we notice that we are at risk of overflowing the aux_q
    //

    __device__ __inline__ CostType GetCutoffCandidate(const CostType current_cutoff,
            const CostType min_cost_in_block,
            const CostType default_beam,
            const int32 q_size,
            const int32 q_capacity) {


        // Doing something simple for now
        // We have to keep beam large enough,
        // the final cutoff will be used for the final
        // prune. If it is too small, we won't keep enough tokens

        CostType beam = default_beam;

        // TODO do something better 
        if(q_size >= q_capacity/2) 
            beam /= 2;

        return fmin(current_cutoff, min_cost_in_block + beam);
    }

    __device__ __forceinline__ int32 binsearch_maxle(const int32 *vec, const int32 val, int32 low, int32 high) {
        while(true) {
            if(low == high)
                return low; //we know it exists
            if((low + 1) == high)
                return (vec[high] <= val) ? high : low;

            int32 mid = low + (high- low) / 2;

            if(vec[mid] > val)
                high = mid-1;
            else
                low = mid;
        }
    }


    //
    // ExpandArc kernel
    // This kernel does the actual work of traversing arcs 
    //
    // Pseudo code :
    // for all token tok in main_q[main_q_offset...end]:
    //      u = tok.next_state
    //      for all arc a(u->v) in the FST:
    //          v_cost = tok.cost + a.cost + accoustic_cost
    // 
    //          if v_cost < cutoff and v_cost < best_state_cost[v]
    //              generate token associated to v, add to aux_q
    //              update best_state_cost[v]
    //              if necessary update cutoff
    //
    // For more information please refer to http://kaldi-asr.org/doc/decoders.html
    //
    // ExpandArc rely on some preprocessed data to be able to function 
    // for instance, it needs the prefix sum of the arc degree of all token.state in the
    // main_q
    // We need to call a Preprocess kernel before ExpandArc
    //
    // ExpandArc is used for both emitting and nonemitting phases
    // Differences between emitting and nonemitting :
    //      1) params.d_q_arc_offset contains offsets to either emitting or nonemitting arcs. 
    //         It is transparent for this kernel. The differentiation was done in the Preprocess kernel,
    //         which is responsible for filling the params.d_q_arc_offset array
    //      2) Computation of the acoustic cost. If nonemitting, it is equal to 0. If emitting, we need
    //         to use values from the acoustic model (through the d_loglikelihoods array)
    //
    //
    //
    // Note : ExpandArc is not the only kernel able to traverse arcs. 
    // NonEmittingLongTail contains a simplified version of expand for only one CUDA block
    //

    void __global__ _expand_arcs_kernel(ExpandArcParams params) {

        // BlockScan that we will use to compute token indexes in the output queue, 
        // and to find the min cost in the block
        typedef hipcub::BlockScan<CostTypeAndInt, KALDI_CUDA_DECODER_KERNEL_EXPAND_ARCS_DIMX> BlockScan;
        __shared__ typename BlockScan::TempStorage temp_storage_scan;

        // This kernel writes the new token to the output queue aux_q
        // We will request a spot to store all the new tokens created by threads in this CUDA block
        // aux_q_index_block_offset indicates where to store them in the aux_q
        // tokens created in this CUDA block will be store in :
        // aux_q[aux_q_index_block_offset], aux_q[aux_q_index_block_offset + 1], ...
        __shared__ int32 aux_q_index_block_offset;

        //
        // Cutoff, stored in shared for caching purposes
        // TODO rely on cache ?
        //
        __shared__ CostType cached_cutoff;

        const int32 total_narcs = *params.d_main_q_narcs;
        const int32 main_q_offset = *params.d_main_q_local_offset;
        const int32 main_q_end = *params.d_main_q_end;

        
        if(threadIdx.x == 0) {
            cached_cutoff = *params.d_cutoff;
        }

        __syncthreads();

        // The condition of this for loop is common for all threads in the block
        // We need to keep all threads in the block alive in the same time
        // We'll have syncs inside the for loop, and we need to have all threads alive during
        // those syncs
        // in the future we may rely on coop groups
        for(int32 main_q_arc_index_block_offset = blockDim.x*blockIdx.x;
                  main_q_arc_index_block_offset < total_narcs;
                  main_q_arc_index_block_offset += gridDim.x*blockDim.x) {

            //
            // Important : this thread is not responsible for a token in the input queue main_q
            // but for an arc, going out of a token in the main_q
            // The main_q contains in total total_narcs
            // and this thread will compute the main_q_arc_index-th arc of the main_q
            // For instance, first thread in the grid with threadIdx.x == 0 and blockIdx.x == 0 
            // will process the first arc of the token in main_q[main_q_offset + 0] 
            // (if that token has at least one arc)
            //
            // This insure a perfect one thread = one arc load balancing
            // but we have work to do to know exactly which arc is the main_q_arc_index-th arc
            // (what's its source ? its destination ? its arc_idx the FST CSR ?)
            //

            int32 main_q_arc_index = main_q_arc_index_block_offset + threadIdx.x;

            // We'll need those variables later in the kernel
            // we declare them outside of the "valid_input" scope
            // to be able to access them later

            int32 main_q_idx;
            int32 arc_idx;
            StateId arc_next_state;
            BaseFloat total_cost = FLT_MAX;

            if(main_q_arc_index < total_narcs) {

                //
                // Current thread must take care of main_q_arc_index-th arc
                // we need to now what's the source of that arc
                // ie which token.state in main_q does it start from ? 
                // We use a binary search in the prefix sum of the token's degree to get that information
                // 
                // Example : main_q contains 3 tokens
                // - First token is associated to a state which has 3 outgoing arc
                // - Second token is associated to a state which has 0 outgoing arc
                // - Third token is associated to a state which has 2 outgoing arc
                //
                // We store the degrees in an array :
                // [3, 0, 2]
                //
                // We then compute the exclusive prefix sum of that array :
                // [0, 3, 3, 5]
                //
                // In total, we have 5 arcs in the main_q. ExpandArc will use 5 threads.
                //
                // Let's say we are the fifth thread in ExpandArc. 
                // we have threadIdx.x == 4, and blockIdx.x == 0
                // it gives us main_q_arc_index == 4
                // From there we have no idea what we're supposed to do next, we need to have information about the
                // arc that we're supposed to traverse
                //
                // To do that, we look for the maximum index maxle_i in the prefix sum array such prefix_sum[i] <= 4
                //
                // [0, 3, 3, 5]
                //         /\
                //         here
                // maxle_i = 2
                // it means that our source token is at index 2 in the main_q
                // and we are computing the arc at index (main_q_arc_index - prefix_sum[maxle_i]) of that token 
                // ie the arc at index (4-3) = 1, the second arc of the second token in main_q
                //

                main_q_idx = binsearch_maxle(params.d_main_q_degrees_prefix_sum, main_q_arc_index, main_q_offset, main_q_end-1); 

                int32 lower_bound = params.d_main_q_degrees_prefix_sum[main_q_idx];
                int32 arc_offset_start = params.d_q_arc_offsets[main_q_idx];

                arc_idx = arc_offset_start + (main_q_arc_index - lower_bound);
                arc_next_state = params.arc_nextstates[arc_idx];

                // Building the total cost incrementally 
                // we'll add the acoustic cost and the old token's cost
                total_cost = params.arc_weights[arc_idx];

                int32 arc_ilabel = params.is_emitting ? params.arc_ilabels[arc_idx] : 0;
                total_cost += (arc_ilabel != 0) ? -params.d_loglikelihoods[arc_ilabel] : 0.0; 
                total_cost += params.d_main_q_cost[main_q_idx];

                // If the total_cost is too large compared to our cutoff (beam search)
                // then let's drop it
                if(total_cost >= cached_cutoff)
                    total_cost = FLT_MAX;
                else {
                    // We need to check if we already have a token going to that next_state,
                    // and if that token has a lower cost that we have
                    // params.d_state_best_cost[state] contains the best cost for that state in the current frame
                    BaseFloat next_state_best_cost = orderedIntToFloat(params.d_state_best_cost[arc_next_state]);

                    // If that token is the best for that state, drop it
                    if(total_cost >= next_state_best_cost)
                        total_cost = FLT_MAX;
                }
            }

            //
            // If total_cost < FLT_MAX, it means that : 
            // - this thread had a valid input (main_q_arc_index < total_narcs)
            // - the total_cost of the generated token is < cutoff
            // - the generated token is the best candidate for that next_state
            // We will then add that new token in the output queue, aux_q
            // We need to know where to put that token in the aux_q
            // we'll first compute its index inside the CUDA block
            // the first valid output token in the CUDA block will have index 0, 
            // the second index 1... We compute that using a prefix sum
            //
            // We also need to find the overall min cost in the CUDA block
            // a prefix sum is a scan operation, and a min a reduce operation
            // we can perform a reduce operation using a scan (using the last value)
            // we compute the prefix sum and the min in one scan, using the data 
            // struct CostTypeAndInt
            //

            int32 has_successor = (total_cost < FLT_MAX) ? 1 : 0; 

            CostTypeAndInt cost_and_index;
            cost_and_index.cost = total_cost; 
            cost_and_index.i = has_successor;

            // This is an /inclusive/ scan
            BlockScan(temp_storage_scan).InclusiveScan(cost_and_index, cost_and_index, MinCostPlusInt());

            if(threadIdx.x == (KALDI_CUDA_DECODER_KERNEL_EXPAND_ARCS_DIMX - 1)) {
                // This is the last thread. The last value of the inclusive scan is the total
                int32 total_successors_in_block = cost_and_index.i;
                
                // Requesting a spot of size total_successors_in_block in the aux_q
                aux_q_index_block_offset = atomicAdd(params.d_aux_q_end, total_successors_in_block);

                //
                // Here we detect an overflow of the aux_q
                // we detect it before actually using the aux_q
                // We try to prevent an overflow from happening using an adaptive beam (cf GetCutoffCandidate)
                //
                if((aux_q_index_block_offset + total_successors_in_block) >= params.q_capacity) {
                    // aux_q_index_block_offset is in shared memory
                    // its value is currently invalid (overflow)
                    // we set it to a special value and use it as a flag to broadcast
                    // the fact that we have an overflow and that all threads should exit
                    aux_q_index_block_offset = params.q_capacity;

                    // We revert the last operation. All threads that detected the overflow 
                    // will revert what they've done. It means that at the end of the kernel,
                    // we'll be back to the last valid state 
                    // We'll be able to continue computation, but quality of the output
                    // may be lower (we weren't able to save all tokens)
                    atomicAdd(params.d_aux_q_end, -total_successors_in_block); 

                    // Setting the flag for the host. It will be used to print a warning to stderr
                    *params.h_q_overflow = 1; 
                } else {

                    /*

                       GetCutoffCandidate takes into account the current value of 
                       d_aux_q_end and compares it with its maximum capacity.
                       If necessary it progressively cuts down the beam 
                       (reducing the cutoff) to only keep the best candidates
                       and avoiding an overflow

                     */

                    CostType cutoff_candidate = GetCutoffCandidate(cached_cutoff,
                            cost_and_index.cost,
                            params.beam,
                            aux_q_index_block_offset + total_successors_in_block,
                            params.q_capacity);

                    cached_cutoff = (cutoff_candidate < cached_cutoff) 
                        ? fmin(fatomicMin(params.d_cutoff, cutoff_candidate), cutoff_candidate)
                        : fmin(*params.d_cutoff, cached_cutoff);
                }
            }

            // Sync'ing for two reasons :
            // - Broadcasting aux_q_index_block_offset
            // - reusing temp_storage (cf CUB's doc)
            __syncthreads(); 


            // The only case where we can have that condition met,
            // if we detected an overflow if the previous lines
            // we need to finalize our work and quit 
            if(aux_q_index_block_offset == params.q_capacity) 
                goto finalize_kernel; // keeping things clean before aborting

            //
            // If we're executing the following lines it means everything
            // is valid and we are not overflowing the aux_q
            //

            cost_and_index.i -= has_successor; // we want the exclusive sum now

            int32 aux_q_block_index = cost_and_index.i;
            int32 aux_q_index = aux_q_index_block_offset + aux_q_block_index;

            if(has_successor) {
                // We save the new token to the aux_q
                
                params.d_aux_q_cost[aux_q_index] = total_cost;
                params.d_aux_q_state[aux_q_index] = arc_next_state;
                
                // Updating the best_state_cost lookup table with our new best cost
                atomicMin(&params.d_state_best_cost[arc_next_state],
                        floatToOrderedInt(total_cost)
                        );

                // We've updated the cached_cutoff since created the new token
                // there's a chance that we no longer need to use that token
                // we've saved the cost and the state to be able to ignore it in the following prune operation
                // but there's no need to write out the infos

                if(total_cost < cached_cutoff) { 
                    InfoToken new_tok_info;
                    // Index of the parent token
                    // the parent is the token used as input 
                    // that parent is at index main_q_idx in the GPU memory
                    // However, the main_q is emptied before processing a new frame
                    // we need to add the offset related to the previous frames index
                    // we add params.main_q_global_offset
                    new_tok_info.prev_token = params.main_q_global_offset + main_q_idx;
                    new_tok_info.arc_idx = arc_idx;

                    params.d_aux_q_info[aux_q_index] = new_tok_info;
                }
            }
        }

        finalize_kernel:

        // We want to be sure that all threads are done before declaring this CUDA block as done
        __syncthreads(); 

        if(threadIdx.x == 0) {
            // Declaring this CTA as done
            int32 old = atomicAdd(params.d_n_CTA_done, 1);

            // If we're the last CTA to exit - detect it
            if(old == (gridDim.x -1)) {
                __threadfence(); // we want last value of d_aux_q_end

                // h_* pointers are in pinned memory, we can update them from the GPU
                *params.h_aux_q_end = *params.d_aux_q_end;
                *params.d_main_q_narcs = 0;
                *params.h_main_q_narcs = 0;
                *params.d_n_CTA_done = 0; 

                if(params.is_emitting) {
                    *params.d_main_q_local_offset = 0; // not needed
                    *params.h_main_q_local_offset = 0; // not needed

                    // It was the last time that we were using tokens in the main_q
                    // flushing it now
                    *params.d_main_q_end = 0;
                    *params.h_main_q_end = 0;
                } else {
                    // Tokens processed in that nonemitting iteration will be ignored in the next iteration
                    *params.d_main_q_local_offset = main_q_end;
                    *params.h_main_q_local_offset = main_q_end;
                }

            }
        }

    }

    void CudaDecoder::ExpandArcs(const ExpandArcParams &params, int32 nthreads) {
        dim3 grid,block;
        block.x = 256;
        grid.x = DIV_ROUND_UP(nthreads, block.x);

        KALDI_ASSERT(grid.x > 0);

        _expand_arcs_kernel<<<grid,block,0,compute_st_>>>(params);
    }



    /*
        
       NonEmittingLongTail
       Meta-kernel (merging preprocess and expand) but only works with 1 CUDA block

       Used to avoid calling multiple "heavy lifting" kernels for the tail of non emitting
       (lots of iterations with small number of arcs)

       Code is greatly simplified because we can have only one CTA alive

       Repeat until new queue empty:
       1) Preprocess 
       2) Expand arcs

       The preprocess stage is not done on the first iteration, because it was
       already done by the ProcessAndContract kernel. We always call ProcessAndContract
       before calling NonEmittingLongTail 

       At the end, this kernel finalize the computation for current frame,
       so that it's ready for next ProcessEmitting

       TODO This kernel could be easily optimized  

       Note : For a detailed description on how the Preprocess and Expand operation work,
       please refer to the PreprocessInPlace and ExpandArc kernel implemention. The algorithm are 
       described there. In this kernel, we compute simplified version of preprocess and expand, because
       we do not need inter-block communication (we launch only one CUDA block)

       Important : in ExpandArc, the input is the main_q, the ouput is the aux_q. We then call PreprocessAndContract
       that move the tokens from the aux_q to the main_q.
       Here we directly output the tokens in the main_q. It helps use simplify the code, and we are not generating a lot
       of tokens anyway (so the pruning stage of PreprocessAndContract is less critical)

     */


    __launch_bounds__(KALDI_CUDA_DECODER_KERNEL_NONEM_LT_DIMX, 1)
        __global__ void _process_nonem_longtail(const uint32_t *d_arc_offsets, 
                ExpandArcParams params) {
            
            // Used to find the minimum cost in the CUDA block
            typedef hipcub::BlockReduce<float, KALDI_CUDA_DECODER_KERNEL_NONEM_LT_DIMX> BlockReduce;
            __shared__ typename BlockReduce::TempStorage temp_storage_reduce;

            // Used to compute the index in the output queue
            typedef hipcub::BlockScan<int32, KALDI_CUDA_DECODER_KERNEL_NONEM_LT_DIMX> BlockScan;
            __shared__ typename BlockScan::TempStorage temp_storage_scan;

            // Cutoff for the beam search. 
            // during the execution of the kernel, it will not be necessary to update params.d_cutoff
            // (from the global memory). We're the only CUDA block executing 
            __shared__ BaseFloat cutoff;

            //
            // main_q is both input and output
            // We are using offsets to differenciate the two subqueue
            //
            // main_q [0 .... input_q_offset ...... output_q_offset ...... output_q_end]
            //                                 /\                     /\ 
            //                            Input queue            Output queue
            //
            // At the beginning of the iteration, output_q_offset == output_q_end
            // we then fill the output queue and increment output_q_end
            //

            int32 input_q_offset = *params.d_main_q_local_offset;
            int32 output_q_offset = *params.d_main_q_end;
            
            int32 output_q_end = output_q_offset;

            int32 total_narcs = *params.d_main_q_narcs;
    
            int32 input_q_size = output_q_offset - input_q_offset;  

            cutoff = *params.d_cutoff;

            // We'll switch queue at the beg of the loop
            output_q_offset = input_q_offset;

            bool first = true;

            while(input_q_size > 0) {

                // cutoff ready
                __syncthreads(); 
                
                // Step 0 : move queues        
                input_q_offset = output_q_offset;
                output_q_offset = output_q_end;

                // Used to clarify the code
                int32 input_q_end = output_q_offset;
                
                if(!first) {
                    total_narcs = 0;

                    // Step 1 : compute_degrees
                    // TODO fuse 1 and 2
                    for(int32 q_idx = input_q_offset + threadIdx.x;
                              q_idx < input_q_end;
                              q_idx += blockDim.x) {

                        StateId token_state = params.d_main_q_state[q_idx];
                        BaseFloat token_cost = params.d_main_q_cost[q_idx];

                        int32 degree = 0;
                        if(token_cost < cutoff) {
                            BaseFloat best_cost = orderedIntToFloat(params.d_state_best_cost[token_state]);

                            if(token_cost == best_cost) {
                                int32 start = d_arc_offsets[token_state];
                                int32 end = d_arc_offsets[token_state+1];
                                degree = end - start;
                                params.d_q_arc_offsets[q_idx] = start;
                            }
                        }

                        params.d_main_q_degrees_prefix_sum[q_idx] = degree;
                    }

                    __syncthreads(); // will be removed

                    // Step 2 : Scan

                    for(int32 block_off = 0;
                              block_off < input_q_size;
                              block_off += blockDim.x) {

                        int32 q_idx = input_q_offset + block_off + threadIdx.x;

                        int32 degree = (q_idx < output_q_offset) 
                            ? params.d_main_q_degrees_prefix_sum[q_idx]
                            : 0;

                        int32 degree_prefix_sum;
                        int32 degree_sum_for_this_iteration;
                        BlockScan(temp_storage_scan).ExclusiveSum(degree, degree_prefix_sum, degree_sum_for_this_iteration);
                        int32 scan = degree_prefix_sum + total_narcs;
                        total_narcs += degree_sum_for_this_iteration;

                        if(q_idx < output_q_offset)
                            params.d_main_q_degrees_prefix_sum[q_idx] = scan;

                         __syncthreads(); // reusing temp_storage_scan + degrees ready
                    }


                } else {
                    first = false;    
                }


                // We already sync'ed

                // Step 3 : expand arcs

                for(int32 main_q_arc_index_block_offset = 0;
                        main_q_arc_index_block_offset < total_narcs;
                        main_q_arc_index_block_offset += blockDim.x) {

                    int32 main_q_arc_index = main_q_arc_index_block_offset + threadIdx.x;
                    bool valid_input = (main_q_arc_index < total_narcs);

                    // For details on how this code works, please refer to ExpandArc's comments


                    BaseFloat total_cost = FLT_MAX;
                    int32 arc_idx;
                    StateId arc_next_state;
                    int32 q_idx;

                    if(valid_input) {
                        q_idx = binsearch_maxle(params.d_main_q_degrees_prefix_sum, main_q_arc_index, input_q_offset, output_q_offset-1); 

                        int32 lower_bound = params.d_main_q_degrees_prefix_sum[q_idx];
                        int32 arc_offset_start = params.d_q_arc_offsets[q_idx];

                        arc_idx = arc_offset_start + (main_q_arc_index - lower_bound);

                        arc_next_state = params.arc_nextstates[arc_idx];
                        BaseFloat arc_weight = params.arc_weights[arc_idx];
                        BaseFloat next_state_cost = orderedIntToFloat(params.d_state_best_cost[arc_next_state]);
                        BaseFloat old_tok_cost = params.d_main_q_cost[q_idx];

                        total_cost = arc_weight + old_tok_cost;

                        if(total_cost >= next_state_cost) {
                            total_cost = FLT_MAX;
                            valid_input = false; 
                        } 
                    }

                    BaseFloat min_cost = BlockReduce(temp_storage_reduce).Reduce(total_cost, hipcub::Min());

                    if(threadIdx.x == 0) {
                        cutoff = GetCutoffCandidate(cutoff,
                                min_cost,
                                params.beam,
                                output_q_end,
                                params.q_capacity);
                    }

                    __syncthreads();

                    int32 has_successor = (total_cost < cutoff && valid_input) ? 1 : 0;

                    if(has_successor) 
                        atomicMin(&params.d_state_best_cost[arc_next_state], floatToOrderedInt(total_cost));

                    int32 new_q_idx_block = has_successor;
                    int32 total_in_blk;
                    BlockScan(temp_storage_scan).ExclusiveSum(new_q_idx_block, new_q_idx_block, total_in_blk);

                    if((output_q_end + total_in_blk) >= params.q_capacity) {
                        *params.h_q_overflow = 1;
                        
                        goto finalize_kernel; // keeping things clean before aborting
                    }

                    if(has_successor) {
                        int32 new_q_index = output_q_end + new_q_idx_block;
                        params.d_main_q_state[new_q_index] = arc_next_state;

                        params.d_main_q_cost[new_q_index] = total_cost;

                        InfoToken new_tok_info;
                        new_tok_info.prev_token = params.main_q_global_offset + q_idx;

                        new_tok_info.arc_idx = arc_idx;
                        params.d_main_q_info[new_q_index] = new_tok_info;
                        
                        //print32f("new q index = %i (%i+%i) (tot=%i) \n", new_q_index, output_q_end, new_q_idx_block,
                        //total_in_blk);
                   }

                    output_q_end += total_in_blk;
                }

                input_q_size = output_q_end - output_q_offset; 
            }

            finalize_kernel:

            if(threadIdx.x == 0) {
                // Next step is ProcessEmitting of next frame, from is currToken_offset
                *params.d_main_q_end = output_q_end; 
                *params.d_main_q_narcs = 0;

                *params.h_main_q_end = output_q_end; 
                *params.h_main_q_narcs = 0; 

                *params.d_main_q_local_offset = 0; 
                *params.h_main_q_local_offset = 0; 

                *params.d_cutoff = cutoff;
            }

        }

    void CudaDecoder::NonEmittingLongTail(const uint32_t *d_arc_offsets, 
            const ExpandArcParams &params) {

        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_KERNEL_NONEM_LT_DIMX;
        grid.x = 1; // it is designed for the long tail
        _process_nonem_longtail<<<grid,block,0,compute_st_>>>(d_arc_offsets, params);
    }


} // end namespace kaldi
