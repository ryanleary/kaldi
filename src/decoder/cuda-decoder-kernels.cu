#include "hip/hip_runtime.h"
// decoder/cuda-decoder-kernels.cu

// See ../../COPYING for clarification regarding multiple authors
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//  http://www.apache.org/licenses/LICENSE-2.0
//
// THIS CODE IS PROVIDED *AS IS* BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, EITHER EXPRESS OR IMPLIED, INCLUDING WITHOUT LIMITATION ANY IMPLIED
// WARRANTIES OR CONDITIONS OF TITLE, FITNESS FOR A PARTICULAR PURPOSE,
// MERCHANTABLITY OR NON-INFRINGEMENT.
// See the Apache 2 License for the specific language governing permissions and
// limitations under the License.

#include <hipcub/hipcub.hpp>
#include "decoder/cuda-decoder.h"

#define DIV_ROUND_UP(a,b) ((a+b-1)/b)

namespace kaldi {

typedef CudaDecoder::StateId StateId;
typedef CudaDecoder::TokenAndArcCount TokenAndArcCount;
typedef CudaDecoder::TokenAndArcCountUnion TokenAndArcCountUnion;
typedef CudaDecoder::CostType CostType;
typedef CudaDecoder::PreprocessParams PreprocessParams; 
typedef CudaDecoder::ExpandArcParams ExpandArcParams; 

//
// Utils device function
//


    //
    // 1:1 Conversion float <---> sortable int
    // We convert floats to sortable ints in order
    // to use native atomics operation, which are 
    // way faster than looping over atomicCAS 
    //

    __device__ int32 floatToOrderedInt(float floatVal) {

        int32 intVal = __float_as_int( floatVal );

        return (intVal >= 0 ) ? intVal : intVal ^ 0x7FFFFFFF;
    }



    __device__ float orderedIntToFloat(int32 intVal) {

        return __int_as_float( (intVal >= 0) ? intVal : intVal ^ 0x7FFFFFFF );

    } 

    // Temporary used for cutoff - will be TODO removed
    __device__ float fatomicMin(float *addr, float value)

    {

        float old = *addr, assumed;
        if(old <= value) return old;

        do
        {
            assumed = old;
            old = atomicCAS((uint32_t*)addr,
                    __float_as_int(assumed),
                    __float_as_int(value));

        } while(old!=assumed); // TODO <

        return old;

    }

    //
    // Kernels
    //

    // For description of what each kernel is doing, please refer to cuda-decoder.h
    // and look for the corresponding wrapper
    // for instance, for a description of _init_lookup_kernel,
    // look for the description of CudaDecoder::InitStateCostLookup() in cuda-decoder.h

    // Used before first frame
    __global__ void _init_state_cost_lookup_kernel(int32 size, int32 *state_cost) {
        for(int32 idx = blockIdx.x*blockDim.x + threadIdx.x;
                idx < size;
                idx += blockDim.x*gridDim.x) {
            state_cost[idx]  = floatToOrderedInt(FLT_MAX);
        }
    }

    void CudaDecoder::InitStateCostLookup() {
        int32 nstates = fst_.numStates;
        KALDI_ASSERT(nstates > 0);

        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_KERNEL_INIT_LOOKUP_DIMX;
        grid.x = DIV_ROUND_UP(nstates, block.x);

        _init_state_cost_lookup_kernel<<<grid,block>>>(nstates, d_state_cost_);
    }

    // Used to reset lookup table between frames
    // Using the queue to reset only the values needed
    // Also takes care of resetting cutoff
    __global__ void _reset_state_cost_lookup_kernel(const StateId *d_main_q_state_, const int32 *d_main_q_end_, int32 *d_state_cost, CostType *d_cutoff) {
        int32 main_q_end = *d_main_q_end_; 

        for(int32 idx = blockIdx.x*blockDim.x + threadIdx.x;
                idx < main_q_end;
                idx += blockDim.x*gridDim.x) {
            // d_main_q_state_ contains the list of states that we've considered in the last frame
            // it corresponds to the list of indexes i such as d_state_cost[i] < +INF
            // faster than init_state_cost_lookup_kernel by a factor of ~10
            StateId state = d_main_q_state_[idx];
            d_state_cost[state]  = floatToOrderedInt(FLT_MAX);
        }

        if(blockIdx.x == 0 && threadIdx.x == 0)
            *d_cutoff = FLT_MAX; // we also reset the cutoff
    }

    void CudaDecoder::ResetStateCostLookup() {
        int32 size = *h_main_q_end_;

        KALDI_ASSERT(size > 0);

        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_KERNEL_INIT_LOOKUP_DIMX;
        grid.x = DIV_ROUND_UP(size, block.x);

        _reset_state_cost_lookup_kernel<<<grid,block,0,compute_st_>>>(d_main_q_state_, d_main_q_end_, d_state_cost_, d_cutoff);
    }


    // Sum operator for the TokenAndArcCount struct (2 ints) 
    // Used in preprocess_and_contract
    struct TokenAndArcCountSum {
        __device__ TokenAndArcCount operator()(const TokenAndArcCount &a, const TokenAndArcCount &b) const {
            TokenAndArcCount c;
            c.ntokens = a.ntokens + b.ntokens;
            c.narcs = a.narcs + b.narcs;

            return c;
        }
    };

    /*
       This kernel preprocess the necessary information for expand (scan of the outgoing degrees) 
       and explicitly prune the tokens

       The ExpandArc kernel writes the new raw token list in the aux_q. However, the cutoff 
       was progressively lowered during the computation, and some tokens now have a cost > cutoff.
       During the contract stage of this kernel, we remove such tokens. 
       We also remove duplicates, i.e. tokens pointing to the same state, but with token.cost > best_cost_for_that_state

       It contracts (by pruning) the queue list:
       raw output in aux_q ----contract----> pruned output in main q

       This kernel is responsible for :

       1) Read a token from the aux queue (raw output from previous expand)

       2) Compute the outgoing degree of that token.next_state. For that :
       -> If that token is suboptimal (cutoff, best_cost), we prune it
       -> Otherwise, we will move it to the main_q. We also read its arc degree in the FST graph 

       3) We move the non-pruned tokens into the main q. After a local prefix sum,
       we request a spot in the main_q for those tokens using the main_q_end_and_narcs counter. 
       main_q_end_and_narcs.split.end contains the number of tokens in the main q until now
       main_q_end_and_narcs.split.narcs contains the number of arcs in the main q until now

       We also compute the degrees prefix sum in one pass using the main_q_end_and_narcs.split.narcs

       This kernel is used before ProcessNonEmitting
    */

    // Important : pass the struct PreprocessParams by copy - passing it using a ref will not work (CPU -> GPU)
    __global__ void _preprocess_and_contract_kernel(PreprocessParams params) {
        
        // Prefix sum operator
        typedef hipcub::BlockScan<TokenAndArcCount, KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX> BlockScan;
        __shared__ typename BlockScan::TempStorage temp_storage;

        // This CUDA block (CTA) will count the number of tokens it has to move to the main_q
        // and store the result in nsurvival_tokens_in_CTA
        __shared__ int32 nsurvival_tokens_in_CTA;

        // We need to move the survival tokens to the main_q
        // 
        // main_q_global_block_offset has two purposes :
        // (1) to know where to store the survival tokens in the main_q
        // (2) to perform the prefix sum degrees of the survival degrees
        //
        // The reason why we store those two values together is because they are linked (see below)
        //
        // (1) We need a spot to store those tokens in the main_q 
        // We will ask the main_q counter where to store those tokens, the answer will be 
        // an offset of the main_q. We will store our tokens in positions :
        // d_main_q_state[main_q_global_block_offset.ntokens], d_main_q_state[main_q_global_block_offset.ntokens+1]...
        //
        // (2) main_q_global_block_offset.narcs contains the number of arcs in the main_q up until index main_q_global_block_offset.ntokens
        // ie the number of arcs going out of all states in d_main_q_state[0..main_q_global_block_offset.ntokens]
        // it is used to compute the global prefix sum of degrees in one pass
        //
        __shared__ TokenAndArcCountUnion main_q_global_block_offset;

        // Final cutoff from last ExpandArc execution
        const BaseFloat cutoff = *params.d_cutoff;

        const int32 aux_q_end = *params.d_aux_q_end;

        // The condition of the for loop is the same for all threads in the CUDA block
        // we want to keep all threads alive at the same time for now
        // otherwise __syncthreads() would fail
        for(int32 block_offset = blockDim.x*blockIdx.x;
                block_offset < aux_q_end;
                block_offset += gridDim.x*blockDim.x) {

            int32 aux_q_idx = block_offset + threadIdx.x;
            int32 degree = 0;
            int32 arc_start = -1;

            StateId token_state;
            CostType token_cost;

            // if aux_q_idx is a valid index in the main_q
            if(aux_q_idx < aux_q_end) {
                // Cost and state associated with the token
                token_cost = params.d_aux_q_cost[aux_q_idx];
                token_state = params.d_aux_q_state[aux_q_idx];

                // Best cost for that token_state
                // We know we have a token associated with token_state in the queue with the cost state_best_cost
                BaseFloat state_best_cost = orderedIntToFloat(params.d_state_cost[token_state]);

                // Cutoff may have decreased since the creation of the token
                if(token_cost < cutoff) {
                    
                    // We can have duplicates, ie token associated with the same states
                    // If this token is not the best candidate, get rid of it
                    if(token_cost == state_best_cost) {
                        arc_start = params.d_arc_offsets[token_state];
                        int32 arc_end = params.d_arc_offsets[token_state+1];
                        degree = arc_end - arc_start;
                    }
                }

                // the d_state_cost lookup table is reset to +INF for all states between frame
                // for perf. reason we only reset states that are in d_main_q_state
                // however if state_best_cost >= cutoff, all tokens associated with token_state 
                // will be pruned, and that state will not be in d_main_q_state
                // we need to reset the lookup table now

                if (state_best_cost >= cutoff)
                    params.d_state_cost[token_state] = floatToOrderedInt(FLT_MAX);

            }

            int32 is_pruned = (arc_start == -1);


            TokenAndArcCount block_prefix_sum_token_arc_count;

            // We now know which tokens will be moved to the main_q, the remaining will be pruned
            // we now compute a prefix sum inside the CUDA block to determine the local indexes of the survival tokens
            // the first survival token will have a index of 0, the second 1, ...
            block_prefix_sum_token_arc_count.ntokens =  is_pruned ? 0 : 1;
            
            // We also need to compute the prefix sum of the degrees
            // we start by doing a local prefix sum inside the CUDA block
            block_prefix_sum_token_arc_count.narcs =  degree;

            TokenAndArcCount zero_struct;
            zero_struct.ntokens = zero_struct.narcs = 0;

            // Computing the prefix sum (exclusive)
            BlockScan(temp_storage).ExclusiveScan(block_prefix_sum_token_arc_count, 
                                                    block_prefix_sum_token_arc_count, 
                                                    zero_struct,
                                                    TokenAndArcCountSum());

            
            TokenAndArcCountUnion token_and_arc_count_block_sum;
            if(threadIdx.x == (KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX-1)) {
                // This conditional branch is entered by the last thread
                // because it is the last, the prefix_sum of that thread contains the sum of all elts

                // We also add the value from this thread - the prefix sum is exclusive
                token_and_arc_count_block_sum.split.ntokens = block_prefix_sum_token_arc_count.ntokens + (is_pruned ? 0 : 1);
                token_and_arc_count_block_sum.split.narcs = block_prefix_sum_token_arc_count.narcs + degree;

                nsurvival_tokens_in_CTA = token_and_arc_count_block_sum.split.ntokens;
                
                // Doing two things at the same time :
                // requesting a spot in the main_q to store the survival tokens from this CTA 
                // (we need space for token_and_arc_count_block_sum.split.ntokens tokens)
                // informing the main_q that our survival tokens contain token_arc_count_block_sum.split.narcs arcs
                //
                // We then store the return value, which is the global offset on where to store those tokens,
                // and the total number of arcs up until that global offset
                main_q_global_block_offset.both = atomicAdd(&params.d_main_q_end_and_narcs_i2->both, token_and_arc_count_block_sum.both);
            }

            // Syncing for three reasons :
            // - Broadcasting main_q_global_block_offset
            // - Broadcasting nsurvival_tokens_in_CTA
            // - We may reuse temp_storage (cf CUB doc)
            __syncthreads(); 

            // Checking if we are overflowing the main_q
            if((main_q_global_block_offset.split.ntokens + nsurvival_tokens_in_CTA) >= params.q_capacity) {
                if(threadIdx.x == (KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX-1)) {
                    // We are overflowing the main_q
                    // We first revert what this CTA has done, ie revert the previous atomicAdd
                    // because all CTAs will revert, we know we will have a valid state after completion of this kernel
                    atomicAdd(&params.d_main_q_end_and_narcs_i2->both, -token_and_arc_count_block_sum.both); // revert

                    // Setting the flag. It will print a warning to stderr
                    *params.h_q_overflow = 1;
                }

                // We abort computation, we no longer have space in the main_q.
                // We still jump to finalize_kernel, to do what's needed before completion
                goto finalize_kernel;
            }

            // If we are executing the following lines it means that we are not overflowing the queue
            // We then continue what we were doing

            if(!is_pruned) {
                // This thread is in charge of a survival token
                // we will move it to the main_q, at index main_q_idx

                // Note : we could remove the branch divergence here 

                int32 main_q_idx = main_q_global_block_offset.split.ntokens + block_prefix_sum_token_arc_count.ntokens;

                InfoToken token_info = params.d_aux_q_info[aux_q_idx];

                // Moving the token to the main q
                params.d_main_q_state[main_q_idx] = token_state;
                params.d_main_q_cost[main_q_idx] = token_cost;
                params.d_main_q_info[main_q_idx] = token_info;

                // Saving the global prefix sum
                // = (narcs until now in the main queue) + (narcs until this thread in the CTA)
                params.d_main_q_degrees_prefix_sum[main_q_idx] = main_q_global_block_offset.split.narcs 
                                                                 + block_prefix_sum_token_arc_count.narcs;

                // Saving the CSR arc offset for that token's state
                // it will be used by the expand kernel, and avoid doing a new random memory access in the expand kernel
                params.d_main_q_arc_offsets[main_q_idx] = arc_start;
            }
        }

        finalize_kernel:

        // Avoiding races 
        // We will write d_aux_q_end
        // And some threads may be still reading it 
        // At the beg of this kernel
        __syncthreads();
        
        if(threadIdx.x == 0) {
            // Declaring the CTA as done
            int32 old = atomicAdd(params.d_n_CTA_done, 1);

            // If we're the last CTA to exit, detect it
            bool is_last_CTA = (old == (gridDim.x -1));

            if(is_last_CTA) {
                __threadfence();

                // We added things to the main_q
                // d_main_q_end was modified
                // we update h_main_q_end to keep it consistent
                // the h_* pointers are in the pinned host memory, we can access them from the device
                *params.h_main_q_end = *params.d_main_q_end;
                *params.h_main_q_narcs = *params.d_main_q_narcs;

                // We moved what we had to move from the aux q to the main q
                // We now empty the aux q 
                *params.d_aux_q_end = 0;
                *params.h_aux_q_end = 0; 

                // Reset the counter for next time
                *params.d_n_CTA_done = 0;
            }
        }

    }


    void CudaDecoder::PreprocessAndContract(const PreprocessParams &params) {
        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX;
        grid.x = DIV_ROUND_UP(*h_aux_q_end_, block.x);

        KALDI_ASSERT(grid.x > 0);

        _preprocess_and_contract_kernel<<<grid,block,0,compute_st_>>>(params);
    }



/*
    PreprocessInPlace
    This kernel is also a preprocessing kernel, but this time does it in place
    ie it will not move tokens from the aux_q to the main_q
    It will do the preprocess operation directly on the main_q
    The tokens are already in the main q (they were placed here by a previous "contract and preprocess").

    We cannot prune non-optimal tokens, because the tokens are already in the main_q (we cannot prune 
    the main_q - it would break the prev_token indexes). To avoid doing unnecessary computation 
    in the expand kernel, we simulate the pruning by setting non-optimal token's degree to 0
    We then rely on the 1 thread = 1 arc exact load balacing of expand to ignore that token

    Please note that even if 0 threads will perform work on an ignored token in expand (degree = 0),
    it is not exactly the same as pruning it : the main_q accesses will not be perfectly coalesced
    in expand, because some "dead" tokens exist between living ones

    For the preprocess stage we have to compute the prefix sum of the tokens arc degrees
    Here we have to do the prefix sum in two passes : first local prefix sums inside CUDA block,
    then in a second kernel (finalize_preprocess_in_place), we add the necessary block offsets to end up 
    with the global prefix sum

    This preprocess step is used in ProcessEmitting. Tokens were placed in main_q by
    the ProcessNonEmitting of the previous frame. We cannot renumber them (it would break
    the prev_token index). We preprocess in place, leaving things as they are in main_q

*/

    __global__ void _preprocess_in_place_kernel(PreprocessParams params) {
   
        // Operator for the prefix sum inside the CUDA block
        typedef hipcub::BlockScan<int32, KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX> BlockScan;
        __shared__ typename BlockScan::TempStorage temp_storage;


        // All threads in the last CUDA block (CTA) alive will have work to do at the end
        // this bool will be needed to broadcast the information from thread0 to all threads in the last CTA 
        __shared__ bool is_last_CTA;

        const int32 main_q_offset = *params.d_main_q_local_offset;
        const int32 main_q_end = *params.d_main_q_end;
        const int32 main_q_size = main_q_end - main_q_offset;

        // Final cutoff from the expand kernel
        const BaseFloat cutoff = *params.d_cutoff;

        // The condition of the for loop is the same for all threads in the CUDA block
        // we want to keep all threads alive at the same time for now
        // otherwise __syncthreads() would fail
        for(int32 block_offset = blockDim.x*blockIdx.x;
                block_offset < main_q_size;
                block_offset += gridDim.x*blockDim.x) {

            // Position of considered token in the main_q
            int32 main_q_idx = main_q_offset + block_offset + threadIdx.x; 

            // Total number of arcs from that token's state
            int32 degree = 0; 

            if(main_q_idx < main_q_end) {
                StateId token_state = params.d_main_q_state[main_q_idx]; 
                BaseFloat token_cost = params.d_main_q_cost[main_q_idx];

                // the cutoff may have decreased since the creation of that token
                if(token_cost < cutoff) {

                    // Best cost for that token_state
                    // We know we have a token associated with token_state in the queue with the cost state_best_cost
                    BaseFloat state_best_cost = orderedIntToFloat(params.d_state_cost[token_state]); 
                    
                    // We can have duplicates, ie token associated with the same states
                    // If this token is not the best candidate, get rid of it
                    if(token_cost == state_best_cost) {
                        int32 start = params.d_arc_offsets[token_state]; 
                        int32 end = params.d_arc_offsets[token_state+1]; 
                        degree  = end - start;
                        
                        // Saving the start offset for the expand kernel
                        // avoid a new random memory access
                        params.d_main_q_arc_offsets[main_q_idx] = start;
                    }
                }
            }

            int32 degree_local_prefix_sum;

            // Computing a local prefix sum inside that CUDA block
            // A second kernel will take care of adding the necessary offset to those local prefix sums
            BlockScan(temp_storage).ExclusiveSum(degree, degree_local_prefix_sum);

            if(main_q_idx < main_q_end) {
                // This is not the final global prefix sum
                // A second kernel will add the necessary offset
                params.d_main_q_degrees_prefix_sum[main_q_idx] = degree_local_prefix_sum; 
            }

            if(threadIdx.x == (KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX-1)) {
                // Saving the local sum of degrees of that CUDA block
                // That's necessary to compute the global offset of that CUDA block,
                // and that offset is what we need to transform the local prefix sum into a global prefix sum

                int local_sum_index = block_offset/KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX;
                int local_sum = degree_local_prefix_sum + degree; // the prefix sum was exclusive, adding missing value
                params.d_main_q_degrees_block_prefix_sum[local_sum_index] = local_sum; 
            }


            // Synchronization for two reasons :
            // - we may need to reuse temp_storage if the for loop iterates (cf CUB's doc)
            // - we need all threads to be done before considering the CTA as done (see below)
            __syncthreads(); 

        }

        //
        // The last CUDA block alive will compute the prefix sum of the block degrees sum
        // We need that prefix sum, because it represents the offsets that each CUDA block has in the global prefix sum
        // we will then add those offsets in finalize_preprocess_in_place

        if(threadIdx.x == 0) {
            // We indicate that this CTA is done
            int32 old = atomicAdd(params.d_n_CTA_done, 1); 
            
            // If we're the last CTA to exit, detect it
            is_last_CTA = (old == (gridDim.x -1));
        }

        // Synchronization for two reasons :
        // - Broadcasting is_last_CTA
        // - reusing temp_storage (cf CUB's doc)
        __syncthreads();
        
        if(is_last_CTA)
        {
            //
            // Our goal here is to compute the prefix sum of the previous local sums
            // What we call local sum is what contains the local_sum variables in the previous lines
            // it is the sum of degrees inside a given CUDA block, at a given for loop iteration
            // all local sums are stored in params.d_main_q_degrees_block_prefix_sum
            // we want to do the prefix sum of that array
            //
            // Once this is done, params.d_main_q_degrees_block_prefix_sum[i] will contain the 
            // offset that we need to add to the local prefix sum #i to convert it to a global
            // prefix sum
            // Right now we are only computing the offsets ; adding them to the local prefix sums will be 
            // done in FinalizePreprocessInPlace
            //

            //
            // We are the last CTA alive
            // which means that all local sums have been written to params.d_main_q_degrees_block_prefix_sum
            // We can now do the prefix sum of that array   
            //

            // Making sure that we see changes from other CTAs 
            __threadfence();

            //
            // How many local sums values do we have ?
            // Please note that this number can be different from gridDim.x
            // We may have applied a upper limit on gridDim.x, and in that case
            // gridDim.x < number_of_local_sums
            //

            int32 number_of_local_sums = DIV_ROUND_UP(main_q_size, KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX);

            // We may iterate the following for loop multiple times
            // on iteration > 0, we will have to consider the offset from previous iterations
            int32 prefix_sum_of_local_sums_offset = 0;

            // local_sum_index is an index in the array d_main_q_degrees_block_prefix
            // 
            // The condition inside the loop is common to all threads in the CTA
            // we want to keep all threads active, we will use syncthreads()
            for(int32 local_sum_index_offset = 0; 
                      local_sum_index_offset < number_of_local_sums; 
                      local_sum_index_offset += blockDim.x) {

                int32 local_sum_index = local_sum_index_offset + threadIdx.x; 

                int32 local_sum = (local_sum_index < number_of_local_sums) 
                                ? params.d_main_q_degrees_block_prefix_sum[local_sum_index] 
                                : 0; // neutral element

                int32 prefix_sum_of_local_sums, total_sum_of_local_sums_for_this_iteration;

                BlockScan(temp_storage).ExclusiveSum(local_sum, prefix_sum_of_local_sums, total_sum_of_local_sums_for_this_iteration);

                prefix_sum_of_local_sums += prefix_sum_of_local_sums_offset;
                prefix_sum_of_local_sums_offset += total_sum_of_local_sums_for_this_iteration;

                if(local_sum_index < number_of_local_sums) {
                    params.d_main_q_degrees_block_prefix_sum[local_sum_index] = prefix_sum_of_local_sums;
                }

                // Sync'ing to be able to reuse temp_storage (cf CUB's doc)
                __syncthreads();
            }

            if(threadIdx.x == 0)
            {
                // Final offset is the overall total
                int total_sum_of_local_sums = prefix_sum_of_local_sums_offset;
                *params.d_main_q_narcs = total_sum_of_local_sums; 
                // h_main_q_narcs is in pinned memory, we can write to it from the device
                *params.h_main_q_narcs = total_sum_of_local_sums; 
                // reset for next time
                *params.d_n_CTA_done = 0;
            }
        }
    }


    void CudaDecoder::PreprocessInPlace(const PreprocessParams &params) {
        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX;
        int32 main_q_size = *h_main_q_end_ - *h_main_q_local_offset_;

        grid.x = DIV_ROUND_UP(main_q_size, block.x);

        KALDI_ASSERT(grid.x > 0);

        _preprocess_in_place_kernel<<<grid,block,0,compute_st_>>>(params);
    }



    /*

       Part 2 of the scan for "PreprocessEmitting". For NonEmitting scan is already final

       Computes global prefix sum with block prefix sum and block offsets

       If we want to speed up expand, we can compute lower and upper bound to restrain 
       the binary search in expand
       This can be done on the fly here, and removes main bottleneck of expand
       Not done for now, because expand is fast enough

     */
    __global__ void _finalize_degrees_scan_kernel(int32 *d_scan, int32 *d_blk_scan, const int32 *d_main_q_local_offset_, const int32
            *d_main_q_end_) {

        int32 q_off = *d_main_q_local_offset_;
        int32 q_end = *d_main_q_end_;
        int32 q_size = q_end - q_off;

        for(int32 idx = q_off + blockDim.x*blockIdx.x + threadIdx.x;
                idx < q_size;
                idx += blockDim.x*gridDim.x) {

            int32 blk_idx = (idx - q_off) / KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX;
            int32 blk_scan_offset = d_blk_scan[blk_idx]; // we rely on L1 for this one, avoiding syncs

            d_scan[idx] += blk_scan_offset;
        }

    }

    void CudaDecoder::FinalizePreprocessInPlace() {
        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_KERNEL_PREPROCESS_DIMX;
        int32 main_q_size = *h_main_q_end_ - *h_main_q_local_offset_;
        grid.x = DIV_ROUND_UP(main_q_size, block.x);

        // If the main_q is empty, we will not be able to continue
        KALDI_ASSERT(grid.x > 0);

        _finalize_degrees_scan_kernel<<<grid,block,0,compute_st_>>>(d_main_q_degrees_prefix_sum_, d_main_q_degrees_block_prefix_sum_, d_main_q_local_offset_,
                d_main_q_end_); 
    }




    /*
       This kernel propagates arcs from the main q [main_q_local_offset, main_q_end[
       to the aux

       The main bottleneck is the first binary search. 
       If we want to remove it, preprocess it on the fly in preprocess

     */

    struct CostTInt {
        CostType cost;
        int32 i;
    };

    struct CISum {
        __device__ CostTInt operator()(const CostTInt &a, const CostTInt &b) const {
            CostTInt c;
            c.cost = fmin(a.cost, b.cost);
            c.i = a.i + b.i;

            return c;
        }
    };


__device__ __inline__ CostType GetCutoffCandidate(const CostType current_cutoff,
                                const CostType min_cost,
                                const CostType default_beam,
                                const int32 q_size,
                                const int32 q_capacity) {
                                 

    // Doing something simple for now
    // We have to keep beam large enough,
    // the final cutoff will be used for the final
    // prune. If it is too small, we won't keep enough tokens

   CostType beam = default_beam;

   if(q_size >= q_capacity/2) 
       beam /= 2;

    return fmin(current_cutoff, min_cost + beam);
}

    __forceinline__ __device__ int32 binsearch_maxle(const int32 *vec, const int32 val, int32 low, int32 high) {
        while(true) {
            if(low == high)
                return low; //we know it exists
            if((low + 1) == high)
                return (vec[high] <= val) ? high : low;

            int32 mid = low + (high- low) / 2;

            if(vec[mid] > val)
                high = mid-1;
            else
                low = mid;
        }
    }


    void __global__ _expand_arcs_kernel(ExpandArcParams params) {
        typedef hipcub::BlockScan<CostTInt, KALDI_CUDA_DECODER_KERNEL_EXPAND_ARCS_DIMX> BlockScan;

        __shared__ typename BlockScan::TempStorage temp_storage_scan;

        __shared__ int32 to_q_block_offset;
        __shared__ CostType blk_cutoff;

        const int32 total_narcs = *params.d_main_q_narcs;
        const int32 main_q_offset = *params.d_main_q_local_offset;
        const int32 main_q_end = *params.d_main_q_end;

        
        if(threadIdx.x == 0) {
            blk_cutoff = *params.d_cutoff;
        }

        __syncthreads();

        // Keeping the whole CTA alive, we'll have syncs
        for(int32 block_offset = blockDim.x*blockIdx.x;
                block_offset < total_narcs;
                block_offset += gridDim.x*blockDim.x) {

            int32 th_idx = block_offset + threadIdx.x;
            bool valid_input = (th_idx < total_narcs);

            BaseFloat total_cost = FLT_MAX;
            int32 arc_idx;
            StateId arc_next_state;
            int32 main_q_idx;

            if(valid_input) {
                //we can do better than that
                main_q_idx = binsearch_maxle(params.d_main_q_degrees_prefix_sum, th_idx, main_q_offset, main_q_end-1); 

                int32 lower_bound = params.d_main_q_degrees_prefix_sum[main_q_idx];
                int32 arc_offset_start = params.d_q_arc_offsets[main_q_idx];

                arc_idx = arc_offset_start + (block_offset + threadIdx.x - lower_bound);
                arc_next_state = params.arc_nextstates[arc_idx];

                total_cost = params.arc_weights[arc_idx];

                int32 arc_ilabel = params.is_emitting ? params.arc_ilabels[arc_idx] : 0;
                total_cost += (arc_ilabel != 0) ? -params.d_loglikelihoods[arc_ilabel] : 0.0; 
                total_cost += params.d_main_q_cost[main_q_idx];

                if(total_cost >= blk_cutoff)
                    valid_input = false;
                else {
                    // switch back to red, worst case is bad
                    BaseFloat next_state_cost = orderedIntToFloat(params.d_lookup[arc_next_state]);

                    if(total_cost >= next_state_cost)
                        valid_input = false;
                }
            }

                            int32 has_successor = valid_input ? 1 : 0;  // Need a spot in the new q
                            CostTInt ci;
                            ci.cost = valid_input ? total_cost : FLT_MAX; 
                            ci.i = has_successor;

                            BlockScan(temp_storage_scan).InclusiveScan(ci, ci, CISum());

                            if(threadIdx.x == (KALDI_CUDA_DECODER_KERNEL_EXPAND_ARCS_DIMX - 1)) {
                                int32 total_successors_in_block = ci.i;
                                to_q_block_offset = atomicAdd(params.d_aux_q_end, total_successors_in_block);
                                if((to_q_block_offset + total_successors_in_block) >= params.q_capacity) {
                                    to_q_block_offset = params.q_capacity; // used to broadcast the info

                                }
                                /*
                                
                                GetCutoffCandidate takes int32o account the current value of 
                                d_aux_q_end and compares it with its maximum capacity.
                                If necessary it progressively cuts down the beam 
                                (reducing the cutoff) to only keep the best candidates
                                and avoiding an overflow

                                */
                                CostType cutoff_candidate = GetCutoffCandidate(blk_cutoff,
                                                                  ci.cost,
                                                                  params.beam,
                                                                  to_q_block_offset + total_successors_in_block,
                                                                  params.q_capacity);

                                blk_cutoff = (cutoff_candidate < blk_cutoff) 
                                             ? fmin(fatomicMin(params.d_cutoff, cutoff_candidate), cutoff_candidate)
                                             : fmin(*params.d_cutoff, blk_cutoff);
                            }

                            __syncthreads(); // to_q_block_offset


                            // aux_q is full. UpdateCutoff should prevent this from happening
                            if(to_q_block_offset == params.q_capacity) {
                                if(threadIdx.x == (KALDI_CUDA_DECODER_KERNEL_EXPAND_ARCS_DIMX - 1)) {
                                    // Revert
                                    int32 total_successors_in_block = ci.i;
                                    atomicAdd(params.d_aux_q_end, -total_successors_in_block); 
                                    *params.h_q_overflow = 1; 
                                }

                                goto finalize_kernel; // keeping things clean before aborting
                            }

                            ci.i -= has_successor; // we want the exclusive sum now
                            int32 to_q_index = to_q_block_offset + ci.i;

                            if(has_successor) {
                                params.d_aux_q_cost[to_q_index] = total_cost;
                                params.d_aux_q_state[to_q_index] = arc_next_state;
                                
                                atomicMin(&params.d_lookup[arc_next_state],
                                floatToOrderedInt(total_cost)
                                );

                                //print32f("cost = %f, cutoff = %f, beam=%f \n", total_cost, blk_cutoff, params.beam);
                                if(total_cost < blk_cutoff) { // cutoff may have changed
                                    // We write the rest of the token only if necessary
                                    // if the cost is higher than cutoff, 
                                    // the token will be ignored anyway 


                                    InfoToken new_tok_info;
                                    new_tok_info.prev_token = params.main_q_global_offset + main_q_idx;
                                    new_tok_info.arc_idx = arc_idx;
                            

                                    params.d_aux_q_info[to_q_index] = new_tok_info;

                                    /*
                                    print32f("expand, adding %i (%i)  -> %i \n", new_tok_info.prev_token,
                                    params.main_q_global_offset, arc_next_state);
                                    */
                                }
                            }
        }

        finalize_kernel:

        __syncthreads(); // avoiding races on d_main_q_narcs for instance

        // Last block alive sets h_aux_q_end_ (pinned memory)
        if(threadIdx.x == 0) {
            int32 old = atomicAdd(params.d_n_CTA_done, 1);
            if(old == (gridDim.x -1)) {
                __threadfence(); // we want last value of d_aux_q_end
                *params.h_aux_q_end = *params.d_aux_q_end;
                *params.d_n_CTA_done = 0;
                *params.d_main_q_narcs = 0;
                *params.h_main_q_narcs = 0;

                if(params.is_emitting) {
                    *params.d_main_q_local_offset = 0; // not needed
                    *params.h_main_q_local_offset = 0; // not needed
                    *params.d_main_q_end = 0;
                    *params.h_main_q_end = 0;
                } else {
                    *params.d_main_q_local_offset = main_q_end;
                    *params.h_main_q_local_offset = main_q_end;
                }

            }
        }

    }

    void CudaDecoder::ExpandArcs(const ExpandArcParams &params, int32 nthreads) {
        dim3 grid,block;
        block.x = 256;
        grid.x = DIV_ROUND_UP(nthreads, block.x);

        // It's possible to have zero threads and still be valid
        if(grid.x > 0)
            _expand_arcs_kernel<<<grid,block,0,compute_st_>>>(params);
    }


    // Wrote for single CTA

    /*

       Persistent kernel

       Used to avoid calling multiple "heavy lifting" kernels for the tail of non emitting
       (lots of iterations with small number of arcs)

       Code is greatly simplified because we can have only one CTA alive

       Repeat until new queue empty:
       1) Computes degrees (cf ComputeDegrees) 
       2) Compute scan
       3) Expand arcs

       1 and 2 are not done on the first iteration, because it's already done
       (by corresponding kernels)

       At the end, this kernel finalize the computation for current frame,
       so that it's ready for next ProcessEmitting

       We could optimize and speed up this kernel
       It will only gives us a better latency for 1 stream, which is low enough
       Instead, we let it compute while we use the GPU for other streams
       This kernel only uses one block

     */


    __launch_bounds__(KALDI_CUDA_DECODER_KERNEL_NONEM_LT_DIMX, 1)
        __global__ void _process_nonem_longtail(const uint32_t *d_arc_offsets, 
                ExpandArcParams params) {

            typedef hipcub::BlockScan<int32, KALDI_CUDA_DECODER_KERNEL_NONEM_LT_DIMX> BlockScan;
            typedef hipcub::BlockReduce<float, KALDI_CUDA_DECODER_KERNEL_NONEM_LT_DIMX> BlockReduce;

            __shared__ typename BlockScan::TempStorage temp_storage_scan;
            __shared__ typename BlockReduce::TempStorage temp_storage_reduce;

            __shared__ BaseFloat cutoff;


            int32 old_q_offset = *params.d_main_q_local_offset;
            int32 new_q_offset = *params.d_main_q_end;
            int32 new_q_end = new_q_offset;

            int32 total_narcs = *params.d_main_q_narcs;
    
            int32 old_q_size = new_q_offset - old_q_offset;  // move to end

            cutoff = *params.d_cutoff;

            // We'll switch queue at the beg of the loop
            // Cleaner that way - we need the offsets ready for
            // the global updates at the very end of this kernel
            new_q_offset = old_q_offset;

            bool first = true;

            while(old_q_size > 0) {
                // Step 0 : move queues        
                old_q_offset = new_q_offset;
                new_q_offset = new_q_end;

                if(!first) {
                    __syncthreads(); // old_q_ready
                    total_narcs = 0;

                    // Step 1 : compute_degrees
                    // TODO fuse 1 and 2
                    for(int32 q_idx = old_q_offset + threadIdx.x;
                            q_idx < new_q_offset; // = old_q_end
                            q_idx += blockDim.x) {

                        StateId state = params.d_main_q_state[q_idx];
                        BaseFloat cost = params.d_main_q_cost[q_idx];

                        int32 degree = 0;
                        if(cost < cutoff) {
                            BaseFloat best_cost = orderedIntToFloat(params.d_lookup[state]);

                            if(cost == best_cost) {
                                int32 start = d_arc_offsets[state];
                                int32 end = d_arc_offsets[state+1];
                                degree = end - start;
                                params.d_q_arc_offsets[q_idx] = start;
                            }
                        }

                        params.d_main_q_degrees_prefix_sum[q_idx] = degree;
                    }

                    __syncthreads(); // will be removed

                    // Step 2 : Scan

                    for(int32 block_off = 0;
                            block_off < old_q_size;
                            block_off += blockDim.x) {

                        int32 q_idx = old_q_offset + block_off + threadIdx.x;

                        int32 degree = (q_idx < new_q_offset) 
                            ? params.d_main_q_degrees_prefix_sum[q_idx]
                            : 0;
                        int32 lscan;
                        int32 total_in_blk;
                        BlockScan(temp_storage_scan).ExclusiveSum(degree, lscan, total_in_blk);
                        int32 scan = lscan + total_narcs;
                        total_narcs += total_in_blk;

                        if(q_idx < new_q_offset)
                            params.d_main_q_degrees_prefix_sum[q_idx] = scan;

                         __syncthreads(); // reusing temp_storage_scan + degrees ready
                    }


                } else {
                    first = false;    
                }


                // We already sync'ed

                // Step 3 : expand arcs

                for(int32 block_offset = 0;
                        block_offset < total_narcs;
                        block_offset += blockDim.x) {

                    int32 th_idx = block_offset + threadIdx.x;
                    bool valid_input = (th_idx < total_narcs);

                    BaseFloat total_cost = FLT_MAX;
                    int32 arc_idx;
                    StateId arc_next_state;
                    int32 q_idx;

                    if(valid_input) {
                        //we can do better than that
                        q_idx = binsearch_maxle(params.d_main_q_degrees_prefix_sum, th_idx, old_q_offset, new_q_offset-1); 

                        int32 lower_bound = params.d_main_q_degrees_prefix_sum[q_idx];
                        int32 arc_offset_start = params.d_q_arc_offsets[q_idx];

                        arc_idx = arc_offset_start + (th_idx - lower_bound);

                        arc_next_state = params.arc_nextstates[arc_idx];
                        BaseFloat arc_weight = params.arc_weights[arc_idx];
                        BaseFloat next_state_cost = orderedIntToFloat(params.d_lookup[arc_next_state]);
                        BaseFloat old_tok_cost = params.d_main_q_cost[q_idx];

                        total_cost = arc_weight + old_tok_cost;

                        if(total_cost >= next_state_cost) {
                            total_cost = FLT_MAX;
                            valid_input = false; 
                        } 
                    }

                    BaseFloat min_cost = BlockReduce(temp_storage_reduce).Reduce(total_cost, hipcub::Min());

                    if(threadIdx.x == 0) {
                        cutoff = GetCutoffCandidate(cutoff,
                                min_cost,
                                params.beam,
                                new_q_end,
                                params.q_capacity);
                    }

                    __syncthreads();

                    int32 has_successor = (total_cost < cutoff && valid_input) ? 1 : 0;

                    if(has_successor) 
                        atomicMin(&params.d_lookup[arc_next_state], floatToOrderedInt(total_cost));

                    int32 new_q_idx_block = has_successor;
                    int32 total_in_blk;
                    BlockScan(temp_storage_scan).ExclusiveSum(new_q_idx_block, new_q_idx_block, total_in_blk);

                    if((new_q_end + total_in_blk) >= params.q_capacity) {
                        *params.h_q_overflow = 1;
                        
                        goto finalize_kernel; // keeping things clean before aborting
                    }

                    if(has_successor) {
                        int32 new_q_index = new_q_end + new_q_idx_block;
                        params.d_main_q_state[new_q_index] = arc_next_state;

                        params.d_main_q_cost[new_q_index] = total_cost;

                        InfoToken new_tok_info;
                        new_tok_info.prev_token = params.main_q_global_offset + q_idx;

                        new_tok_info.arc_idx = arc_idx;
                        params.d_main_q_info[new_q_index] = new_tok_info;
                        
                        //print32f("new q index = %i (%i+%i) (tot=%i) \n", new_q_index, new_q_end, new_q_idx_block,
                        //total_in_blk);
                   }

                    new_q_end += total_in_blk;
                }

                old_q_size = new_q_end - new_q_offset; 
            }

            finalize_kernel:

            if(threadIdx.x == 0) {
                // Next step is ProcessEmitting of next frame, from is currToken_offset
                *params.d_main_q_end = new_q_end; 
                *params.d_main_q_narcs = 0;

                *params.h_main_q_end = new_q_end; 
                *params.h_main_q_narcs = 0; 

                *params.d_main_q_local_offset = 0; 
                *params.h_main_q_local_offset = 0; 

                *params.d_cutoff = cutoff;
            }

        }

    void CudaDecoder::NonEmittingLongTail(const uint32_t *d_arc_offsets, 
            const ExpandArcParams &params) {

        dim3 grid,block;
        block.x = KALDI_CUDA_DECODER_KERNEL_NONEM_LT_DIMX;
        grid.x = 1; // it is designed for the long tail
        _process_nonem_longtail<<<grid,block,0,compute_st_>>>(d_arc_offsets, params);
    }


} // end namespace kaldi
