// decoder/cuda-decoder-utils.cu

// 2018 - Hugo Braun, Justin Luitjens, Ryan Leary

// See ../../COPYING for clarification regarding multiple authors
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//  http://www.apache.org/licenses/LICENSE-2.0
//
// THIS CODE IS PROVIDED *AS IS* BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, EITHER EXPRESS OR IMPLIED, INCLUDING WITHOUT LIMITATION ANY IMPLIED
// WARRANTIES OR CONDITIONS OF TITLE, FITNESS FOR A PARTICULAR PURPOSE,
// MERCHANTABLITY OR NON-INFRINGEMENT.
// See the Apache 2 License for the specific language governing permissions and
// limitations under the License.


#include "decoder/cuda-decoder-utils.h"
#include <nvToolsExt.h>

namespace kaldi {

    /***************************************CudaFst Implementation*****************************************/

    void CudaFst::initialize(const fst::Fst<StdArc> &fst) {
        nvtxRangePushA("CudaFst constructor");

        //count states since Fst doesn't provide this functionality
        numStates=0;
        for( fst::StateIterator<fst::Fst<StdArc> > iter(fst); !iter.Done(); iter.Next()) {
            numStates++;
        }
        start=fst.Start();
        hipHostMalloc(&h_final,sizeof(float)*numStates);

        //allocate and initialize offset arrays
        hipHostMalloc(&h_e_offsets, (numStates+1)*sizeof(unsigned int));
        hipHostMalloc(&h_ne_offsets, (numStates+1)*sizeof(unsigned int));

        hipMalloc((void**)&d_e_offsets,sizeof(unsigned int)*(numStates+1));
        hipMalloc((void**)&d_ne_offsets,sizeof(unsigned int)*(numStates+1));
 
       //iterate through states and arcs and count number of arcs per state
        e_count=0;
        ne_count=0;
        max_ilabel=0;
       
        // Init first offsets
        h_ne_offsets[0] = 0; 
        h_e_offsets[0] = 0; 

        for(int i=0;i<numStates;i++) {
            h_final[i]=fst.Final(i).Value();

            //count emiting and non_emitting arcs
            for (fst::ArcIterator<fst::Fst<StdArc> > aiter(fst, i); !aiter.Done(); aiter.Next()) {
                StdArc arc = aiter.Value();
                int32 ilabel = arc.ilabel;
                int32 olabel = arc.olabel;

                if(ilabel>max_ilabel) {
                    max_ilabel=ilabel;
                }

                if(ilabel!=0) { //emitting
                    e_count++;
                } else { //non-emitting
                    ne_count++;
                }
            }
            h_ne_offsets[i+1]=ne_count;
            h_e_offsets[i+1]=e_count;
        }

        // We put the emitting arcs before the nonemitting arcs in the arc list
        // adding offset to the non emitting arcs
        // we go to numStates+1 to take into account the last offset
        for(int i=0;i<numStates+1;i++) 
            h_ne_offsets[i]+=e_count;   //e_arcs before

        arc_count=e_count+ne_count;

        hipMemcpy(d_e_offsets,h_e_offsets,sizeof(unsigned int)*(numStates+1),hipMemcpyHostToDevice);
        hipMemcpy(d_ne_offsets,h_ne_offsets,sizeof(unsigned int)*(numStates+1),hipMemcpyHostToDevice);

        hipHostMalloc(&h_arc_weights,arc_count*sizeof(BaseFloat));
        hipHostMalloc(&h_arc_nextstates,arc_count*sizeof(StateId));
        hipHostMalloc(&h_arc_ilabels,arc_count*sizeof(int32));
        hipHostMalloc(&h_arc_olabels,arc_count*sizeof(int32));

        hipMalloc(&d_arc_weights,arc_count*sizeof(BaseFloat));
        hipMalloc(&d_arc_nextstates,arc_count*sizeof(StateId));

        // Only the ilabels for the e_arc are needed on the device
        hipMalloc(&d_arc_ilabels,e_count*sizeof(int32)); 
        // We do not need the olabels on the device - GetBestPath is on CPU

        //now populate arc data
        int e_idx=0;
        int ne_idx=e_count; //starts where e_offsets ends

        for(int i=0;i<numStates;i++) {
            for (fst::ArcIterator<fst::Fst<StdArc> > aiter(fst, i); !aiter.Done(); aiter.Next()) {
                StdArc arc = aiter.Value();
                int idx;
                if(arc.ilabel!=0) { //emitting
                    idx=e_idx++;
                } else {
                    idx=ne_idx++;
                }
                h_arc_weights[idx]=arc.weight.Value();
                h_arc_nextstates[idx]=arc.nextstate;
                h_arc_ilabels[idx]=arc.ilabel;
                h_arc_olabels[idx]=arc.olabel;
            }
        }

        hipMemcpy(d_arc_weights,h_arc_weights,arc_count*sizeof(BaseFloat),hipMemcpyHostToDevice);
        hipMemcpy(d_arc_nextstates,h_arc_nextstates,arc_count*sizeof(StateId),hipMemcpyHostToDevice);
        hipMemcpy(d_arc_ilabels,h_arc_ilabels, e_count*sizeof(int32),hipMemcpyHostToDevice);
        
        // Making sure the graph is ready
        hipDeviceSynchronize();

        cudaCheckError();

        nvtxRangePop();
    }

    void CudaFst::finalize() {
        nvtxRangePushA("CudaFst destructor");
        hipHostFree(h_final);
        hipHostFree(h_e_offsets);
        hipHostFree(h_ne_offsets);

        hipFree(d_e_offsets);
        hipFree(d_ne_offsets);

        hipHostFree(h_arc_weights);
        hipHostFree(h_arc_nextstates);
        hipHostFree(h_arc_ilabels);
        hipHostFree(h_arc_olabels);

        hipFree(d_arc_weights);
        hipFree(d_arc_nextstates);
        hipFree(d_arc_ilabels);
        nvtxRangePop();
    }


    /***************************************End CudaFst****************************************************/


    // Constructor always takes an initial capacity for the vector
    // even if the vector can grow if necessary, it damages performance
    // we need to have an appropriate initial capacity (is set using a parameter in CudaDecoderConfig)
    InfoTokenVector::InfoTokenVector(int capacity) {
        capacity_ = capacity;
        KALDI_LOG << "Allocating InfoTokenVector with capacity = " << capacity_ << " tokens";
        hipHostMalloc(&h_data_, capacity_ * sizeof(InfoToken)); 
        SetCudaStream(0); // using default stream
        Reset();
    }

    void InfoTokenVector::Reset() {
        size_ = 0;
    };

    void InfoTokenVector::SetCudaStream(hipStream_t st) {
        copy_st_ = st;
    }

    void InfoTokenVector::CopyFromDevice(size_t offset, InfoToken *d_ptr, size_t count) {
        Reserve(size_+count); // making sure we have the space

        hipMemcpyAsync(&h_data_[offset], d_ptr, count*sizeof(InfoToken), hipMemcpyDeviceToHost, copy_st_);
        size_ += count;
    }

    void InfoTokenVector::Reserve(size_t min_capacity) {
        if(min_capacity <= capacity_)
            return;

        while(capacity_ < min_capacity)
            capacity_ *= 2;

        KALDI_LOG << "Reallocating InfoTokenVector on host (new capacity = " << capacity_ << " tokens).";

        hipStreamSynchronize(copy_st_);
        InfoToken *h_old_data = h_data_;
        hipHostMalloc(&h_data_, capacity_ * sizeof(InfoToken)); 

        if(!h_data_)
            KALDI_ERR << "Host ran out of memory to store tokens. Exiting.";

        if(size_ > 0)
            hipMemcpyAsync(h_data_, h_old_data, size_ * sizeof(InfoToken), hipMemcpyHostToHost, copy_st_);

        hipStreamSynchronize(copy_st_);
        hipHostFree(h_old_data);
    }

    InfoToken * InfoTokenVector::GetRawPointer() const {
        return h_data_;
    }

    InfoTokenVector::~InfoTokenVector() {
        hipHostFree(h_data_);
    }

} // end namespace kaldi
